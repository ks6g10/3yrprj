#include "hip/hip_runtime.h"
/**
 em	 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <ctype.h>
#include <stdint.h>
#include <sys/types.h>
#include <assert.h>
#include <limits.h>
#include <float.h>


typedef uint32_t myint;
#define WORDSIZE (sizeof(myint)*8)
#define SIZE (1)
struct allocation {
	myint a[SIZE];
};

struct bid {
	struct allocation alloc;
	unsigned int id;
	unsigned int bin;
	unsigned int dummy;
	float offer;
	float average;
	struct bid * next;
	struct bid * prev;
};

struct configuration {
	struct allocation * allocation;
	unsigned int * bin;
	unsigned int * id;
	// each bids dummy, if any else is set to 0
	unsigned int * dummies;
	unsigned int * bin_count;
	unsigned int * max_offset;
	float * offer;
	float * average;
	// the allocation dummy
	unsigned int goods;
	unsigned int words;
	unsigned int bids;
	unsigned int dummy;
	unsigned int singletons;
	unsigned int * allocation_id;
	unsigned int * allocation_dummy;
	unsigned int allocation_id_index;
	float allocation_value;
};

unsigned int ints = 0;
/* unsigned int * MASK = void; */

unsigned int next_index(unsigned int a_index) {
	return __builtin_ffs(a_index) - 1;
//	tmp  &= ~(1 << index);
//	upper_bound +=  v(bins[x],pi_conf,bin_counts[x]);

}

void print_binary(unsigned int * allocation, unsigned int goods) {
	int x;
	for (x = goods - 1; x >= 0; x--) {
		printf("%u", !!(*allocation & (1 << x)));
	}
	printf("\n");
}

struct configuration * get_configuration(FILE * fp) {
	const char * s_goods = "goods";
	const char * s_bids = "bids";
	const char * s_dummy = "dummy";

	ssize_t read;
	char * line = NULL;
	size_t len = 0;
	int got_dummy = 0;
	unsigned int all = 0;
	unsigned int goods =0;
	unsigned int bids = 0;
	unsigned int dummy = 0;


	while ((read = getline(&line, &len, fp)) != -1 && !all) {
		if (line[0] == '%' || line[0] == '\n') {
			continue;
		}
		if (strncmp(line, s_goods, strlen(s_goods)) == 0) {
			goods = atoi(line + strlen(s_goods) + 1);
			printf("Number of goods %u\n", goods);
		} else if (strncmp(line, s_bids, strlen(s_bids)) == 0) {
			bids = atoi(line + strlen(s_bids) + 1);
			printf("Number of bids %u\n", bids);
		} else if (strncmp(line, s_dummy, strlen(s_dummy)) == 0) {
			dummy = atoi(line + strlen(s_dummy) + 1);
			got_dummy = 1;
			printf("Number of dummy %u\n", dummy);
//			ints = 1+(goods-1)/32;
		}
//			total_goods = goods + dummy;
		all = !!(goods && bids && got_dummy);
	}
	free(line);

	//if(goods <= 32) {
	struct configuration * ret = (struct configuration *) malloc(
					sizeof(struct configuration));


	//}

	ret->words = SIZE;
	return ret;
}

unsigned int * get_bincount(FILE * fp, struct configuration * conf,
		unsigned int * have_singelton) {
	unsigned int * bin_count = (unsigned int *) malloc(
			sizeof(int) * conf->goods);
	int x;
	for (x = 0; x < conf->goods; x++) {
		bin_count[x] = 0;
	}

	char * head = NULL;
	char * tail = NULL;
	char * line = NULL;
	size_t len = 0;
	ssize_t read;
	printf("hello1\n");
	while ((read = getline(&line, &len, fp)) != -1) {
		if (!isdigit(line[0])) {
			continue;
		}
		head = tail = line;
		int tab_count = 0;
		head++;
		while (tab_count < 3) {
			if (*head == '\t') {
				tab_count++;
				if (tab_count <= 2) {
					tail = head;
					head++;
				}
			} else {
				head++;
			}
		}
		int which_bin = strtol(tail, &head, 10);
		head++;
		bin_count[which_bin]++;
		printf("abin %u count %u\n", which_bin, bin_count[which_bin]);
		int goods_count = 1;
		while (*head != '#' && *head != '\0') {
			if (*head == '\t') {
				goods_count++;
			}
			head++;
		}
		if (goods_count == 1) {
			have_singelton[which_bin] = 1;
		}
//		printf("Bin %d count %u\n",which_bin,bin_count[which_bin]);
	}
	free(line);

	return bin_count;
}

int compare_int(const void* p1, const void* p2) {
//	struct bid b1 =
	float i1 = ((struct bid*) p1)->average;
	float i2 = ((struct bid*) p2)->average;
	assert(((struct bid*)p1)->bin == ((struct bid*)p2)->bin);
	return i1 > i2 ? -1 : i1 < i2 ? 1 : 0;

}

struct bid * remove_from_list(struct bid * curr, struct bid * root) {

	if (curr->prev) { // if current node is not the first in the list
		curr->prev->next = curr->next; // then point prev to the next in the list
	} else {
		root = curr->next;
	}
	if (curr->next) { //if current node is not last in the list
		curr->next->prev = curr->prev; // then point the next node to the prev
		assert(curr != curr->next->prev);
	}

	return root;
}

int get_next_best_good(struct configuration * conf, struct bid * curr) {
	int x;
	unsigned int total_goods_count[conf->goods];
	unsigned int numbids_count[conf->goods];
	for (x = 0; x < conf->goods; x++) {
		total_goods_count[x] = numbids_count[x] = 0;
	}

	while (curr) {
		int goods_count = 0;
		for (x = 0; x < SIZE; x++) {
			goods_count += __builtin_popcount((unsigned int)curr->alloc.a[x]);
		}
		for (x = 0; x < conf->goods; x++) {
			int word_index = x / WORDSIZE;
			int bit_index = x % WORDSIZE;
			int result = !!(curr->alloc.a[word_index] & (1 << bit_index));
			total_goods_count[x] += result * goods_count;
			numbids_count[x] += result;

		}
		curr = curr->next;
	}
	int min_pos = 0;
	float min = FLT_MAX;
	for (x = 0; x < conf->goods; x++) {
		float score = 0.0f;
		if (numbids_count[x]) {
			double avg = ((double) total_goods_count[x])
					/ ((double) numbids_count[x]);
			score = ((double) numbids_count[x]) / avg;
			//printf("x %d score %.3f\n",x,score);
			if (score < min) {
				min = score;
				min_pos = x;
			}
		}
	}

	return min_pos;
}

void allocate_all_bids(FILE * fp, struct configuration * conf,
		unsigned int * have_singelton, unsigned int * bin_count) {
	conf->allocation = (struct allocation *) malloc(
			sizeof(struct allocation) * conf->bids);

	conf->id = (unsigned int *) malloc(sizeof(unsigned int) * conf->bids);
	conf->dummies = (unsigned int *) malloc(sizeof(unsigned int) * conf->bids);
	conf->bin = (unsigned int *) malloc(sizeof(unsigned int) * conf->bids);
	conf->offer = (float *) malloc(sizeof(float) * conf->bids);
	conf->average = (float *) malloc(sizeof(float) * conf->bids);
	conf->max_offset = (unsigned int *) malloc(
			sizeof(unsigned int) * conf->goods);

	conf->bin_count = bin_count;

	char * head = NULL;
	char * tail = NULL;
	char *line = NULL;
	unsigned long total_goods_count[conf->goods];

	unsigned int numbids_count[conf->goods];
	unsigned int goods[conf->goods];
	size_t len = 0;
	ssize_t read;
	int x;
	unsigned int bin_index[conf->goods];
	bin_index[0] = 0;
	total_goods_count[0] = numbids_count[0] = 0;
	for (x = 1; x < conf->goods; x++) {
		bin_index[x] = bin_count[x - 1] + bin_index[x - 1];
		total_goods_count[x] = numbids_count[x] = 0;
	}
	struct bid * tmp_bids = (struct bid *) malloc(
			sizeof(struct bid) * conf->bids);
	struct bid * root = &tmp_bids[0]; //malloc(sizeof(struct bid));
	struct bid * curr = root;
	curr->next = NULL;
	curr->prev = NULL;
	for (x = 1; x < conf->bids; x++) {
		curr->next = &tmp_bids[x]; //malloc(sizeof(struct bid));
		curr->next->prev = curr;
		curr = curr->next;
		curr->next = NULL;
	}

	curr = root;
	while ((read = getline(&line, &len, fp)) != -1) {
		if (!isdigit(line[0])) {
			continue;
		}
		head = tail = line;

		while (*head != '\t' && *head != '\0') {
			head++;
		}
		int id = strtol(tail, &head, 10);
		tail = head;
		head++;
		//get offer or value
		while (*head != '\t' && *head != '\0') {
			head++;
		}
		float offer = strtod(tail, &head);
		tail = head;
		head++;
		unsigned int goods_count = 0;
		unsigned int good = 0;
		unsigned int dummy = 0;
		unsigned int tmp_allocation[SIZE];
		for (x = 0; x < SIZE; x++) {
			tmp_allocation[x] = 0;
		}
		//reset the temporary goods array, used to determin the score
		goods_count = 0;
		for (x = 0; x < conf->goods; x++) {
			goods[x] = 0;
		}

		while (*head != '#' && *head != '\0') {
			if (*head == '\t') {
				good = strtol(tail, &head, 10);

				//sscanf(tail,"\t%u\t",tmp2);
				if (good < conf->goods) {
					tmp_allocation[(good / WORDSIZE)] += (1 << good);
					goods[goods_count] = good;
				} else {
					dummy = good;
				}
				tail = head;
				goods_count++;
			}
			head++;

		}
		if (dummy > 0) {
			goods_count--;
		}
		curr->average = (float) offer / (goods_count);
		for (x = 0; x < goods_count; x++) {
			total_goods_count[goods[x]] += goods_count;
			numbids_count[goods[x]]++;
		}
		curr->offer = offer;
		curr->bin = goods[0];
		curr->dummy = dummy;
		curr->id = id;
		for (x = 0; x < SIZE; x++) {
			curr->alloc.a[x] = tmp_allocation[x];
		}
		curr = curr->next;
//		printf("id %d bin %u count %u value %.3lf\n",bid_count,bin_for_bid,tmp_count[bin_for_bid],0);
		bin_index[goods[0]]++;
		//printf("hello\n");
	}
	free(line);
	float min = FLT_MAX;
	int min_pos = 0;
	int singleton_count = conf->bids - conf->singletons;
	for (x = 0; x < conf->goods; x++) {
		if (!have_singelton[x]) {
			int y;
			for (y = 1; y < SIZE; y++) {
				curr->alloc.a[x] = 0;
			}
			int word_index = x / WORDSIZE;
			int bit_index = x % WORDSIZE;
			curr->alloc.a[word_index] = (1 << bit_index);
			curr->offer = curr->average = 0.0f;
			curr->dummy = 0;
			curr->bin = x;
			curr->id = singleton_count;

			total_goods_count[x] += 1; //add one more to the score stat
			numbids_count[x] += 1; // also add one more to the number of bids to the score stat
			singleton_count++; // next singleton bid will have an consecutive bid id
			curr = curr->next;
		}
		double score = 0;
		double avg;
		if (numbids_count[x]) {
			printf("x %d total good count %lu, numbids_count %d\n", x,
					total_goods_count[x], numbids_count[x]);
			avg = ((double) total_goods_count[x]) / ((double) numbids_count[x]);
			score = ((double) numbids_count[x]) / avg;
		}
		if (score < min) {
			min = score;
			min_pos = x;
		}
	}
	unsigned int bid_to_bit[conf->goods];

	for (x = 0; x < conf->goods; x++) {
		bid_to_bit[x] = 0;
	}
	printf("min %.3f pos %d\n", min, min_pos);
	int bid_bit_count = -1;
	struct bid * new_root = NULL;
	struct bid * new_curr = NULL;
	int bid_count = 0;
	while (root) {
		bid_count = 0;
		bid_bit_count++;
		bid_to_bit[min_pos] = bid_bit_count;
		curr = root;
		while (curr) {

			int word_index = min_pos / WORDSIZE;
			int bit_index = min_pos % WORDSIZE;
			struct bid * next = curr->next;
			if (curr->alloc.a[word_index] & (1 << bit_index)) {
				curr->bin = bid_bit_count;
				if (!new_root) {
					root = remove_from_list(curr, root);
					new_root = curr;
					//curr = curr->next;
					new_curr = new_root;
					new_curr->prev = NULL;

				} else {
					root = remove_from_list(curr, root);
					new_curr->next = curr;
					new_curr->next->prev = new_curr;
					//curr = curr->next;

					new_curr = new_curr->next;
				}
				new_curr->next = NULL;
				bid_count++;
			}
			curr = next;

		}
		curr = root;
		conf->bin_count[bid_bit_count] = bid_count;
		conf->max_offset[bid_bit_count] = bid_count - 1;
		min_pos = get_next_best_good(conf, curr);
		printf("min pos %u\n", min_pos);
	}
	new_curr = new_root;
	while (new_curr) {
		struct allocation tmp;
		for (x = 0; x < SIZE; x++) {
			tmp.a[x] = 0;
		}
		for (x = 0; x < conf->goods; x++) {
			int bit_index = x % WORDSIZE;
			int word_index = x / WORDSIZE;
			if ((new_curr->alloc.a[word_index] & (1 << bit_index))) {
				tmp.a[word_index] |= (1 << bid_to_bit[x]);
//				printf("good %d translation %d\n",x,bid_to_bit[x]);
			}
		}
		for (x = 0; x < SIZE; x++) {
			new_curr->alloc.a[x] = tmp.a[x];
		}
		/* printf("id %u res %u\n",new_curr->id,new_curr->alloc.a[0]); */
		/* exit(0); */
		/* printf("%u bin %u\n",new_curr->id,bit_to_bid[new_curr->bin]); */
		new_curr = new_curr->next;
	}
	printf("total bids %u\n", conf->bids);

	for (x = 0; x < conf->goods; x++) {
		printf("%d %u\n", x, conf->bin_count[x]);
	}
	exit(0);

//	exit(0);
	int y;

	int bin_index2[conf->goods];
	bin_index2[0] = 0;

	for (x = 1; x < conf->goods; x++) {
		bin_index2[x] = bin_count[x - 1] + bin_index2[x - 1];
	}

	x = 0;

	struct bid * lhead, *ltail;
	ltail = new_root;
	while (ltail) {
		int good = ltail->bin;
		lhead = ltail->next;
		while (lhead && lhead->bin == good) {
			if (lhead->average > ltail->average) {
				if (lhead->prev == ltail) {

					if (ltail->prev)
						ltail->prev->next = lhead;
					else
						new_root = lhead;
					if (lhead->next)
						lhead->next->prev = ltail;
					lhead->prev = ltail->prev;
					ltail->next = lhead->next;
					lhead->next = ltail;
					ltail->prev = lhead;
					struct bid * tmp = lhead;
					lhead = ltail;
					ltail = tmp;
				} else {

					struct bid *ltailprev, *ltailnext;
					assert(ltail->next != lhead);
					assert(lhead->prev != ltail);
					assert(lhead->next != ltail);
					assert(ltail->prev != lhead);
					ltailnext = ltail->next;
					ltailprev = ltail->prev;
					if (ltail->prev)
						ltail->prev->next = lhead;
					else
						new_root = lhead;
					if (ltail->next)
						ltail->next->prev = lhead;
					if (lhead->prev)
						lhead->prev->next = ltail;
					if (lhead->next)
						lhead->next->prev = ltail;
					ltail->next = lhead->next;
					ltail->prev = lhead->prev;
					lhead->next = ltailnext;
					lhead->prev = ltailprev;
					struct bid * tmp = lhead;
					lhead = ltail;
					ltail = tmp;
				}
			}
			lhead = lhead->next;
		}
		ltail = ltail->next;
	}
	new_curr = new_root;

	while (new_curr) {

		int index = x;
		for (y = 0; y < SIZE; y++) {
			conf->allocation[index].a[y] = new_curr->alloc.a[y];
			assert(conf->allocation[index].a[y] == new_curr->alloc.a[y]);
		}
		conf->bin[index] = new_curr->bin;
		assert(
				conf->allocation[index].a[conf->bin[index]/WORDSIZE] & (1<< conf->bin[index]));
		assert(x >= bin_index2[conf->bin[index]]);
		assert(x < bin_count[conf->bin[index]]+bin_index2[conf->bin[index]]);

		conf->offer[index] = new_curr->offer;
		conf->dummies[index] = new_curr->dummy;
		conf->id[index] = new_curr->id;
		conf->average[index] = new_curr->average;
		new_curr = new_curr->next;
		x++;
	}
	free(tmp_bids);

	return;
}

//wordsize in bits
#define WORD (32)
#define BIN (0)
#define INDEX (1)

void print_debug(struct configuration * conf, unsigned int (*allocation_count),
		unsigned int allocation_id_index, unsigned int low_order_good,
		unsigned int (*bin_count)) {
	return;
	printf("low order good %u allocation_id_index %u\n", low_order_good,
			allocation_id_index);
	int x;
	printf("allocation:\n");
	for (x = 0; x < conf->goods; x++) {
		printf("%u/%u\t", allocation_count[x], bin_count[x]);
	}
	printf("\n\n");

}

float h(struct configuration * conf, struct allocation * curr_allocation,
		int good, unsigned int * bin_index, unsigned int t) {
	int y, x;
	float value = 0.0f;
	assert(curr_allocation->a[0] == t);
	const int words = SIZE;
	for (y = 0; y < conf->goods; y++) {
		float partial_val = 0.0f;
		if ((curr_allocation->a[y / WORDSIZE] & (1 << y % WORDSIZE))== 0){
			int count = 0;
			assert((curr_allocation->a[y/WORDSIZE] & (1<< y%WORDSIZE)) == 0);
			unsigned int status;
			int max_offset = conf->max_offset[y];
			int index;
			for (; status != 0 && count <= max_offset; count++) {

				//if(count[good] > conf->max_offset[good]){ break;}
				status = 0;
				index = count + bin_index[y];

				assert(y == conf->bin[index]);
				if (conf->dummies[index]) {
					for (x = 0; x < conf->allocation_id_index; x++) {
						status |= (conf->allocation_dummy[x]
								== conf->dummies[index]);
					}
				}

				if (status)
					continue;
				for (x = 0; x < words; x++) {
					status |= (conf->allocation[index].a[x])
							& (curr_allocation->a[x]);
				}
				//if(conf->id[index] == 75) {printf("hello %u\n",status);}
			}
			if (!status) {
				partial_val = conf->average[index];
			}

		}
		value += partial_val;

	}
	printf("value %.3f\n", value);
	return value;
}

#define DEBUG (0)
#define H (1)
#define THREADS (1024)
#define WARPS (THREADS/32)
#define GOOD_I (0)
#define COUNT_I (1)
#define GOODS (32)
template<int uses_dummy,int single_word>
__global__ void calc_best2(unsigned int * max,
			   unsigned int * _max_index,
			   unsigned short * bin_index,
			   struct allocation * allocation,
			   unsigned int * offer,
			   unsigned int * dummies,
			   unsigned int * bin) {

	__shared__ struct allocation curr_allocation[WARPS];
	__shared__ unsigned short max_index[GOODS];
	__shared__ unsigned short count[WARPS][GOODS];
	__shared__ unsigned int value[WARPS];
	__shared__ short shared_vars[2];
	__shared__ unsigned int shared_max;
	__shared__ unsigned char allocation_id_index[WARPS];

	__shared__ unsigned short allocation_dummy[(WARPS)*uses_dummy][(GOODS-1)*uses_dummy];

	__shared__ unsigned short allocation_id[WARPS][GOODS-1];

	const char laneid = threadIdx.x  % 32;
	const char warpid = threadIdx.x / 32;
	//each warp reset the allocation
	if (laneid == 0) {
		curr_allocation[warpid] = allocation[blockIdx.x];
		value[warpid] = offer[blockIdx.x];
	}
	if (threadIdx.x < goods) {
		max_index[threadIdx.x] = _max_index[threadIdx.x];
	}
	if (laneid == 0) {
		allocation_id_index[warpid] = 0;
	}
	int x;
	for(x=laneid;x < (GOODS);x += 32) {
		
		count[warpid][x] = 0;
	}
	

	//start add the second bid ---------------------------------------------------------------------------------
	//----------------------------------------------------------------------------------------------------------

	int good = 0;
	if (threadIdx.x == 0) {
		shared_max = *max;
		while ((curr_allocation[warpid].a[good / WORDSIZE]
			& (1 << good % WORDSIZE))&&
		       good < goods) {
			good++;
		}
		shared_vars[GOOD_I] = good;
		shared_vars[COUNT_I] = 32;
	}
	__syncthreads();
	good = shared_vars[GOOD_I];

	int binindex = bin_index[good];
	//if there is not enough bids in the start bin to continue
	//could later change such that it chooses the bin for which most bids exists
	if (warpid > max_index[good]) {
		return; //potentially fatal if i sync below
	}
	int index = binindex + warpid;
	int status = 1;

re_add_second_bid:
	;
	while (status) {
		status = 0;
		if (laneid < words) {
			status = curr_allocation[warpid].a[laneid]
				& allocation[index].a[laneid];
		}
		if (__any(status)) {
			if (laneid == 0) {
				index = atomicAdd((unsigned int *)&shared_vars[COUNT_I], 1);
			}
			index = __shfl(index, 0);
		}
		if (index > max_index[good]) {
			return;
		}
	}

	if (laneid < words) {
		curr_allocation[warpid].a[laneid] |= allocation[index].a[laneid];
	}

	if (laneid == 0) {
		value[warpid] += offer[index];
	}

	if (laneid == 0) {
		if(uses_dummy) {
			allocation_dummy[warpid][allocation_id_index[warpid]] = dummies[index];
		}
		//which bid we allocated
		allocation_id[warpid][allocation_id_index[warpid]] = index;
		allocation_id_index[warpid]++;

		if (value[warpid] > shared_max) {
			atomicMax(max, value[warpid]);
			atomicMax(&shared_max, value[warpid]);
		}
	}

	//end add the second bid -----------------------------------------------------------------------------------
	//----------------------------------------------------------------------------------------------------------



	int allocate = 1; //could remove and just use goto
	int dealloc = 0;
	while (allocate || dealloc) {

		while (allocate) {
			// whilst the good have already been allocated
			status = 0;
			x = laneid;
			while (status == 0) {
				if (x < GOODS) { // if the good index is larger than the amount of goods
					if(single_word) {//template
						status = curr_allocation[warpid].a[0] & (1 << (x % WORDSIZE));
					} else {
						status = curr_allocation[warpid].a[x/WORDSIZE] & (1 << (x % WORDSIZE));
					}
				}
				status = __ballot((status == 0));

				if (status) {
					if (laneid == 0) {
						good = __ffs(status) - 1 + x; //which thread is the first to find empty good
					}
					good = __shfl(good, 0);
				} else {
					
					x += 32;
					if (x >= goods) {
						good = goods;
						break;
					}

				}
				
			}

			//if there are no more goods to allocate, exit allocation loop
			if (good >= goods) {
				//printf("dealloc full good\n");
				allocate = 0;
				dealloc = 1;
				break;
			}
			int max_offset = max_index[good];
			binindex = bin_index[good];

			index = 0;

			status = 0;
			while (status == 0) {

				index = count[warpid][good] + laneid;

				if(index >= max_offset) {
					status = 1;					
				}

				if (index < max_offset) {
					int y;
					if(uses_dummy) {
						unsigned int t_dummy = dummies[binindex + index];
						if (t_dummy) {
					
							for (y = 0; y < allocation_id_index[warpid]; x++) {
								status |= (allocation_dummy[warpid][y] == t_dummy);
							}
						}
					}
//					if(!__ballot(status == 0)) {
//						x += 32;
//						continue;
//					}
					for (y = 0; y < SIZE; y++) {
						status |= allocation[index + binindex].a[y]
							& (curr_allocation[warpid].a[y]);
					}
					status = __ballot((status == 0));
					
					if (status == 0) { //if no thread have found a compatible bid
						if (laneid == 0) {
							count[warpid][good] += 32;
						}						
					} else if (laneid == 0) {
						index = __ffs(status) - 1 + count[warpid][good];
						count[warpid][good] = index + 1;
					}

				}
				//if the count index is greater than the maximum offset
				if (count[warpid][good] > max_offset) {
					dealloc = 1;
					break;
				}

			}
			status = __shfl(status,0);
			
			//if count is greater than maximum offset and we did not find a suitable bid
			if (dealloc && !status) {
				if (laneid == 0) {
					count[warpid][good] = 0;
				}
				allocate = 0;
				break;
			}
			index = __shfl(index,0);
			
			//parallel --------------------------------------------------------------------------------------------------
			// add the goods
			if (laneid < SIZE) {
				curr_allocation[warpid].a[laneid] |=
					allocation[index].a[laneid];
			}

			//dummy bid for the bid we allocation
			if (laneid == 0) {
				value[warpid] += offer[index];
				if(uses_dummy) {
					allocation_dummy[warpid][allocation_id_index[warpid]] =
						dummies[index];
				}
				//which bid we allocated
				allocation_id[warpid][allocation_id_index[warpid]] = index;
				allocation_id_index[warpid]++;

				if (value[warpid] > shared_max) {										
					atomicMax(max, value[warpid]);
					atomicMax(&shared_max, value[warpid]);
					printf("new max %u\n", value[warpid]);
//				printf("bid id separated by tab\n");
//				for (x = 0; x < conf->allocation_id_index; x++) {
//					printf("%u\t", conf->id[conf->allocation_id[x]]);
//				}
//				printf("\n");
				}
			}
			//parallel --------------------------------------------------------------------------------------------------
		}

		while (dealloc) {
			dealloc = 0;
			allocate = 1;
			if (laneid == 0) {
				allocation_id_index[warpid] -= 1;
			}

			//printf("index %u\n",conf->allocation_id_index);
			int dealloc_index = allocation_id[warpid][allocation_id_index[warpid]];
			int dealloc_good = bin[dealloc_index];
			if (laneid == 0) {
				value[warpid] -= offer[dealloc_index];
			}

			if (laneid < SIZE) {
				curr_allocation[warpid].a[laneid] ^= allocation[dealloc_index].a[laneid];
			}



			if (count[warpid][dealloc_good] >= max_index[dealloc_good]) {
				//	printf("re-de-alloc good %u\n",dealloc_good);
				if (laneid == 0) {
					count[warpid][dealloc_good] = 0;
				}
				dealloc = 1;
				allocate = 0;
			}
			good = 0;
			if(allocation_id_index[warpid] == 0) {//if we deallocated the second bid
				dealloc = 0;
				allocate = 1;
				good = shared_vars[GOOD_I];
				if (laneid == 0) {
					index = atomicAdd((unsigned int *)&shared_vars[COUNT_I], 1);
				}
				index = __shfl(index, 0);				

				if(index > max_index[good]){
					return;
				}
				goto re_add_second_bid;								
			}			
		}

	}
}

/**
 * This macro checks return value of the CUDA runtime call and exits
 * the application if the call failed.
 */
#define CUDA_CHECK_RETURN(value) {											\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		fprintf(stderr, "Error %s at line %d in file %s\n",					\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		exit(1);															\
	} }

/**
 * Host function that prepares data array and passes it to the CUDA kernel.
 */
int setup_mem_and_run(struct configuration * conf) {
	void *d = NULL;
	int i;

	struct configuration transfer;
	unsigned short bin_index[conf->goods];
	bin_index[0] = 0;
	for(i = 1; i < conf->goods;i++) {
		bin_index[i] = bin_index[i-1] + conf->bin_count[i-1];
	}
	
	int bids = conf->bids;
	int goods = conf->goods;
	unsigned int * max;
	unsigned int _max = 0;

	unsigned short * d_bin_index;
	
	CUDA_CHECK_RETURN(hipMalloc((void**) &d_bin_index, sizeof(unsigned short)));
	CUDA_CHECK_RETURN(hipMalloc((void**) &max, sizeof(unsigned int)));
	CUDA_CHECK_RETURN(
			hipMalloc((void**) &transfer.allocation, sizeof(struct allocation) * bids));
	CUDA_CHECK_RETURN(hipMalloc((void**) &transfer.bin, sizeof(int) * bids));
	CUDA_CHECK_RETURN(hipMalloc((void**) &transfer.id, sizeof(int) * bids));
	CUDA_CHECK_RETURN(
			hipMalloc((void**) &transfer.dummies, sizeof(int) * bids));
	CUDA_CHECK_RETURN(
			hipMalloc((void**) &transfer.offer, sizeof(float) * bids));
	CUDA_CHECK_RETURN(
			hipMalloc((void**) &transfer.average, sizeof(float) * bids));
	CUDA_CHECK_RETURN(
			hipMalloc((void**) &transfer.bin_count, sizeof(int) * goods));
	CUDA_CHECK_RETURN(
			hipMalloc((void**) &transfer.max_offset, sizeof(int) * goods));
	//CUDA_CHECK_RETURN(hipMalloc((void**) &d, sizeof(int) * WORK_SIZE));
	CUDA_CHECK_RETURN(
		hipMemcpy(d_bin_index, bin_index, sizeof(unsigned short) * goods, hipMemcpyHostToDevice));

	CUDA_CHECK_RETURN(
			hipMemcpy(transfer.allocation, conf->allocation, sizeof(struct allocation) * bids, hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(
			hipMemcpy(transfer.bin, conf->bin, sizeof(int) * bids, hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(
			hipMemcpy(transfer.id, conf->id, sizeof(int) * bids, hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(
			hipMemcpy(transfer.dummies, conf->dummies, sizeof(int) * bids, hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(
			hipMemcpy(transfer.offer, conf->offer, sizeof(unsigned int) * bids, hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(
			hipMemcpy(transfer.average, conf->average, sizeof(float) * bids, hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(
			hipMemcpy(transfer.bin_count, conf->bin_count, sizeof(int) * goods, hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(
			hipMemcpy(transfer.max_offset, conf->max_offset, sizeof(int) * goods, hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(
			hipMemcpy(max, &_max, sizeof(unsigned int)*1, hipMemcpyHostToDevice));
	
	//CUDA_CHECK_RETURN(hipMemcpy(d, idata, sizeof(int) * WORK_SIZE, hipMemcpyHostToDevice));
	int blocks = conf->bin_count[0];
	calc_best2<0,1><<<blocks,THREADS,0>>>(max,transfer.max_offset,d_bin_index,transfer.allocation,transfer.offer,transfer.dummies,transfer.bin);
	
//	bitreverse<<<1, WORK_SIZE, WORK_SIZE * sizeof(int)>>>(d);

	CUDA_CHECK_RETURN(hipDeviceSynchronize());
	// Wait for the GPU launched work to complete
	CUDA_CHECK_RETURN(hipGetLastError());
//	CUDA_CHECK_RETURN(
	//		hipMemcpy(odata, d, sizeof(int) * WORK_SIZE, hipMemcpyDeviceToHost));

//	for (i = 0; i < WORK_SIZE; i++)
//		printf("Input value: %u, device output: %u\n", idata[i], odata[i]);

	CUDA_CHECK_RETURN(
		hipMemcpy(&_max , max, sizeof(unsigned int)*1, hipMemcpyDeviceToHost));
	
	CUDA_CHECK_RETURN(hipDeviceReset());

	return 0;
}

int main(int argc, char *argv[]) {
	int x;
	FILE * fp;
	fp = fopen(argv[1], "r");
	if (fp == NULL) {
		printf("Could not open file\n");
		exit(EXIT_FAILURE);
	}
	printf("hello\n");
	struct configuration * conf = get_configuration(fp);
	conf->singletons = 0;
	unsigned int * have_singleton = (unsigned int *) malloc(
			sizeof(int) * conf->goods);
	for (x = 0; x < conf->goods; x++) {
		have_singleton[x] = 0;
	}
	unsigned int * bin_count = get_bincount(fp, conf, have_singleton);

	for (x = 0; x < conf->goods; x++) {
		if (!have_singleton[x]) {
			conf->singletons++;
			conf->bids++;
			bin_count[x]++;
		}
		printf("bin %d count %u\n", x, bin_count[x]);
	}

	fclose(fp);
	fp = fopen(argv[1], "r");
	if (fp == NULL) {
		printf("Could not open file\n");
		exit(EXIT_FAILURE);
	}
	allocate_all_bids(fp, conf, have_singleton, bin_count);
	fclose(fp);
	for (x = 0; x < conf->bids; x++) {
		printf("x %d id %u, offer %.3f, bin %u, alloc %u, bin_count %u\n", x,
				conf->id[x], conf->offer[x], conf->bin[x],
				conf->allocation[x].a[0], conf->bin_count[conf->bin[x]]);

	}
	printf("words %u wordsize %lu\n", SIZE, WORDSIZE);
	free(have_singleton);
	setup_mem_and_run(conf);
	//calc_best2(conf);
	free(conf->allocation);
	free(conf->bin);
	free(conf->id);
	free(conf->dummies);
	free(conf->bin_count);
	free(conf->max_offset);
	free(conf->offer);
	free(conf->average);
	free(conf->allocation_id);
	free(conf->allocation_dummy);
	free(conf);
	printf("Bye\n");
	exit(EXIT_SUCCESS);
}
