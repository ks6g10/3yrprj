
#include <stdio.h>
#include <string.h> // for ffs
#include <stdint.h>
#include <math.h>
#include <time.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

/*Debug enabled gives more print statements of bids and how the "Matrix" gets evaluated*/
#define DEBUG 0
/*Test sets all bids to one, which should give you n=|ITEMS| bids on output*/
#define TEST 1
/*Defines from 0-Range the random will give out*/
#define RANGE 20
#define ITEMS 23
#define MAX (2 << (ITEMS-1))
#if ITEMS < 8
#define dint uint8_t
#elif ITEMS < 16
#define dint uint16_t
#elif ITEMS < 32
#define dint uint32_t
#elif ITEMS > 32
#define dint uint64_t
#endif

static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

/*                    1         2        4       8        16*/

char * assets[3] = {"apple","mapple","potato"};
/*		           0 1 2 3 4 5 6 7 8			*/
dint bids[MAX];
// dint bids[MAX] ={0,2,3,6,4,7,6,9}; //conf 5 and 2
//dint bids[MAX] =  {0,2,3,6,4,6,6,9}; //conf 3 and 4
// dint bids[MAX] =  {0,20,3,6,4,6,10,9}; //conf 1 and 6
dint wopt[MAX] = {0};
dint f[MAX] = {0};
dint O[MAX] = {0};

inline  dint intersect( dint seta,  dint setb) {
     return (seta & setb);
}

inline  dint _union( dint seta,  dint setb) {
     return (seta | setb);
}

#define setdiff(seta,setb) (seta & ~setb)

inline  dint cardinality( dint seta) {
     return __builtin_popcount(seta);
}

void gen_rand_bids(dint MAXVAL) {
     register dint i = 0;
#if TEST
     for(i = 1; i < MAXVAL;i++) {
	  bids[i] = 1;
     }
#else
     for(i = 1; i < MAX;i++) {
	  bids[i] = rand() % RANGE;
     }
#endif

#if DEBUG
     printf("i =\t");
     for(i = 1; i < MAX;i++) {
	  printf("%u\t",i);
     }
     printf("\n");
     printf("val =\t");
     for(i = 1; i < MAX;i++) {
	  printf("%u\t",bids[i]);
     }
     printf("\n");
#endif
}

/*Reminder of sets
 *f[]
 *O[]
 *bids[]
 */
inline void printfo() {
#if DEBUG
     dint i;
     printf("i\t");
     for(i =1; i < MAX; i++) {
	  printf("%u\t",i);
     }
     printf("\n");
     printf("f[]\t");
     for(i =1; i < MAX; i++) {
	  printf("%u\t",f[i]);
     }
     printf("\n");
     printf("O[]\t");
     for(i =1; i < MAX; i++) {
	  printf("%u\t",O[i]);
     }
     printf("\n");
#endif
}

/*Sets all bids with one element in it, |n| = 1*/
inline void set_singleton_bid(dint MAXVAL) {
     register  dint i;
     for(i =1; i< MAXVAL; i*=2) {
	  f[i] = bids[i];
	  if(bids[i] > 0)
	       O[i] = i;
     }
}


/* conf a e.g. 1101
 * (~a+i) & a gives a subset of a
 *  i is a integer from 1 to |a|
 *
 * ~1101 = 0010
 * i = 0001
 * (0010+0001)&1101 =
 * 0011&1101 = 0001
 *
 *i = 0011
 * (0010+0011)&1101
 *(0101)&1101 = 0101
 */



struct _stack {
	dint conf;
	struct _stack * next;
} typedef stack;


void parse_wopt(dint MAXVAL) {
     //wopt at start contain MAX at wopt[0] which is the combination that goes in bids[wopt[n]]
     stack * root =(stack *) malloc(sizeof(stack));
     //DO NOT REMOVE -1
     root->conf = (MAXVAL)-1;
     stack * curr = root;
     while(curr != NULL) {

	     dint conf = curr->conf;
	     if(conf == 0) {
	    	 printf("EXIT FAILURE\n");
	    	 return;
	     }
	     if(conf != O[conf]) {
		     dint diff = setdiff(conf,O[conf]);
		     curr->conf = O[conf];
		     stack * tmp = (stack *) malloc(sizeof(stack));
		     tmp->conf = diff;
		     tmp->next = curr;
		     root = tmp;
		     curr = root;
		     continue;
	     }
	     curr = curr->next;
     }
     curr = root;
     dint tmp = 0;
     while(curr != NULL) {
	  printf("conf %u value %u\n",curr->conf,bids[curr->conf]);
	  tmp++;
	  stack * tmp = curr;
	  curr = curr->next;
	  free(tmp);
     }
     printf("n = %u",tmp);
}

/*n 15 t 9 n 16 t 42*/
void max2(dint conf) {
     register dint card = cardinality(conf)/2;
     register dint combinations = 1 << (cardinality(conf)-1);
     register dint max = bids[conf];
     register dint set = conf;
     register dint tmp = 0;
     register dint subset;
     register dint inverse = ~conf;
     register dint i;
     for(i = 1;i<combinations; i++) {
	  subset = (inverse+i)&conf;
	  if(cardinality(subset) > card)
	       continue;
	  tmp = f[setdiff(conf,subset)] + f[subset];
	  if(max < tmp) {
	       max = tmp;
	       set = subset;
	  }
     }
     f[conf] = max;
     O[conf] = set;
}

__global__ void add(unsigned int * p, unsigned int * f, unsigned int * O)
{
	int tid = blockIdx.x;

	unsigned int conf = p[tid];
	unsigned int card = (unsigned int) __popc(conf)/2;
	unsigned int combinations = 1 << (__popc(conf) -1);
	unsigned int max = f[conf];
	unsigned int set = p[tid];

	unsigned int tmp = 0;
	unsigned int subset;
	unsigned int inverse = ~set;
	unsigned int i;
	if(max == 0) {
		printf("hello");
		return;
	}

	for(i = 1;i<combinations; i++) {
		  subset = (inverse+i)&conf;
		  if(__popc(subset) > card)
		       continue;
		  tmp = f[setdiff(conf,subset)] + f[subset];
		  if(max < tmp) {
		       max = tmp;
		       set = subset;
		  }
	     }
	     f[conf] = max;
	     O[conf] = set;
}















__global__ void gen_subset(void) {



}

/*
 *
 * 1. gen all combinations of card n
 * 2. for each combination, generate all subset with condition |s| < |c|/2
 * 3. for each subset check if |s| < |c|/2 then compute the sum
 * 4. 
 *
 *
 *
 */

void run_test(dint MAXVAL,dint items) {
/*Setup the environment*/
     dint perm[MAXVAL];
     gen_rand_bids(MAXVAL);
     set_singleton_bid(MAXVAL);
     printfo();
     dint i, c;

 	dint *dev_f,*dev_o,*dev_p;
 	HANDLE_ERROR(hipMalloc((void **)&dev_f, MAXVAL*sizeof(int)));
 	HANDLE_ERROR(hipMalloc((void **)&dev_o, MAXVAL*sizeof(int)));
 	HANDLE_ERROR(hipMalloc((void **)&dev_p, MAXVAL*sizeof(int)));
 	//add<<<1,1>>>(2,7,dev_c);

 	HANDLE_ERROR(hipMemcpy(dev_f,bids,MAXVAL*sizeof(int),hipMemcpyHostToDevice));
 	HANDLE_ERROR(hipMemcpy(dev_o,O,MAXVAL*sizeof(int),hipMemcpyHostToDevice));
 //	printf("2+7 = %d \n",c);
     /*2.*/
 	 int count = 0;
/*     items represent the maximum cardinality a set can have*/
     for(i = 2; i <= items; i++) {
    	 count = 0;
/*1.*/	     for(c = (1 << i) -1; c <= MAXVAL;) {
	    	 perm[count++] = c;
		     //bit hacks "Compute the lexicographically next bit permutation"
		     dint t = c | (c-1);
		     c = (t + 1) | (((~t & -~t) - 1) >> (__builtin_ctz(c) + 1));
		     //end ref
	     }
	     HANDLE_ERROR(hipMemcpy(dev_p,perm,count*sizeof(int),hipMemcpyHostToDevice));
	     add<<<count,1>>>(dev_p,dev_f,dev_o);
	     printfo();
     }
     HANDLE_ERROR(hipMemcpy(f,dev_f,MAXVAL*sizeof(int),hipMemcpyDeviceToHost));
     HANDLE_ERROR(hipMemcpy(O,dev_o,MAXVAL*sizeof(int),hipMemcpyDeviceToHost));
     //int i;
     hipFree(dev_f);
     hipFree(dev_o);
     hipFree(dev_p);

     parse_wopt(MAXVAL);

}



int main(void) {
     /*Start n amount of assets*/
     dint from = 14;
     /*End amount of assets, inclusive*/
     dint till = 14;
     dint MAXVAL = (2 << (from-1));
     if(till > ITEMS) {
	  printf("More than maximum allowed\n");
	  return 1;
     }

     //time_t start,end,t;

     /*Run all tests*/
     for(;from <= till;from++) {
	  MAXVAL = (2 << (from-1));
	  printf("maxval %u\n",MAXVAL);
	//  start=clock();//predefined  function in c
	  run_test(MAXVAL,from);
	//  end=clock();
	//  t=(end-start)/CLOCKS_PER_SEC;
	//  printf("\nTime taken =%lu for n= %u\n", (unsigned long) t,from);
/*Reset the arrays*/
	  memset(&f,'\0',sizeof(f));
	  memset(&O,'\0',sizeof(O));
     }

     return 0;
}

void old_test(void) {
     /*Testing facility*/
     dint i;
     for(i = 1; i < 8; i++) {
	  dint z = intersect(i,i-1);
	  dint x = _union(i,i-1);
	  dint f = setdiff(i,i-1);
	  dint t = cardinality(i);
	  printf("i %u \tinter %u\t union %u\t diff %u\t card %u\n",i,z,x,f,t);
     }
     return;
}


//#include <cuda.h> // helper utility functions
//#include <cuda_runtime.h>
//#include <cuda.h>
//#include <book.h>





__global__ void kernel(void){

}

int m2ain(void)
{

	return 0;
}
