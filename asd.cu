#include <stdio.h> 
#include <string.h> // for ffs
#include <stdint.h>
#include <math.h>
#include <time.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

/*Debug enabled gives more print statements of bids and how the "Matrix" gets evaluated*/
#define DEBUG 0
#define TRUE 1
#define FALSE 0
/*Test sets all bids to one, which should give you n=|ITEMS| bids on output*/
#define TEST 1
/*Defines from 0-Range the random will give out*/
#define RANGE 10000
#define ITEMS 25

#define MAX (2 << (ITEMS-1))
#if ITEMS < 8
#define dint uint8_t  
#elif ITEMS < 16
#undef dint
#define dint uint16_t
#elif ITEMS < 32
#undef dint
#define dint uint32_t
#elif ITEMS > 32
#undef dint
#define dint uint64_t
#endif

#define SUBSET(X)((~_conf+(X+1))&_conf)
#define SETSUM(X)(f[setdiff(_conf,X)]+f[X])
#define I (threadIdx.x + blockDim.x * blockIdx.x)

static void HandleError( hipError_t err, const char *file,int line ) {
	if (err != hipSuccess) {
		printf( "%s in %s at line %d\n", hipGetErrorString( err ),file, line );
		exit( EXIT_FAILURE );
	}
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))



/*		           0 1 2 3 4 5 6 7 8			*/
dint * bids;
// dint bids[MAX] ={0,2,3,6,4,7,6,9}; //conf 5 and 2
//dint bids[MAX] =  {0,2,3,6,4,6,6,9}; //conf 3 and 4
// dint bids[MAX] =  {0,20,3,6,4,6,10,9}; //conf 1 and 6
dint * f;
dint * O;
 
struct _stack {
	dint conf;
	struct _stack * next;
} typedef stack;

struct _locklist {
	unsigned int size;
	unsigned int conf;
	unsigned int * dev_f;
	unsigned int * dev_o;
	struct _locklist * next;
} typedef locklist;

struct _lockstruct {
	unsigned int * dev_lock;
	struct _lockstruct * next;
} typedef lockstruct;


#define setdiff(seta,setb) (seta & ~setb)

inline  dint cardinality( dint seta) {
	return __builtin_popcount(seta);
}

void gen_rand_bids(dint MAXVAL) {
	register dint i = 0;
#if TEST
	for(i = 1; i < MAXVAL;i++) {
		bids[i] = 1;
		O[i] = i;
	}
//	bids[1] =0;
//	bids[2] = 0;
//	bids[32769] = 20;
#else
	for(i = 1; i < MAXVAL;i++) {
		bids[i] = rand() % RANGE;
		O[i] = i;
	}
	for(i = 1; i < MAXVAL;i*=2) {
		bids[i] = rand() % RANGE;

	}
#endif

#if DEBUG
	printf("i =\t");
	for(i = 1; i < MAX;i++) {
		printf("%u\t",i);
	}
	printf("\n");
	printf("val =\t");
	for(i = 1; i < MAX;i++) {
		printf("%u\t",bids[i]);
	}
	printf("\n");
#endif
}

const char *btb(dint y)
{
	int x = y;
	static char b[9];
	b[0] = '\0';
	
	int z;
	for (z = 128; z > 0; z >>= 1)
	{
		strcat(b, ((x & z) == z) ? "1" : "0");
	}	
	return b;
}

inline void printfo() {
#if DEBUG
	dint i;
	printf("i\t");
	for(i =1; i < MAX; i++) {
		printf("%u\t",i);
	}
	printf("\n");
	printf("f[]\t");
	for(i =1; i < MAX; i++) {
		printf("%u\t",f[i]);
	}
	printf("\n");
	printf("O[]\t");
	for(i =1; i < MAX; i++) {
		printf("%u\t",O[i]);
	}
	printf("\n");
#endif
}

/*Sets all bids with one element in it, |n| = 1*/
inline void set_singleton_bid(dint MAXVAL) {
	register  dint i;
	for(i =1; i< MAXVAL; i*=2) {
		f[i] = bids[i];
		if(bids[i] > 0)
			O[i] = i;
	}
}

/* conf a e.g. 1101
 * (~a+i) & a gives a subset of a
 *  i is a integer from 1 to |a|
 *
 * ~1101 = 0010
 * i = 0001
 * (0010+0001)&1101 =
 * 0011&1101 = 0001
 *
 *i = 0011
 * (0010+0011)&1101
 *(0101)&1101 = 0101
 */



void printfo(dint MAXVAL) {
	int i;
	printf("\n");
	for(i = 1; i< MAXVAL; i++)
	{
		printf("Bid[%d]\t%u\tF[%d]\t%u\tO[%d]\t%u\tbin\t%s\n",i,bids[i],i,f[i],i,O[i],btb(i));

	}
}

void parse_wopt(dint MAXVAL) {
	printf("parse maxval = %u\n",MAXVAL);
	//wopt at start contain MAX at wopt[0] which is the combination that goes in bids[wopt[n]]
	stack * root =(stack *) malloc(sizeof(stack));
	stack * sroot = NULL;
	stack * scurr = NULL;
	//DO N	OT REMOVE -1
	root->conf = (MAXVAL)-1;
	int count = 0;
	root->next = NULL;
	stack * curr = root;
	while(curr) {
		dint conf = curr->conf;
		if(conf == 0) {
			printf("EXIT FAILURE\n");
			return;
		}
		/*if something is wrong*/
		if(count > 40) {
			fprintf(stderr,"Something went wrong at line %d in %s\n",__LINE__,__FILE__);
			return;
		}
			printf("curr %u\t",curr->conf);
		if(conf != O[conf]) {

			dint diff = setdiff(conf,O[conf]);
			curr->conf = O[conf];
			stack * tmp = (stack *) malloc(sizeof(stack));
			printf("diff %u\t O[conf] %u f %u\n",diff,O[conf],f[conf]);
			tmp->conf = diff;
			tmp->next = curr;
			root = tmp;
			curr = root;
			count++;
			continue;
		}
		if(sroot == NULL) {
			sroot = curr;
			scurr = curr;
		} else {
			/*set next pointer to the next singelton*/
			scurr->next = curr;
		}

		/*set the current singleton to be the pointer*/
		scurr = curr;
		curr = curr->next;
		/*clear the pinter to avoid infinite loop if cross-referenced*/
		scurr->next = NULL;

	}
	curr = sroot;
	dint tmp = 0;
	while(curr != NULL) {
		printf("conf %u value %u\n",curr->conf,bids[curr->conf]);
		tmp++;
		stack * tmp = curr;
		curr = curr->next;
		free(tmp);
	}
	printf("n = %u\n",tmp);
}


/*
 *
 * 1. gen all combinations of card n
 * 2. for each combination, generate all subset with condition |s| < |c|/2
 * 3. for each subset check if |s| < |c|/2 then compute the sum
 * 4. 
 *
 *
 *
 */ 

#define MAXBLOCKSIZE 256U
#define NAGENTS 23
#define NSTREAMS 16 
#if NAGENTS > 2
#define NPERBLOCK 8
#else
#define NPERBLOCK 4
#endif
__global__ void subsetcomp22(
		 	     unsigned int * __restrict__ f, /*Bid value*/
			     unsigned int * __restrict__ O, /*The move array*/
			     unsigned int * __restrict__ lock,
			     unsigned int _conf, /*The configuration*/
			     unsigned int cardmax, /*cardinality of max allowance*/
			     unsigned int maxval,
			     unsigned int count,
			     unsigned int offset,
			     unsigned int defbid)
{

	__shared__ unsigned int share[MAXBLOCKSIZE];
	__shared__ unsigned int step[MAXBLOCKSIZE];
		/* printf("threadid.x\t%d\tblockDim.x\t%d\tblockIdx.x\t%d\tI\t%u\n", */
	/*        threadIdx.x, */
	/*        blockDim.x, */
	/*        blockIdx.x, */
	/*        conf); */
//	unsigned int i = I+offset;
//	unsigned int ispec = (threadIdx.x*NPERBLOCK + blockDim.x * blockIdx.x) + offset;
	unsigned int ispec = (threadIdx.x + blockDim.x * blockIdx.x) + offset;
//	unsigned int i = I+offset;
	/*subset var, but also for indexing later  on*/
//	unsigned int s;// = SUBSET(i);


//	unsigned int tid = threadIdx.x;

	unsigned int val11,val12,step1,val21,val22,step2,val31,val32,step3,val41,val42,step4;
	/* unsigned int vals[NPERBLOCK][2]; */
	/* unsigned int step[NPERBLOCK]; */
	
	step[threadIdx.x] = share[threadIdx.x] = 0U;
	if(ispec <= maxval) {

		/* unsigned int tmp = ispec; */
		/* tmp += ((maxval - ispec > NPERBLOCK) ? NPERBLOCK : maxval - ispec); */

		/* unsigned int i; */
		/* for(;ispec<tmp; ispec++) { */
		/* 	step[i] = SUBSET(ispec); */
		/* 	if(__popc(step1) <= cardmax) { */
		/* 		vals[isp] */

		/* 	} */
		/* } */

 		step1 = SUBSET(ispec);
		val11 = val12 = 0U;
		val41 = val42 = 0U;
		val21 = val22 = 0U;
		val31 = val32 = 0U;
		if(__popc(step1) <= cardmax) {
			val11 = f[setdiff(_conf,step1)];
			val12 = f[step1];	
		}
		ispec += blockDim.x;
		step2 = SUBSET(ispec);
		if(__popc(step2) <= cardmax && ispec <= maxval) {
			val21 = f[setdiff(_conf,step2)];
			val22 = f[step2];	
		}
		ispec += blockDim.x;
		step3 = SUBSET(ispec);

		if(__popc(step3) <= cardmax && ispec <= maxval) {
			val31 = f[setdiff(_conf,step3)];
			val32 = f[step3];	
		}
		ispec += blockDim.x;
		step4 = SUBSET(ispec);

		if(__popc(step4) <= cardmax && ispec <= maxval) {
			val41 = f[setdiff(_conf,step4)];
			val42 = f[step4];	
		}
		ispec += blockDim.x;
		val11 += val12;
		val21 += val22;
		val31 += val32;
		val41 += val42;

		if(val21 > val11) {
			val11 = val21; 
			step1 = step2;
		}
		if(val41 > val31) {
			val31 = val41;
			step3 = step4;
		}

		if(val31 > val11) {
			val11 = val31;
			step1 = step3;
		}
		share[threadIdx.x] = val11;
		step[threadIdx.x] = step1;
		
#if NPERBLOCK >= 8		
		step1 = SUBSET(ispec);
		val11 = val12 = 0U;
		val41 = val42 = 0U;
		val21 = val22 = 0U;
		val31 = val32 = 0U;
		if(__popc(step1) <= cardmax && ispec < maxval) {
			val11 = f[setdiff(_conf,step1)];
			val12 = f[step1];	
		}
		ispec += blockDim.x;
		step2 = SUBSET(ispec);

		if(__popc(step2) <= cardmax && ispec < maxval) {
			val21 = f[setdiff(_conf,step2)];
			val22 = f[step2];	
		}
		ispec += blockDim.x;
		step3 = SUBSET(ispec);

		if(__popc(step3) <= cardmax && ispec < maxval) {
			val31 = f[setdiff(_conf,step3)];
			val32 = f[step3];	
		}
		ispec += blockDim.x;
		step4 = SUBSET(ispec);

		if(__popc(step4) <= cardmax && ispec < maxval) {
			val41 = f[setdiff(_conf,step4)];
			val42 = f[step4];	
		}
//		ispec += blockDim.x;
		
		val11 += val12;
		val21 += val22;
		val31 += val32;
		val41 += val42;

		if(val21 > val11) {
			val11 = val21; 
			step1 = step2;
		}
		if(val41 > val31) {
			val31 = val41;
			step3 = step4;
		}

		if(val31 > val11) {
			val11 = val31;
			step1 = step3;
		}

		if(val11 > share[threadIdx.x]) {
			share[threadIdx.x] = val11;
			step[threadIdx.x] = step1;
		}
#endif
		/* if(__popc(s) <= cardmax ) { */
		/* 	share[tid] = f[setdiff(_conf,s)] + f[s]; */
		/* 	step[tid] = s; */
		/* } */
	}
	ispec = I;
       
	val11= blockDim.x >> 1U;
	__syncthreads();
#pragma unroll
	for (; val11>0U; val11>>=1U) {
		if (threadIdx.x < val11 && (ispec < maxval)) {
			if(share[threadIdx.x] < share[threadIdx.x + val11]) {
				step[threadIdx.x] = step[threadIdx.x+val11];
				share[threadIdx.x] = share[threadIdx.x+val11];
			}
		}
		__syncthreads();
	}

	if(threadIdx.x == 0U) {
		//unsigned int tshare = share[0U];
		val11 = share[0U];
		if(defbid>val11)
			return;
		if(lock[count] < val11) {
			if(atomicMax(&(lock[count]),val11) < val11) {
				//lock[count] = val11;
				O[_conf] = step[0U];
				f[_conf] = val11;
				
			} 
		}
	}
}



#define COMBS(X) ((1 << cardinality(X)) - 1)

int run_test(dint MAXVAL,dint items) {
/*Setup the environment*/
	//dint perm[MAXVAL];
	printfo();
	register unsigned int i, c,t,count =0;
//	f = bids;
 	unsigned int *dev_f,*dev_o;

	i = items/2;
	count = 0;
	/* for(c = (1 << i) -1; c <= MAXVAL;) { */
	/* 	count++; */
	/* 	t = c | (c-1); */
	/* 	c = (t + 1) | (((~t & -~t) - 1) >> (__builtin_ctz(c) + 1)); */
	/* } */
	HANDLE_ERROR(hipDeviceReset());
	unsigned int * dev_lock1,*dev_lock2,*dev_ptr;
	const	unsigned int devcount = 1024;// count;
	unsigned int streams = NSTREAMS;
	unsigned int count2 = 0;
	unsigned int streamcount = 0;
	hipStream_t stream[streams];
	for(int i = 0;i < streams; i++)
		HANDLE_ERROR(hipStreamCreate(&stream[i]));
	//HANDLE_ERROR(cudaStreamCreate(&stream[1]));
	printf("count %u\n",devcount);
//	HANDLE_ERROR(cudaDeviceSetCacheConfig(cudaFuncCachePreferL1));
//	cudaProfilerStart();
	count = 0;
	HANDLE_ERROR(hipMalloc((void **)&dev_lock1,(10+devcount)*sizeof(int)));
	HANDLE_ERROR(hipMalloc((void **)&dev_lock2,(10+devcount)*sizeof(int)));
 	HANDLE_ERROR(hipMalloc((void **)&dev_f, MAXVAL*sizeof(int)));
 	HANDLE_ERROR(hipMalloc((void **)&dev_o, MAXVAL*sizeof(int)));

 	HANDLE_ERROR(hipMemcpy(dev_f,bids,MAXVAL*sizeof(int),hipMemcpyHostToDevice));
 	HANDLE_ERROR(hipMemcpy(dev_o,O,MAXVAL*sizeof(int),hipMemcpyHostToDevice));
//	HANDLE_ERROR(cudaMemcpy(dev_lock,cpy_lock,(10+devcount)*sizeof(int),cudaMemcpyHostToDevice));
	HANDLE_ERROR(hipMemset(dev_lock1,0,devcount*sizeof(int)));
	HANDLE_ERROR(hipMemset(dev_lock2,0,devcount*sizeof(int)));
	/*2.*/
//	printfo(MAXVAL); printf("before\n");
	dev_ptr = dev_lock1;
	double bsize = 0;
	count2 = 0;
	for(i = 2; i <= items; i++) {
		//	count =0;
		/*Generate all combinations of cardinality i*/
		// bsize = 0;
		//	c = (1 << i) -1;
		//	printf("blocks %d\n",(COMBS(c)/BLOCKSIZE)+1);
		for(c = (1 << i) -1; c <= MAXVAL;) {

			double tmp = (double) COMBS(c)/NPERBLOCK;
			
			while( bsize < MAXBLOCKSIZE && tmp > bsize) {
				bsize += 32;
			}
			int blocks =(int)  ceil((tmp/bsize));
#if __CUDA_ARCH__ < 300
			int remaindern = blocks - 65535;
			while( blocks > 65535 ) {
				bsize += 32;
				blocks =(int)  ceil((tmp/bsize));
			}
			//double bsize = BLOCKSIZE;
			if(remaindern > 0) {
				blocks =65535;
				subsetcomp22<<<remaindern,bsize,0,stream[streamcount]>>>(dev_f,dev_o,dev_ptr,c,i/2,tmp,count2,65535*bsize,bids[c]);
			}
#endif
			subsetcomp22<<<blocks,bsize,0,stream[streamcount]>>>(dev_f,dev_o,dev_ptr,c,i/2,tmp,count2,0,bids[c]);
		
			t = c | (c-1);
			c = (t + 1) | (((~t & -~t) - 1) >> (__builtin_ctz(c) + 1));
			count++;
			count2++;	
			streamcount++;
			if(streamcount >= streams)
				streamcount = 0;
			if(count2 < devcount)
				continue;
			HANDLE_ERROR(hipMemset(dev_ptr,0,devcount*sizeof(int)));

			if(dev_ptr == dev_lock1)
				dev_ptr = dev_lock2;
			else
				dev_ptr = dev_lock1;
			count2 = 0;
		}
		for (int i = 0; i < streams; ++i)
			HANDLE_ERROR(hipStreamSynchronize(stream[i]));

		HANDLE_ERROR(hipDeviceSynchronize());
		//HANDLE_ERROR(cudaMemset(dev_lock,0,devcount*sizeof(int)));
//		HANDLE_ERROR(cudaMemcpy(dev_lock,cpy_lock,(10+devcount)*sizeof(int),cudaMemcpyHostToDevice));
      
		printfo();
	}
	for (int i = 0; i < streams; ++i)
	hipStreamDestroy(stream[i]);

	HANDLE_ERROR(hipDeviceSynchronize());
	HANDLE_ERROR(hipMemcpy(f,dev_f,MAXVAL*sizeof(int),hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipMemcpy(O,dev_o,MAXVAL*sizeof(int),hipMemcpyDeviceToHost));
	//int i;
	HANDLE_ERROR(hipFree(dev_f));
	HANDLE_ERROR(hipFree(dev_o));
	HANDLE_ERROR(hipFree(dev_lock1));
	HANDLE_ERROR(hipFree(dev_lock2));

	HANDLE_ERROR(hipDeviceReset());
//	cudaProfilerStop();

//	printfo(MAXVAL);
	//printf("items %u F[%u] = %u\n",items,MAXVAL,f[MAXVAL-1]);

	return count;
}



int main(void) {
	/*Start n amount of assets*/
	dint from = NAGENTS;
	/*End amount of assets , inclusive*/
	dint MAXVAL = (2 << (from-1));

	time_t start,end,t;
	O = (dint * ) malloc(sizeof(dint)*(2 << (from-1)));
	bids = (dint * ) malloc(sizeof(dint)*(2 << (from-1)));
	f = bids;
	/*Run all tests*/

	MAXVAL = (2 << (from-1));
	gen_rand_bids(MAXVAL);
	set_singleton_bid(MAXVAL);
	printf("maxval %u from %u\n",MAXVAL,from);
	start=clock();//predefined  function in c
	int count = run_test(MAXVAL,from);
	end=clock();
	t=(end-start)/CLOCKS_PER_SEC;
	parse_wopt(MAXVAL);
	printf("\nTime taken =%lu for n= %u with count %d average per count %lf\n", (unsigned long) t,from,count,(double)t/count);
/*Reset the arrays*/

	free(O);
	free(f);

	return 0;
}








///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff///old stuff

/* __global__ void setglobal(unsigned int * f, /\*Bid value*\/ */
/* 			  unsigned int * O, /\*The move array*\/ */
/* 			  unsigned int * tf, */
/* 			  unsigned int * to, /\*The configuration*\/ */
/* 			  unsigned int conf /\*cardinality of max allowance*\/ */
/* 			  ) { */

/* 	unsigned int tid = threadIdx.x; */
/* //	while(to[threadIdx.x] == 0); */
/* //	while(tf[threadIdx.x] == 0); */
/* 	extern __shared__ unsigned int share[]; */
/* 	extern __shared__ unsigned int step[]; */
/* 	share[tid] = tf[tid]; */
/* 	step[tid]  = to[tid]; */

/* 	__syncthreads(); */
/* 	for (unsigned int s=blockDim.x/2; s>0; s>>=1) { */
/* 		if (tid < s) { */
/* 			if(share[tid] < share[tid + s]) { */
/* 				share[tid] = share[tid+s]; */
/* 				step[tid] = step[tid+s]; */
/* 			} */
/* 		} */
/* 		__syncthreads(); */
/* 	} */

/* 	if(tid == 0) { */
/* 		f[conf] = share[0]; */
/* 		O[conf] = step[0]; */

/* 		__threadfence(); */
/* 	} */
	
/* } */

/* __global__ void subsetcomp2(unsigned int * f, /\*Bid value*\/ */
/* 			   unsigned int * O, /\*The move array*\/ */
/* 			   unsigned int cardset, /\*The configuration*\/ */
/* 			    unsigned int cardmax, /\*cardinality of max allowance*\/ */
/* 			    unsigned int bidperthread, /\*how many bids should be looked at per thread*\/ */
/* 			    unsigned int maxval)  */
/* /\*The maximum value it can take, e.g tid can not be greater than 129*\/ */
/* { */
/* 	__shared__ unsigned int _conf; */
/* //	__shared__ unsigned int max[128]; */
/* //	__shared__ unsigned int step[128]; */
/* 	unsigned int i,tid; */
/* 	unsigned int subset=0,sum = 0; */
/* 	unsigned int tmpset = 0,tmpsum = 0; */
/* 	/\*thread 0 sets up variables*\/ */
/* 	if(threadIdx.x == 0) { */
/* 		/\*set up first permutaion eg 0011 for cardset 2*\/ */
/* 		_conf = (1 << cardset) -1; */
		
/* 		/\*generate the conf value*\/ */
/* 		for(i=0;i<blockIdx.x;i++) { */
/* 			tid = _conf | (_conf-1); */
/* 			_conf = (tid + 1) | (((~tid & -~tid) - 1) >> (__ffs(_conf))); */
/* 		} */
/* 		/\*set the configuration value to shared memory*\/ */
/* //		_conf = c; */
/* 		/\*put it also in the global memory*\/ */

/* 		/\*make sure that all blocks sees the change, could possibly discard it*\/ */
/* 		__threadfence(); */
/* 		O[_conf] = _conf; */
/* 		/\* printf("threadid.x\t%d\tblockDim.x\t%d\tblockIdx.x\t%d\tI\t%u\n", *\/ */
/* 		/\*        threadIdx.x, *\/ */
/* 		/\*        blockDim.x, *\/ */
/* 		/\*        blockIdx.x, *\/ */
/* 		/\*        _conf); *\/ */
/* 	} */
/* //	max[threadIdx.x] = 0; */
/* //	step[threadIdx.x] = 0; */
/* 	__syncthreads();		 */
/* //	unsigned int comb = (1 << (__popc(_conf)-1)); */
	
/* 	tid = threadIdx.x;//\*2; */
	
/* 	for(i = 1; i<= bidperthread && tid < maxval;i++) { */
/* 		tmpset = SUBSET(tid); */
/* 		if(__popc(tmpset) <= cardmax) { */
/* 			tmpsum = f[tmpset]+f[setdiff(_conf,tmpset)]; */
/* 			if(sum < tmpsum) { */
/* 				subset = tmpset; */
/* 				sum = tmpsum; */
				
/* 			} */
			
/* 		} */
/* 		tid += blockDim.x; */
/* 		//	printf("subset %u sum %u\n",tmpset,f[tmpset]+f[setdiff(_conf,tmpset)]); */
/* 	} */

/* 	__syncthreads(); */

/* //	unsigned int temp = 0; */

/* 	for(i = 0; i < blockDim.x;i++) { */
/* 		if(threadIdx.x == i) { */
/* 			if(f[_conf] < sum) { */
/* 				f[_conf] = sum; */
/* 				O[_conf] = subset; */
/* 			} */
/* 		} */
/* 		__syncthreads(); */
/* 	} */
/* } */
/* __global__ void subsetcomp(unsigned int * f, /\*Bid value*\/ */
/* 			   unsigned int * O, /\*The move array*\/ */
/* 			   unsigned int conf, /\*The configuration*\/ */
/* 			   unsigned int cardinality) /\*cardinality of max allowance*\/ */
/* { */
/* 	unsigned int max; */
/* 	/\*tmp_max is a temporary max variable that is not subject to mutex lock*\/ */
/* //	__shared__ unsigned int tmp_max; */
/* 	__shared__ unsigned int tmpstore[192]; */
/* 	/\* printf("threadid.x\t%d\tblockDim.x\t%d\tblockIdx.x\t%d\tI\t%d\n", *\/ */
/* 	/\*        threadIdx.x, *\/ */
/* 	/\*        blockDim.x, *\/ */
/* 	/\*        blockIdx.x, *\/ */
/* 	/\*        I); *\/ */
/* 	unsigned int subset = (~conf+(I+1))&conf; */
/* 	if(__popc(subset) <= cardinality) */
/* 		tmpstore[I] = f[setdiff(conf,subset)] + f[subset]; */
/* 	__syncthreads(); */
/* //	__threadfence(); */
/* 	if(threadIdx.x == 0) { */
/* 		unsigned int i = 0; */
/* 		unsigned int c = 0; */
/* 		for(;i < blockDim.x;i++) */
/* 		{ */
/* 			if(tmpstore[i] > max) { */
/* 				max = tmpstore[i]; */
/* 				c = i; */
/* 			} */
/* 		} */
/* 		if(atomicMax(&f[conf],max) < max) { */
/* 			subset = (~conf+((c+blockDim.x*blockIdx.x)+1))&conf; */
/* 			atomicExch(&O[conf],subset); */
/* 		} */
/* 	} */
/* } */
/* __global__ void add(unsigned int * p, unsigned int * f, unsigned int * O) */
/* { */
/* 	int tid = blockIdx.x; */

/* 	unsigned int conf = p[tid]; */
/* 	unsigned int card = (unsigned int) __popc(conf)/2; */
/* 	unsigned int combinations = 1 << (__popc(conf) -1); */
/* 	unsigned int max = f[conf]; */
/* 	unsigned int set = p[tid]; */

/* 	unsigned int tmp = 0; */
/* 	unsigned int subset; */
/* 	unsigned int inverse = ~set; */
/* 	unsigned int i; */
/* 	if(max == 0) { */
/* 		printf("hello"); */
/* 		return; */
/* 	} */
/* /\**\/ */
	
/* 	for(i = 1;i<combinations; i++) { */
/* 		subset = (inverse+i)&conf; */
/* 		if(__popc(subset) > card) */
/* 			continue; */
/* 		tmp = f[setdiff(conf,subset)] + f[subset]; */
/* 		if(max < tmp) { */
/* 			max = tmp; */
/* 			set = subset; */
/* 		} */
/* 	} */
/* 	f[conf] = max; */
/* 	O[conf] = set; */
/* } */
