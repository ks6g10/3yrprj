#include <stdio.h> 
#include <string.h> // for ffs
#include <stdint.h>
#include <math.h>
#include <time.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

/*Debug enabled gives more print statements of bids and how the "Matrix" gets evaluated*/
#define DEBUG 0
#define TRUE 1
#define FALSE 0
/*Test sets all bids to one, which should give you n=|ITEMS| bids on output*/
#define TEST 1
/*Defines from 0-Range the random will give out*/
#define RANGE 10000
#define ITEMS 25

#define MAX (2 << (ITEMS-1))
#if ITEMS < 8
#define dint uint8_t  
#elif ITEMS < 16
#undef dint
#define dint uint16_t
#elif ITEMS < 32
#undef dint
#define dint uint32_t
#elif ITEMS > 32
#undef dint
#define dint uint64_t
#endif

#define SUBSET(X)((~_conf+(X+1))&_conf)
#define SETSUM(X)(f[setdiff(_conf,X)]+f[X])
#define I (threadIdx.x + blockDim.x * blockIdx.x)

static void HandleError( hipError_t err, const char *file,int line ) {
	if (err != hipSuccess) {
		printf( "%s in %s at line %d\n", hipGetErrorString( err ),file, line );
		exit( EXIT_FAILURE );
	}
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))



/*		           0 1 2 3 4 5 6 7 8			*/
dint * bids;
// dint bids[MAX] ={0,2,3,6,4,7,6,9}; //conf 5 and 2
//dint bids[MAX] =  {0,2,3,6,4,6,6,9}; //conf 3 and 4
// dint bids[MAX] =  {0,20,3,6,4,6,10,9}; //conf 1 and 6
dint * f;
dint * O;
 
struct _stack {
	dint conf;
	struct _stack * next;
} typedef stack;

struct _locklist {
	unsigned int size;
	unsigned int conf;
	unsigned int * dev_f;
	unsigned int * dev_o;
	struct _locklist * next;
} typedef locklist;

struct _lockstruct {
	unsigned int * dev_lock;
	struct _lockstruct * next;
} typedef lockstruct;


#define setdiff(seta,setb) (seta & ~setb)

inline  dint cardinality( dint seta) {
	return __builtin_popcount(seta);
}

void gen_rand_bids(dint MAXVAL) {
	register dint i = 0;
#if TEST
	for(i = 1; i < MAXVAL;i++) {
		bids[i] = 1;
		O[i] = i;
	}
//	bids[1] =0;
//	bids[2] = 0;
//	bids[32769] = 20;
#else
	for(i = 1; i < MAXVAL;i++) {
		bids[i] = rand() % RANGE;
		O[i] = i;
	}
	for(i = 1; i < MAXVAL;i*=2) {
		bids[i] = rand() % RANGE;

	}
#endif

#if DEBUG
	printf("i =\t");
	for(i = 1; i < MAX;i++) {
		printf("%u\t",i);
	}
	printf("\n");
	printf("val =\t");
	for(i = 1; i < MAX;i++) {
		printf("%u\t",bids[i]);
	}
	printf("\n");
#endif
}

const char *btb(dint y)
{
	int x = y;
	static char b[9];
	b[0] = '\0';
	
	int z;
	for (z = 128; z > 0; z >>= 1)
	{
		strcat(b, ((x & z) == z) ? "1" : "0");
	}	
	return b;
}

inline void printfo() {
#if DEBUG
	dint i;
	printf("i\t");
	for(i =1; i < MAX; i++) {
		printf("%u\t",i);
	}
	printf("\n");
	printf("f[]\t");
	for(i =1; i < MAX; i++) {
		printf("%u\t",f[i]);
	}
	printf("\n");
	printf("O[]\t");
	for(i =1; i < MAX; i++) {
		printf("%u\t",O[i]);
	}
	printf("\n");
#endif
}

/*Sets all bids with one element in it, |n| = 1*/
inline void set_singleton_bid(dint MAXVAL) {
	register  dint i;
	for(i =1; i< MAXVAL; i*=2) {
		f[i] = bids[i];
		if(bids[i] > 0)
			O[i] = i;
	}
}

/* conf a e.g. 1101
 * (~a+i) & a gives a subset of a
 *  i is a integer from 1 to |a|
 *
 * ~1101 = 0010
 * i = 0001
 * (0010+0001)&1101 =
 * 0011&1101 = 0001
 *
 *i = 0011
 * (0010+0011)&1101
 *(0101)&1101 = 0101
 */



void printfo(dint MAXVAL) {
	int i;
	printf("\n");
	for(i = 1; i< MAXVAL; i++)
	{
		printf("Bid[%d]\t%u\tF[%d]\t%u\tO[%d]\t%u\tbin\t%s\n",i,bids[i],i,f[i],i,O[i],btb(i));

	}
}

void parse_wopt(dint MAXVAL) {
	printf("parse maxval = %u\n",MAXVAL);
	//wopt at start contain MAX at wopt[0] which is the combination that goes in bids[wopt[n]]
	stack * root =(stack *) malloc(sizeof(stack));
	stack * sroot = NULL;
	stack * scurr = NULL;
	//DO N	OT REMOVE -1
	root->conf = (MAXVAL)-1;
	int count = 0;
	root->next = NULL;
	stack * curr = root;
	while(curr) {
		dint conf = curr->conf;
		if(conf == 0) {
			printf("EXIT FAILURE\n");
			return;
		}
		/*if something is wrong*/
		if(count > 40) {
			fprintf(stderr,"Something went wrong at line %d in %s\n",__LINE__,__FILE__);
			return;
		}
			printf("curr %u\t",curr->conf);
		if(conf != O[conf]) {

			dint diff = setdiff(conf,O[conf]);
			curr->conf = O[conf];
			stack * tmp = (stack *) malloc(sizeof(stack));
			printf("diff %u\t O[conf] %u f %u\n",diff,O[conf],f[conf]);
			tmp->conf = diff;
			tmp->next = curr;
			root = tmp;
			curr = root;
			count++;
			continue;
		}
		if(sroot == NULL) {
			sroot = curr;
			scurr = curr;
		} else {
			/*set next pointer to the next singelton*/
			scurr->next = curr;
		}

		/*set the current singleton to be the pointer*/
		scurr = curr;
		curr = curr->next;
		/*clear the pinter to avoid infinite loop if cross-referenced*/
		scurr->next = NULL;

	}
	curr = sroot;
	dint tmp = 0;
	while(curr != NULL) {
		printf("conf %u value %u\n",curr->conf,bids[curr->conf]);
		tmp++;
		stack * tmp = curr;
		curr = curr->next;
		free(tmp);
	}
	printf("n = %u\n",tmp);
}


/*
 *
 * 1. gen all combinations of card n
 * 2. for each combination, generate all subset with condition |s| < |c|/2
 * 3. for each subset check if |s| < |c|/2 then compute the sum
 * 4. 
 *
 *
 *
 */ 

#define SET_TEST_FETCH(STEP,S1,S2) {				\
	S1 = S2 = 0U;						\
	STEP = SUBSET(ispec);					\
	if(__popc(STEP) <= cardmax && ispec <= maxval) {	\
	S1 = f[setdiff(_conf,STEP)];				\
	S2 = f[STEP];						\
	}							\
	ispec += blockDim.x;					\
	}



#define MAXBLOCKSIZE 256U
#define NAGENTS 24 
#define NSTREAMS 16 
#define NPERBLOCK 10


__global__ void subsetcomp22(
		 	     unsigned int * __restrict__ f, /*Bid value*/
			     unsigned int * __restrict__ O, /*The move array*/
			     unsigned int * __restrict__ lock,
			     unsigned int _conf, /*The configuration*/
			     unsigned int cardmax, /*cardinality of max allowance*/
			     unsigned int maxval,
			     unsigned int count,
			     unsigned int offset,
			     unsigned int defbid)
{

	__shared__ unsigned int share[MAXBLOCKSIZE];
	__shared__ unsigned int step[MAXBLOCKSIZE];     

    		/* printf("threadid.x\t%d\tblockDim.x\t%d\tblockIdx.x\t%d\tI\t%u\n", */
	/*        threadIdx.x, */
	/*        blockDim.x, */
	/*        blockIdx.x, */
	/*        conf); */
//	unsigned int i = I+offset;
//	unsigned int ispec = (threadIdx.x*NPERBLOCK + blockDim.x * blockIdx.x) + offset;
	unsigned int ispec = (threadIdx.x + blockDim.x * blockIdx.x) + offset;
//	unsigned int i = I+offset;
	/*subset var, but also for indexing later  on*/
//	unsigned int s;// = SUBSET(i);


//	unsigned int tid = threadIdx.x;

	unsigned int val11,val12,step1,val21,val22,step2,val31,val32,step3,val41,val42,step4;
	unsigned int val51,val52,step5;
	/* unsigned int vals[NPERBLOCK][2]; */
	/* unsigned int step[NPERBLOCK]; */
	
	step[threadIdx.x] = share[threadIdx.x] = 0U;
	if(ispec <= maxval) {

		/* unsigned int tmp = ispec; */
		/* tmp += ((maxval - ispec > NPERBLOCK) ? NPERBLOCK : maxval - ispec); */

		/* unsigned int i; */
		/* for(;ispec<tmp; ispec++) { */
		/* 	step[i] = SUBSET(ispec); */
		/* 	if(__popc(step1) <= cardmax) { */
		/* 		vals[isp] */

		/* 	} */
		/* } */
		SET_TEST_FETCH(step1,val11,val12);
 		/* step1 = SUBSET(ispec); */
		/* val11 = val12 = 0U; */
		/* if(__popc(step1) <= cardmax) { */
		/* 	val11 = f[setdiff(_conf,step1)]; */
		/* 	val12 = f[step1];	 */
		/* } */
		/* ispec += blockDim.x; */
		SET_TEST_FETCH(step2,val21,val22);
		
		/* step2 = SUBSET(ispec); */
		/* val21 = val22 = 0U; */
		/* if(__popc(step2) <= cardmax && ispec <= maxval) { */
		/* 	val21 = f[setdiff(_conf,step2)]; */
		/* 	val22 = f[step2];	 */
		/* } */
		/* ispec += blockDim.x; */
		SET_TEST_FETCH(step3,val31,val32);
		/* step3 = SUBSET(ispec); */
		/* val31 = val32 = 0U; */
		/* if(__popc(step3) <= cardmax && ispec <= maxval) { */
		/* 	val31 = f[setdiff(_conf,step3)]; */
		/* 	val32 = f[step3];	 */
		/* } */
		/* ispec += blockDim.x; */
		SET_TEST_FETCH(step4,val41,val42);
		/* step4 = SUBSET(ispec); */
		/* val41 = val42 = 0U; */
		/* if(__popc(step4) <= cardmax && ispec <= maxval) { */
		/* 	val41 = f[setdiff(_conf,step4)]; */
		/* 	val42 = f[step4];	 */
		/* } */
		/* ispec += blockDim.x; */

		/*step5*/
		SET_TEST_FETCH(step5,val51,val52);
		/* step5 = SUBSET(ispec); */
		/* val51 = val52 = 0U; */
		/* if(__popc(step5) <= cardmax && ispec <= maxval) { */
		/* 	val51 = f[setdiff(_conf,step5)]; */
		/* 	val52 = f[step5];	 */
		/* } */
		/* ispec += blockDim.x; */


		val11 += val12;
		val21 += val22;
		

		if(val21 > val11) {
			val11 = val21; 
			step1 = step2;
		}

		/*pipelined fetch*/
		SET_TEST_FETCH(step2,val21,val22);
		/* val21 = val22 = 0U; */
		/* step2 = SUBSET(ispec); */
		/* if(__popc(step2) <= cardmax && ispec <= maxval) { */
		/* 	val21 = f[setdiff(_conf,step2)]; */
		/* 	val22 = f[step2];	 */
		/* } */
		/* ispec += blockDim.x; */
		

		val31 += val32;		
		if(val31 > val11) {
			val11 = val31;
			step1 = step3;
		}

		/*pipelined fetch*/
		SET_TEST_FETCH(step3,val31,val32);
		/* val31 = val32 = 0U; */
		/* step3 = SUBSET(ispec); */
		/* if(__popc(step3) <= cardmax && ispec <= maxval) { */
		/* 	val31 = f[setdiff(_conf,step3)]; */
		/* 	val32 = f[step3];	 */
		/* } */
		/* ispec += blockDim.x; */

		val41 += val42;
		if(val41 > val11) {
			val11 = val41;
			step1 = step4;
		}

		/*pipelined fetch*/
		SET_TEST_FETCH(step4,val41,val42);
		/* val41 = val42 = 0U; */
		/* step4 = SUBSET(ispec); */
		/* if(__popc(step4) <= cardmax && ispec <= maxval) { */
		/* 	val41 = f[setdiff(_conf,step4)]; */
		/* 	val42 = f[step4];	 */
		/* } */
		/* ispec += blockDim.x; */

		val51 += val52;
		if(val51 > val11) {
			val11 = val51;
			step1 = step5;
		}

		/*step5*/
		SET_TEST_FETCH(step5,val51,val52);
		/* step5 = SUBSET(ispec); */
		/* val51 = val52 = 0U; */
		/* if(__popc(step5) <= cardmax && ispec <= maxval) { */
		/* 	val51 = f[setdiff(_conf,step5)]; */
		/* 	val52 = f[step5];	 */
		/* } */
		/* ispec += blockDim.x; */


		share[threadIdx.x] = val11;
		step[threadIdx.x] = step1;

		/*pipelined fetch*/
		SET_TEST_FETCH(step1,val11,val12);
		/* step1 = SUBSET(ispec); */
		/* val11 = val12 = 0U; */
		/* if(__popc(step1) <= cardmax && ispec <= maxval) { */
		/* 	val11 = f[setdiff(_conf,step1)]; */
		/* 	val12 = f[step1];	 */
		/* } */
//		ispec += blockDim.x;

		val21 += val22;
		val31 += val32;
		if(val31 > val21) {
			val21 = val31; 
			step2 = step2;
		}
		val41 += val42;
		if(val41 > val21) {
			val21 = val41;
			step2 = step4;
		}

		val51 += val52;
		if(val51 > val11) {
			val11 = val51;
			step1 = step5;
		}

		val11 += val12;
		if(val11 > val21) {
			val21 = val11;
			step2 = step1;
		}

		if(val21 > share[threadIdx.x]) {
			share[threadIdx.x] = val21;
			step[threadIdx.x] = step2;
		}

	}
	ispec = I;
       
	val11= blockDim.x >> 1U;
	__syncthreads();
#pragma unroll
	for (; val11>0U; val11>>=1U) {
		if (threadIdx.x < val11 && (ispec <= maxval)) {
			if(share[threadIdx.x] < share[threadIdx.x + val11]) {
				step[threadIdx.x] = step[threadIdx.x+val11];
				share[threadIdx.x] = share[threadIdx.x+val11];
			}
		}
		__syncthreads();
	}

	if(threadIdx.x == 0U) {
		val11 = share[0U];
		if(defbid>val11)
			return;
		if(lock[count] < val11) {
			if(atomicMax(&(lock[count]),val11) < val11) {
				O[_conf] = step[0U];
				f[_conf] = val11;
				
			} 
		}
	}
}



#define COMBS(X) ((1 << cardinality(X)) - 1)

int run_test(dint MAXVAL,dint items) {
/*Setup the environment*/
	//dint perm[MAXVAL];
	printfo();
	register unsigned int i, c,t,count =0;
 	unsigned int *dev_f,*dev_o;

	i = items/2;
	count = 0;

	HANDLE_ERROR(hipDeviceReset());

	unsigned int * dev_lock1,*dev_lock2,*dev_ptr;
	const	unsigned int devcount = 1024;// count;
	unsigned int streams = NSTREAMS;
	unsigned int count2 = 0;
	unsigned int streamcount = 0;
	hipStream_t stream[streams];
	for(int i = 0;i < streams; i++)
		HANDLE_ERROR(hipStreamCreate(&stream[i]));
	printf("count %u\n",devcount);
	count = 0;
	HANDLE_ERROR(hipMalloc((void **)&dev_lock1,(10+devcount)*sizeof(int)));
	HANDLE_ERROR(hipMalloc((void **)&dev_lock2,(10+devcount)*sizeof(int)));
 	HANDLE_ERROR(hipMalloc((void **)&dev_f, MAXVAL*sizeof(int)));
 	HANDLE_ERROR(hipMalloc((void **)&dev_o, MAXVAL*sizeof(int)));

 	HANDLE_ERROR(hipMemcpy(dev_f,bids,MAXVAL*sizeof(int),hipMemcpyHostToDevice));
 	HANDLE_ERROR(hipMemcpy(dev_o,O,MAXVAL*sizeof(int),hipMemcpyHostToDevice));

	HANDLE_ERROR(hipMemset(dev_lock1,0,devcount*sizeof(int)));
	HANDLE_ERROR(hipMemset(dev_lock2,0,devcount*sizeof(int)));
	/*2.*/
//	printfo(MAXVAL); printf("before\n");
	dev_ptr = dev_lock1;
	double bsize = 0;
	count2 = 0;
	for(i = 2; i <= items; i++) {
		for(c = (1 << i) -1; c <= MAXVAL;) {

			double tmp = (double) COMBS(c)/NPERBLOCK;
			
			while( bsize < MAXBLOCKSIZE && tmp > bsize) {
				bsize += 32;
			}
			int blocks =(int)  ceil((tmp/bsize));
#if __CUDA_ARCH__ < 300
			int remaindern = blocks - 65535;
			while( blocks > 65535 ) {
				bsize += 32;
				blocks =(int)  ceil((tmp/bsize));
			}
			//double bsize = BLOCKSIZE;
			if(remaindern > 0) {
				blocks =65535;
				subsetcomp22<<<remaindern,bsize,0,stream[streamcount]>>>(dev_f,dev_o,dev_ptr,c,i/2,tmp,count2,65535*bsize,bids[c]);
			}
#endif
			subsetcomp22<<<blocks,bsize,0,stream[streamcount]>>>(dev_f,dev_o,dev_ptr,c,i/2,tmp,count2,0,bids[c]);
		
			t = c | (c-1);
			c = (t + 1) | (((~t & -~t) - 1) >> (__builtin_ctz(c) + 1));
			count++;
			count2++;	
			streamcount++;
			if(streamcount >= streams)
				streamcount = 0;
			if(count2 < devcount)
				continue;
			HANDLE_ERROR(hipMemset(dev_ptr,0,devcount*sizeof(int)));

			if(dev_ptr == dev_lock1)
				dev_ptr = dev_lock2;
			else
				dev_ptr = dev_lock1;
			count2 = 0;
		}
		for (int i = 0; i < streams; ++i)
			HANDLE_ERROR(hipStreamSynchronize(stream[i]));

		HANDLE_ERROR(hipDeviceSynchronize());
		printfo();
	}
	for (int i = 0; i < streams; ++i)
	hipStreamDestroy(stream[i]);

	HANDLE_ERROR(hipDeviceSynchronize());
	HANDLE_ERROR(hipMemcpy(f,dev_f,MAXVAL*sizeof(int),hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipMemcpy(O,dev_o,MAXVAL*sizeof(int),hipMemcpyDeviceToHost));
	//int i;
	HANDLE_ERROR(hipFree(dev_f));
	HANDLE_ERROR(hipFree(dev_o));
	HANDLE_ERROR(hipFree(dev_lock1));
	HANDLE_ERROR(hipFree(dev_lock2));

	HANDLE_ERROR(hipDeviceReset());
//	printfo(MAXVAL);
	return count;
}



int main(void) {
	/*Start n amount of assets*/
	dint from = NAGENTS;
	/*End amount of assets , inclusive*/
	dint MAXVAL = (2 << (from-1));

	time_t start,end,t;
	O = (dint * ) malloc(sizeof(dint)*(2 << (from-1)));
	bids = (dint * ) malloc(sizeof(dint)*(2 << (from-1)));
	f = bids;
	/*Run all tests*/

	MAXVAL = (2 << (from-1));
	gen_rand_bids(MAXVAL);
	set_singleton_bid(MAXVAL);
	printf("maxval %u from %u\n",MAXVAL,from);
	start=clock();//predefined  function in c
	int count = run_test(MAXVAL,from);
	end=clock();
	t=(end-start)/CLOCKS_PER_SEC;
	parse_wopt(MAXVAL);
	printf("\nTime taken =%lu for n= %u with count %d average per count %lf\n", (unsigned long) t,from,count,(double)t/count);
/*Reset the arrays*/

	free(O);
	free(f);

	return 0;
}
