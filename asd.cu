#include <stdio.h>
#include <string.h> // for ffs
#include <stdint.h>
#include <math.h>
#include <time.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

/*Debug enabled gives more print statements of bids and how the "Matrix" gets evaluated*/
#define DEBUG 0
#define TRUE 1
#define FALSE 0
/*Test sets all bids to one, which should give you n=|ITEMS| bids on output*/
#define TEST 1
/*Defines from 0-Range the random will give out*/
#define RANGE 10000
#define ITEMS 25

//#define MAX (2 << (ITEMS-1))
#if ITEMS < 8
#define dint uint8_t
#elif ITEMS < 16
#undef dint
#define dint uint16_t
#elif ITEMS < 32
#undef dint
#define dint uint32_t
#endif

static void HandleError( hipError_t err, const char *file,int line ) {
	if (err != hipSuccess) {
		printf( "%s in %s at line %d\n", hipGetErrorString( err ),file, line );
		exit( EXIT_FAILURE );
	}
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


/*		           0 1 2 3 4 5 6 7 8			*/
uint32_t  * O;
unsigned short  * f, * bids;
// dint bids[MAX] ={0,2,3,6,4,7,6,9}; //conf 5 and 2
//dint bids[MAX] =  {0,2,3,6,4,6,6,9}; //conf 3 and 4
// dint bids[MAX] =  {0,20,3,6,4,6,10,9}; //conf 1 and 6

struct _stack {
	dint conf;
	struct _stack * next;
} typedef stack;


inline  dint cardinality( dint seta) {
	return __builtin_popcount(seta);
}
int indexa =0;
void gen_rand_bids(dint MAXVAL) {
	register dint i = 0;
#if TEST
	unsigned int seed = (unsigned)time ( NULL );
	srand(seed);
	indexa++;
	for(i = 1; i < MAXVAL;i++) {
		bids[i] = 1;//rand()%10+1;
		O[i] = i;
	}

	//indexa = rand() % MAXVAL;
	bids[indexa] = 100;
	printf("index %d \n",indexa);
	if(indexa >= MAXVAL) {
		printf("No error\n");
		exit(0);
	}

//	bids[1] =0;
//	bids[2] = 0;
//	bids[32769] = 20;
#else
	for(i = 1; i < MAXVAL;i++) {
		bids[i] = rand() % RANGE;
		O[i] = i;
	}
	for(i = 1; i < MAXVAL;i*=2) {
		bids[i] = rand() % RANGE;

	}
#endif

}

const char *btb(dint y)
{
	int x = y;
	static char b[9];
	b[0] = '\0';

	int z;
	for (z = 128; z > 0; z >>= 1)
	{
		strcat(b, ((x & z) == z) ? "1" : "0");
	}
	return b;
}

/*Sets all bids with one element in it, |n| = 1*/
inline void set_singleton_bid(dint MAXVAL) {
	register  dint i;
	for(i =1; i< MAXVAL; i*=2) {
		f[i] = bids[i];
		if(bids[i] > 0)
			O[i] = i;
	}
}



void printfo(dint MAXVAL) {
	int i;
	printf("\n");
	for(i = 1; i< MAXVAL; i++)
	{
		printf("Bid[%d]\t%u\tF[%d]\t%u\tO[%d]\t%u\tbin\t%s\n",i,bids[i],i,f[i],i,O[i],btb(i));

	}
}


#define SUBSET(Y,X)(((~Y+1)+(X))&Y)

#define setdiff(seta,setb) (seta & ~setb)
//maxreg 32
//256
#define MAXBLOCKSIZE (1024)
//32
#define WARPSIZE (32)
//0
#define MIN_BLOCKS_PER_MP 4
#define NAGENTS (25)
//32  
#define NSTREAMS (16)
//2
#define NPERBLOCK (2)
#define confpwarp (2)
//32
#define CONFPKERNEL ((MAXBLOCKSIZE/32)*confpwarp)
//4
#define parasplittings (8)
#define NPARALLELCONF (4)
#define TIMING (0)

#define COMP(Z) {							\
		if(shared_value[tid][Z] < shared_value[tid+i][Z]) {	\
			shared_value[tid][Z] = shared_value[tid+i][Z];	\
		}							\
	}

#define CHECKPOINT(X) {							\
		stop_time = clock();					\
		if(tid == 0 && blockIdx.x == 0) {			\
			total = stop_time - start_time;			\
			printf(X,stop_time - start_time);		\
		}							\
		start_time =clock();					\
	}

#if (TIMING == 0)
#undef CHECKPOINT
#define CHECKPOINT(X) {}
#endif

#define MAX(X,Y) ((X) > (Y) ? (X) : (Y))


template<int blockSize, int overlastval>
__global__ void
__launch_bounds__(blockSize,MIN_BLOCKS_PER_MP)
	subsetcomp33(
		/*0*/	unsigned short * __restrict__ f, /*Bid value*/
		/*5*/	const unsigned int splittings,		
		const unsigned int lastval,//the value a permutation can not exceed.
		const unsigned int card,
		const unsigned int conf1,
		const unsigned int conf2,
		const unsigned int conf3,
		const unsigned int conf4
)
{
	__shared__ unsigned int conf[confpwarp][(blockSize/32)+1];
	__shared__ unsigned short value[confpwarp][(blockSize/32)+1];
	__shared__  unsigned char shift [(blockSize/32)*confpwarp][NAGENTS];// the shift matrix/array
	const unsigned int tid = threadIdx.x;
	const unsigned int laneId = (tid%32);
	const unsigned int warpId = tid/32;
	const unsigned int specsplittings = (!!(splittings%32))+(splittings/32);
//(laneId < splittings)*(splittings/32)+(laneId < (splittings%32));
	const unsigned int initsplit = tid*specsplittings;//+(laneId >= (splittings%32))*(splittings%32);
	// const unsigned int specsplittings = (splittings/32)+!!(splittings%32);
	// const unsigned int initsplit = tid*specsplittings;
	unsigned int leafsplit[2];
//	unsigned int rootsplit[confpwarp];
	unsigned int rvalue[confpwarp][parasplittings];
	/*Thread 0 of each warp*/
	if(!tid) {
		unsigned int tmp;
		unsigned int conftmp;
		if(blockIdx.x == 0) {
			conftmp = conf[0][warpId] = conf1;
		} else if(blockIdx.x == 1) {
			conftmp = conf[0][warpId] = conf2;
		} else if(blockIdx.x == 2) {
			conftmp = conf[0][warpId] = conf3;
		} else {
			conftmp = conf[0][warpId] = conf4;
		}
		int x;
		tmp = conftmp | (conftmp-1);
		conftmp = (tmp + 1) | (((~tmp & -~tmp) - 1) >> (__ffs(conftmp)));
		conf[1][warpId] = conftmp;
		for(x =1; x < (blockSize/32);x++) {
			tmp = conftmp | (conftmp-1);
			conftmp = (tmp + 1) | (((~tmp & -~tmp) - 1) >> (__ffs(conftmp)));
			conf[0][x] = conftmp;
			// if(conftmp > lastval) {
			// 	conf[1][x] = conftmp;
			// 	continue;
			// }
			tmp = conftmp | (conftmp-1);
			conftmp = (tmp + 1) | (((~tmp & -~tmp) - 1) >> (__ffs(conftmp)));
			conf[1][x] = conftmp;
		}
	}
	__syncthreads();
	if(conf[0][warpId] > lastval) {
		return;
	}
	if((__popc(conf[1][warpId] & conf[0][warpId]) == (card - 1)) && conf[1][warpId] < lastval) {
		if(laneId == 0) {
			unsigned int index;
			unsigned int count = 0;
			unsigned int conftmp = conf[0][warpId] & conf[1][warpId];

			//if(conf[1][warpId] < lastval) {
			value[1][warpId] = f[conf[1][warpId]];
			//}
			value[0][warpId] = f[conf[0][warpId]];
//#pragma unroll
			while(conftmp) {
				index = __ffs(conftmp) - 1; //find which index is first bit
				conftmp &= ~(1 << index);//set nth bit to 0
				shift[warpId][count] = index;
				shift[warpId+1][count] = index;
				count++;
			}
		}
	} else if(laneId < confpwarp) { //generate the shift arrays
		unsigned int index;
		unsigned int conftmp = conf[laneId][warpId];

		if(conftmp < lastval) {
			value[laneId][warpId] = f[conftmp];
		}
#pragma unroll
		for(int x = 0; x < card;x++) {//could put card in template to unroll
			index = __ffs(conftmp) - 1; //find which index is first bit
			conftmp &= ~(1 << index);//set nth bit to 0
			shift[warpId+laneId][x] = index;
		}

	}
	//__syncthreads();



	if(conf[1][warpId] < lastval) {
		unsigned int index;
		unsigned int splittmp= initsplit;
		leafsplit[0] = leafsplit[1] = 0;
		while(splittmp) {
			index = __ffs(splittmp)-1;
			leafsplit[1] += (1 << shift[warpId+1][index]);
			leafsplit[0] += (1 << shift[warpId][index]);//CHECK
			splittmp &= ~(1 << index);
		}
		if((__popc(conf[1][warpId] & conf[0][warpId]) == (card - 1))) {
			leafsplit[0] = leafsplit[1] = SUBSET((conf[1][warpId] & conf[0][warpId]),leafsplit[1]);
		} else {
			leafsplit[1] = SUBSET(conf[1][warpId],leafsplit[1]);
			leafsplit[0] = SUBSET(conf[0][warpId],leafsplit[0]);
		}
	} else {
		unsigned int index;
		unsigned int splittmp= initsplit;
		leafsplit[0] = 0;
		while(splittmp) {
			index = __ffs(splittmp)-1;
			leafsplit[0] += (1 << shift[warpId][index]);//CHECK
			splittmp &= ~(1 << index);
		}
		leafsplit[0] = SUBSET(conf[0][warpId],leafsplit[0]);
	}
	if(!specsplittings) {
		return;
	}

	int x,y;


	for(x = 0; x < specsplittings;x += parasplittings) {

		if((__popc(conf[1][warpId] & conf[0][warpId]) == (card - 1)) && (conf[1][warpId] < lastval)) {
#pragma unroll 8
			for(y = 0;y < parasplittings ;y++) {
				rvalue[0][y] = 	rvalue[1][y] = 0;
				if(x+y+initsplit < splittings) {
					int tmp = __popc(leafsplit[0]);
					if((NAGENTS-card) <= tmp && (NAGENTS-card) <= (card-tmp)) {
						rvalue[1][y] = rvalue[0][y] = f[leafsplit[0]];
						rvalue[0][y] += f[setdiff(conf[0][warpId],leafsplit[0])];
						rvalue[1][y] += f[setdiff(conf[1][warpId],leafsplit[1])];
					}
					leafsplit[1] = SUBSET((conf[1][warpId] & conf[0][warpId]),leafsplit[1]);
					leafsplit[0] = SUBSET((conf[1][warpId] & conf[0][warpId]),leafsplit[0]);
				}
			}

		} else {			
#pragma unroll 8			
			for(y = 0;y < parasplittings ;y++) {
				rvalue[0][y] = rvalue[1][y] = 0;
				if(x+y+initsplit < splittings) {
					int tmp = __popc(leafsplit[0]);
					if((NAGENTS-card) <= tmp && (NAGENTS-card) <= (card-tmp)) {
						rvalue[0][y] = f[leafsplit[0]];
						rvalue[0][y] += f[setdiff(conf[0][warpId],leafsplit[0])];
						if(overlastval) {
						if(conf[1][warpId] < lastval) {
							rvalue[1][y] = f[leafsplit[1]];	
							rvalue[1][y] += f[setdiff(conf[1][warpId],leafsplit[1])];
						}
						} else {
							rvalue[1][y] = f[leafsplit[1]];	
							rvalue[1][y] += f[setdiff(conf[1][warpId],leafsplit[1])];
						}
					}
					leafsplit[0] = SUBSET(conf[0][warpId],leafsplit[0]);
					leafsplit[1] = SUBSET(conf[1][warpId],leafsplit[1]);
					
				}
			}

		}

#pragma unroll	       
		for(y = 1;y < parasplittings;y++) {
			if(x+y+initsplit >= splittings) {
				continue;
			}
			if(rvalue[0][0] < rvalue[0][y]) {
				rvalue[0][0] = rvalue[0][y];
			}
			if(rvalue[1][0] < rvalue[1][y]) {
				rvalue[1][0] = rvalue[1][y];
			}
		}


		// 	if(rvalue[1][0] + rvalue[0][0] >= 100) {

		// 	printf("hello %u\n",rvalue[1][0] + rvalue[0][0]);
		// }
		if(__ballot( ( (rvalue[0][0] > value[0][warpId]) || (rvalue[1][0] > value[1][warpId]) ) ) == 0) {
			continue;
		}


		if(__ballot( ( rvalue[0][0] > value[0][warpId] ) ) ) {
			rvalue[0][1] = __shfl_xor((int)rvalue[0][0],16,32);
			rvalue[0][0] = MAX(rvalue[0][0],rvalue[0][1]);
			rvalue[0][1] = __shfl_xor((int)rvalue[0][0],8,32);
			rvalue[0][0] = MAX(rvalue[0][0],rvalue[0][1]);
			rvalue[0][1] = __shfl_xor((int)rvalue[0][0],4,32);
			rvalue[0][0] = MAX(rvalue[0][0],rvalue[0][1]);
			rvalue[0][1] = __shfl_xor((int)rvalue[0][0],2,32);
			rvalue[0][0] = MAX(rvalue[0][0],rvalue[0][1]);
			rvalue[0][1] = __shfl_xor((int)rvalue[0][0],1,32);
			rvalue[0][0] = MAX(rvalue[0][0],rvalue[0][1]);
		}
		
		if(__ballot( ( rvalue[1][0] > value[1][warpId] ) ) ) {
			if(conf[1][warpId] < lastval) {
			rvalue[1][1] = __shfl_xor((int)rvalue[1][0],16,32);
			rvalue[1][0] = MAX(rvalue[1][0],rvalue[1][1]);
			rvalue[1][1] = __shfl_xor((int)rvalue[1][0],8,32);
			rvalue[1][0] = MAX(rvalue[1][0],rvalue[1][1]);
			rvalue[1][1] = __shfl_xor((int)rvalue[1][0],4,32);
			rvalue[1][0] = MAX(rvalue[1][0],rvalue[1][1]);
			rvalue[1][1] = __shfl_xor((int)rvalue[1][0],2,32);
			rvalue[1][0] = MAX(rvalue[1][0],rvalue[1][1]);
			rvalue[1][1] = __shfl_xor((int)rvalue[1][0],1,32);
			rvalue[1][0] = MAX(rvalue[1][0],rvalue[1][1]);
			}
		}
// #pragma unroll
// 		for(int i = 16;i >=1;i >>=1) {
// 				rvalue[0][1] = __shfl_xor((int)rvalue[0][0],i,32);				
// 				if(rvalue[0][1] > rvalue[0][0]) {
// 					rvalue[0][0] = rvalue[0][1];
// 				}

// 				if(!overlastval) {
// 					rvalue[1][1] = __shfl_xor((int)rvalue[1][0],i,32);
// 					if(rvalue[1][1] > rvalue[1][0]) {
// 						rvalue[1][0] = rvalue[1][1];
// 					}
					
// 				} else {
// 					if(conf[1][warpId] < lastval) {
// 						rvalue[1][1] = __shfl_xor((int)rvalue[1][0],i,32);
// 						if(rvalue[1][1] > rvalue[1][0]) {
// 							rvalue[1][0] = rvalue[1][1];
// 						}
// 					}
// 				}
// 		}

		if(laneId == 0) {
			if(rvalue[0][0] > value[0][warpId]) {
				value[0][warpId] = rvalue[0][0];
			}
			if(rvalue[1][0] > value[1][warpId]) {
				value[1][warpId] = rvalue[1][0];
			}
		}
	}

	if(laneId == 0) {
		if(conf[0][warpId] < lastval) {
			if(value[0][warpId] > f[conf[0][warpId]]) {
				f[conf[0][warpId]] = value[0][warpId];
			}
		}
		if(conf[1][warpId] < lastval) {
			if(value[1][warpId] > f[conf[1][warpId]]) {
				f[conf[1][warpId]] = value[1][warpId];
			}
		}
	}
	return;
}



#define COMBS(X) ((1 << cardinality(X)-1) - 1)

int run_test(unsigned int MAXVAL,dint items) {

	register unsigned int i,c1,count =0;
	unsigned short *dev_bids;

	count = 0;

	HANDLE_ERROR(hipDeviceReset());
	HANDLE_ERROR(hipSetDeviceFlags(hipDeviceScheduleYield));
  	HANDLE_ERROR(hipDeviceSetCacheConfig(hipFuncCachePreferL1));
//	HANDLE_ERROR(cudaDeviceSetLimit(cudaLimitMallocHeapSize,0));
	HANDLE_ERROR(hipDeviceSetLimit(hipLimitStackSize,0));
	HANDLE_ERROR(hipDeviceSetLimit(hipLimitMallocHeapSize,0));
	HANDLE_ERROR(hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte));

	register unsigned int streams = NSTREAMS;
	register unsigned int streamcount = 0;
	register hipStream_t stream[streams];
	for(i = 0;i < streams; i++)
		HANDLE_ERROR(hipStreamCreate(&stream[i]));

	count = 0;

 	HANDLE_ERROR(hipMalloc((void **)&dev_bids, MAXVAL*sizeof(short)));

 	HANDLE_ERROR(hipMemcpy(dev_bids,bids,MAXVAL*sizeof(short),hipMemcpyHostToDevice));

	register unsigned int bsize = MAXBLOCKSIZE;
	register int blocks;
	int prev =0;
//	lock_count = 0;
	time_t rstart,rend,rt;
	rstart=clock();
	for(i = 2; i <= items; i++) {
		time_t start,end,t;

		start=clock();
		unsigned int splittings;
		blocks =4;//(int)  ceil((threads/bsize));
		double threads;
		c1 = (1 << i) -1;
		unsigned int c2 = c1;
		unsigned int c3;
		unsigned int ca[blocks];
		unsigned int cacount;
		splittings =  COMBS(c1);///NPERBLOCK;
		threads = ((double) splittings)/ NPERBLOCK;
		threads = ceil(threads);
		
		for(; c1 <= MAXVAL;) {

			cacount = 1;
			ca[0] = c1;			
			for(int x = 0, cacount = 1; x < CONFPKERNEL*blocks;x++) {
				t = c2 | (c2-1);
				c2 = (t + 1) | (((~t & -~t) - 1) >> (__builtin_ctz(c2) + 1));
				if(x%CONFPKERNEL == 0 && x > 1) {
					ca[cacount] = c2;
					cacount++;
				}
			}
	
			switch((c2 > MAXVAL)) {
			case 1:
				subsetcomp33 < MAXBLOCKSIZE , 1> <<<blocks,MAXBLOCKSIZE,0,stream[streamcount]>>>(dev_bids,splittings,MAXVAL,i,ca[0],ca[1],ca[2],ca[3]);
				break;
			case 0:
				subsetcomp33 < MAXBLOCKSIZE , 0> <<<blocks,MAXBLOCKSIZE,0,stream[streamcount]>>>(dev_bids,splittings,MAXVAL,i,ca[0],ca[1],ca[2],ca[3]);
				break;

			}
			c1 = c2;


			streamcount++;
			count++;

			if(streamcount >= streams)
				streamcount = 0;
		}
		

		for (int t = 0; t < streams; ++t) {
			HANDLE_ERROR(hipStreamSynchronize(stream[t]));
		}

		HANDLE_ERROR(hipDeviceSynchronize());


		end=clock();
		t=(end-start)/(CLOCKS_PER_SEC/1000);
		printf("ended card %d blocks\t %d threads/block %u, n kernels %u \t time %lu \t splittings %d time per kernel %u\n",i,blocks,bsize,count-prev,t,splittings,t/(count-prev));
		prev =	count;

	}
	for (int i = 0; i < streams; ++i)
		hipStreamDestroy(stream[i]);

	HANDLE_ERROR(hipDeviceSynchronize());

	rend=clock();
	rt=(rend-rstart)/(CLOCKS_PER_SEC/1000);
	printf("real time %lu\n",rt);

	HANDLE_ERROR(hipMemcpy(f,dev_bids,MAXVAL*sizeof(short),hipMemcpyDeviceToHost));
//	HANDLE_ERROR(cudaMemcpy(O,dev_o,MAXVAL*sizeof(int),cudaMemcpyDeviceToHost));
	//int i;
	HANDLE_ERROR(hipFree(dev_bids));
//	HANDLE_ERROR(cudaFree(dev_o));
	// HANDLE_ERROR(cudaFree(dev_lock1));
	// HANDLE_ERROR(cudaFree(dev_lock2));

	HANDLE_ERROR(hipDeviceReset());
//	printfo(MAXVAL);
	return count;
}

dint max2(dint conf) {
     register dint card = cardinality(conf)/2;
     register dint combinations = (1 << cardinality(conf)-1)-1;
     register dint max = f[conf];
     register dint tmp = 0;
     register dint subset = 0;
     register const dint inverse = ~conf;
     register dint i;
     for(i = 1;i<=combinations; i++) {
	     subset = ((inverse+1)+subset)&conf;
	     tmp = f[setdiff(conf,subset)] + f[subset];
	     if(max == tmp) {
		     break;
		     //  return subset;
	     }

     }
     return subset;
}

int recur_parse_wopt(dint MAXVAL) {
	stack * root = (stack *) malloc(sizeof(stack));
	stack * sroot = NULL;
	stack * scurr = NULL;
		root->conf = (MAXVAL)-1;
	int count = 0;
	root->next = NULL;
	stack * curr = root;
	while(curr) {
		dint conf = curr->conf;
		if(conf == 0) {
			printf("EXIT FAILURE\n");
			return 1;
		}
		/*if something is wrong*/
		if(count > 40) {
			fprintf(stderr,"Something went wrong at line %d in %s\n",__LINE__,__FILE__);
			return 1;
		}
//		printf("curr %u\t\n",curr->conf);
		if(f[conf] != bids[conf]) {
			dint proper_subset = max2(conf);
			dint diff = setdiff(conf,proper_subset);
			curr->conf = proper_subset;
			stack * tmp = (stack *) malloc(sizeof(stack));
			//printf("diff %u\t conf %u\t O[diff] %u\t O[conf]\t f %u\n",diff,conf,O[diff],O[conf],f[conf]);
			tmp->conf = diff;
			tmp->next = curr;
			root = tmp;
			curr = root;
			count++;
			continue;
		}
		if(sroot == NULL) {
			sroot = curr;
			scurr = curr;
		} else {
			/*set next pointer to the next singelton*/
			scurr->next = curr;
		}

		/*set the current singleton to be the pointer*/
		scurr = curr;
		curr = curr->next;
		/*clear the pinter to avoid infinite loop if cross-referenced*/
		scurr->next = NULL;

	}
	printf("\n");
	curr = sroot;
	dint tmp = 0;
	while(curr != NULL) {
		if(bids[curr->conf]) {
			if(curr->conf == indexa) {
				printf("correct bid\n");
				tmp++;
				return 0;
			}
			//printf("conf %u value %u\n",curr->conf,bids[curr->conf]);

		}
		stack * tmp = curr;
		curr = curr->next;
		free(tmp);
	}
	if(tmp < 1) {
		printf("something is wrong, no bids\n");
		return 1;

	}
	printf("n = %u\n",tmp);
	return 0;
}

int parse_wopt(dint MAXVAL) {
	//printf("parse maxval = %u\n",MAXVAL);
	//wopt at start contain MAX at wopt[0] which is the combination that goes in bids[wopt[n]]
	stack * root =(stack *) malloc(sizeof(stack));
	stack * sroot = NULL;
	stack * scurr = NULL;
	//DO N	OT REMOVE -1
	root->conf = (MAXVAL)-1;
	int count = 0;
	root->next = NULL;
	stack * curr = root;
	while(curr) {
		dint conf = curr->conf;
		if(conf == 0) {
			printf("EXIT FAILURE\n");
			return 1;
		}
		/*if something is wrong*/
		if(count > 40) {
			fprintf(stderr,"Something went wrong at line %d in %s\n",__LINE__,__FILE__);
			return 1;
		}
		printf("curr %u\t\n",curr->conf);
		if(conf != O[conf]) {

			dint diff = setdiff(conf,O[conf]);
			curr->conf = O[conf];
			stack * tmp = (stack *) malloc(sizeof(stack));
			printf("diff %u\t conf %u\t O[diff] %u\t O[conf] %u\t f %u\n",diff,conf,O[diff],O[conf],f[conf]);
			tmp->conf = diff;
			tmp->next = curr;
			root = tmp;
			curr = root;
			count++;
			continue;
		}
		if(sroot == NULL) {
			sroot = curr;
			scurr = curr;
		} else {
			/*set next pointer to the next singelton*/
			scurr->next = curr;
		}

		/*set the current singleton to be the pointer*/
		scurr = curr;
		curr = curr->next;
		/*clear the pinter to avoid infinite loop if cross-referenced*/
		scurr->next = NULL;

	}
	printf("\n");
	curr = sroot;
	dint tmp = 0;
	while(curr != NULL) {
		if(bids[curr->conf]) {
			if(curr->conf == indexa) {
				printf("correct bid\n");
				tmp++;
				return 0;
			}
			printf("conf %u value %u\n",curr->conf,bids[curr->conf]);

		}
		stack * tmp = curr;
		curr = curr->next;
		free(tmp);
	}
	if(tmp < 1) {
		printf("something is wrong, no bids\n");

		return 1;

	}
	printf("n = %u\n",tmp);
	return 0;
}

int main(void) {
	/*Start n amount of assets*/
	dint from = NAGENTS;
	dint MAXVAL = (2 << (from-1));

	time_t start,end,t;
	O = (dint * ) malloc(sizeof(dint)*(2 << (from-1)));
	bids = (unsigned short * ) malloc(sizeof(short)*(2 << (from-1)));
	f = (unsigned short * ) malloc(sizeof(short)*(2 << (from-1)));
//	f = bids;
	int ret_val =0;
	int count;
	while(ret_val == 0) {

		MAXVAL = (2 << (from-1));
		gen_rand_bids(MAXVAL);
		set_singleton_bid(MAXVAL);
		printf("maxval %u from %u\n",MAXVAL,from);
		start=clock();//predefined  function in c
		count = run_test(MAXVAL,from);
		end=clock();
		t=(end-start)/CLOCKS_PER_SEC;
		ret_val= recur_parse_wopt(MAXVAL);// parse_wopt(MAXVAL);
		printf("\nTime taken =%lu for n= %u with count %d average per count %lf\n", (unsigned long) t,from,count,(double)t/count);
	}


	free(O);
	free(f);

	return 0;
}


// template<int blockSize,int nparallelconf,int confpkernel,int nperblock,int currblocksize>
// __global__ void
// __launch_bounds__(currblocksize,MIN_BLOCKS_PER_MP)
// 	subsetcomp32(
// 		/*0*/	unsigned short * __restrict__ f, /*Bid value*/
// 		/*1*/	unsigned int * __restrict__ O, /*The move array*/
// 		/*2*/	unsigned int * __restrict__ lock,
// 		/*5*/	unsigned int maxval,
// 		/*6*/	unsigned short count1,
// 		unsigned int conf1,
// 		unsigned int lastval,//the value a permutation can not exceed.
// 		unsigned int card)
// {
// 	//confpkernel = how many configurations the kernel will evaluate
// 	//nparallelconf = how many configurations the kernel will evaluate at the same time
// 	__shared__  unsigned short shared_value[(currblocksize >> 5)+1][nparallelconf];
// 	__shared__  unsigned int shared_conf[(currblocksize >> 5)+1][nparallelconf];
// 	__shared__  unsigned int conf[confpkernel];// the configurations needed for the whole execution
// 	__shared__  unsigned short shift [confpkernel][NAGENTS];// the shift matrix/array
// 	//__shared__ volatile unsigned int tmp [confpkernel][2];
//  	register unsigned int subset_value[2][8];//the value for one of the subset sums
// 	register unsigned int subset_conf[2][8];

// 	register unsigned int count = count1;
// 	register unsigned int const tid = threadIdx.x;
// //	__shared__ unsigned int
// 	register unsigned int ispec = nperblock*(threadIdx.x + blockDim.x * blockIdx.x);//I + offset;
// 	register int x,i,z; //counter

// #if (TIMING == 1)
// 	clock_t stop_time, total,start_time;
// 	total = 0;
// 	start_time = clock();
// #endif

// 	if(tid == 0) {
// 		conf[0] = conf1;
// #pragma unroll
// 		for(x = 1;x < confpkernel; x++) { // generate the configurations
// 			conf[x] = conf[x-1];
// 			z = conf[x] | (conf[x]-1);
// 			conf[x] = (z + 1) | (((~z & -~z) - 1) >> (__ffs(conf[x])));
// 		}
// 		CHECKPOINT("l1 %d\n");
// 	}
// 	if(confpkernel > 32) {
// 		__syncthreads();
// 	}

// 	count = count1;
// 	if(tid < confpkernel) { //generate the shift arrays
// 		subset_conf[0][0] = conf[tid]; // re-use registers tmpval
// 		subset_conf[0][1] = 0;// re-use registers index
// #pragma unroll
// 		for(x = 0,i=0; x < card;x++) {//could put card in template to unroll
// 			subset_conf[0][1] = __ffs(subset_conf[0][0]) - 1; //find which index is first bit
// 			subset_conf[0][0] &= ~(1 << subset_conf[0][1]);//set nth bit to 0
// 			shift[tid][x] = subset_conf[0][1];
// 		}
// 	}

// 	CHECKPOINT("l2 %d\n");

// 	__syncthreads();

// #pragma unroll
// 	for(x =0; x < confpkernel; x += nparallelconf) {

// 		if(conf[x] > lastval) {//if the permutation is larger than the full set, c> (1 << NAGENTS)
// 			continue;
// 		}
// //#pragma unroll
// //		for(i = 0; i < nperblock; i++) {
// #pragma unroll
//  		for(z=0; z < nparallelconf;z++) {
//  			subset_value[0][z] = subset_value[1][z] = 0U;
//  		}
// //		}
// 		CHECKPOINT("l3 %d\n");
// 		if(ispec >= maxval) {
// 			goto postfetch;
// 		}
// 			//This for loop initilize the first subset configuration.

// #pragma unroll
// 		for(i = 0; i < nparallelconf; i++) {
// 			unsigned int tmp = ispec;
// 			//unsigned int const tcar = __popc(tmp);
// 			subset_conf[0][i] = 0;
// 			while(tmp) {
// 				unsigned short index = __ffs(tmp)-1;
// 				subset_conf[0][i] += (1 << shift[x+i][index]);//CHECK
// 				tmp &= ~(1 << index);
// 			}
// 		}
// 		CHECKPOINT("l4 %d\n");
// #pragma unroll
// 		for(z=0;z < nparallelconf;z++) {
// 			if(conf[z+x] > lastval) {
// 				continue;
// 			}
// 			subset_conf[0][z] = SUBSET(conf[z+x],subset_conf[0][z]);
// 			subset_value[0][z] = f[(setdiff(conf[z+x],subset_conf[0][z]))] + f[subset_conf[0][z]];
// 			//ispec++;
// 			if((ispec+1) >= maxval) {
// 				continue;
// 			}
// 			subset_conf[1][z] = SUBSET(conf[z+x],subset_conf[0][z]);
// 			subset_value[1][z] = f[(setdiff(conf[z+x],subset_conf[1][z]))] + f[subset_conf[1][z]];
// 		}
// 		CHECKPOINT("l6 %d\n");

// 	postfetch:

// #pragma unroll
// 		for(z = 0; z < nparallelconf;z++) {//warp reduction
// 			if(subset_value[1][z] > subset_value[0][z]) {
// 				subset_value[0][z] = subset_value[1][z];
// 				subset_conf[0][z] = subset_conf[1][z];
// 			}
// #pragma unroll
// 			for(i = 16;i >=1;i >>=1) {
// 				int warp_value = __shfl_xor((int)subset_value[0][z],i,32);
// 				int warp_conf = __shfl_xor((int)subset_conf[0][z],i,32);
// 				if(warp_value > subset_value[0][z]) {
// 					subset_value[0][z] =(unsigned int) warp_value;
// 					subset_conf[0][z] =(unsigned int) warp_conf;
// 				}
// 			}
// 			//tid&(WARPSIZE-1) == tid%WARPSIZE
// 			//Only threads with line id == 0 is allowed to update in the shared memory,
// 			//i.e. the first thread in each warp
// 			if(!(tid&(31))) {
// 				unsigned int index = tid >> 5; // tid >> 5 == tid / 32 which warp it is
// 				shared_value[index][z] = subset_value[0][z];
// 				shared_conf[index][z] = subset_conf[0][z];
// 			}

// 		}
// 		CHECKPOINT("l7 %d\n");

// 		//	CHECKPOINT("l8 %d\n");
// 		if((currblocksize/32) > 1) {
//  		__syncthreads();
// 		}
// 		//how many warps is it, block dimension divided by warp size
// 		//e.g. 256/32 == 256 >> 5
// 		if((currblocksize/32) > 1) {//evaluated by the pre-processor
// 			i = (currblocksize >> 6);//blockDim.x >> 6;
// 			if(tid<i) {//reduction mby move down if you get wrong results gained ~1000 cycles

// #pragma unroll
// 			for(; i > 0; i >>= 1) {
// #pragma unroll
//   					for(z=0; z < nparallelconf;z +=4) {
// 						COMP(z);
//   					}
// 			}
// 			}
// 		}

// 		CHECKPOINT("l9 %d\n");
// 		if((currblocksize) >= blockSize) {//evaluated by the pre-processor
// 			//__syncthreads();
// 			if(tid == 0) {
// #pragma unroll
// 				for(z=0; z < nparallelconf;z++) {
// 					if(f[conf[z+x]] < shared_value[0][z]) {
// 						//	printf("lock val %u shared_val %s\n",lock[count+z] ,shared_value[0][z]);
// 						if(atomicMax(&(lock[count+z]),shared_value[0][z]) < shared_value[0][z]) {
// 							//	O[conf[z+x]] = shared_conf[0][z];
// 							f[conf[z+x]] = shared_value[0][z];
// 						}
// 					}
// 				}
// 			}
// 		} else {
// #pragma unroll
// 			for(z=0; z < nparallelconf;z++) {
// 				if(f[conf[z+x]] < shared_value[0][z]) {
// 					//	O[conf[z+x]] = shared_conf[0][z];
// 					f[conf[z+x]] = shared_value[0][z];

// 				}

// 			}
// 		}

// 		count += nparallelconf;
// 		ispec = NPERBLOCK*(threadIdx.x + blockDim.x * blockIdx.x);//I + offset;

// 	}
// }
// template<int blockSize,int nparallelconf,int confpkernel,int nperblock,int currblocksize>
// __global__ void
// __launch_bounds__(currblocksize,MIN_BLOCKS_PER_MP)
// 	subsetcomp35(
// 				/*0*/	unsigned short * __restrict__ f, /*Bid value*/
// 		/*1*/	unsigned int * __restrict__ O, /*The move array*/
// 		/*2*/	unsigned int * __restrict__ lock,
// 		/*5*/	unsigned int maxval,
// 		/*6*/	unsigned short count1,
// 		unsigned int conf1,
// 		unsigned int lastval,//the value a permutation can not exceed.
// 		unsigned int card)
// {
// 	//confpkernel = how many configurations the kernel will evaluate
// 	//nparallelconf = how many configurations the kernel will evaluate at the same time
// 	__shared__  unsigned short shared_value[(currblocksize >> 5)+1][nparallelconf];
// 	__shared__  unsigned int conf[confpkernel];// the configurations needed for the whole execution
// //	__shared__ unsigned short old_values[]
// 	__shared__  unsigned char shift [confpkernel][NAGENTS];// the shift matrix/array
// 	//__shared__ volatile unsigned int tmp [confpkernel][2];
//  	register unsigned int subset_value[nperblock][nparallelconf];//the value for one of the subset sums
// 	register unsigned int subset_conf[2];//[nparallelconf];
// //	register unsigned int count = count1;
// 	register unsigned int const tid = threadIdx.x;
// 	register unsigned int const ispec = nperblock*(threadIdx.x + blockDim.x * blockIdx.x);//I + offset;
// 	register int x,i,z; //counter

// #if (TIMING == 1)
// 	clock_t stop_time, total,start_time;
// 	total = 0;
// 	start_time = clock();
// #endif

// 	if(tid == 0) {

// 		subset_conf[0] = conf[0] = conf1;
// #pragma unroll
// 		for(x = 1;x < confpkernel; x++) { // generate the configurations
// 			//conf[x] = conf[x-1];
// 			z = subset_conf[0] | (subset_conf[0]-1);
// 			subset_conf[0] = (z + 1) | (((~z & -~z) - 1) >> (__ffs(subset_conf[0])));
// 			conf[x]= subset_conf[0];
// 		}
// 		CHECKPOINT("l1\t %d\n");
// 	}
// //	__syncthreads();

// #if (TIMING == 1)
// 	start_time = clock();
// #endif
// 	/* Generates the shift array in order in the future to create subsets.
// 	 *
// 	 */

// 	if(tid < confpkernel) { //generate the shift arrays

// 		subset_conf[0] = conf[tid]; // re-use registers tmpval
// 		subset_conf[1] = 0;// re-use registers index
// #pragma unroll
// 		for(x = 0,i=0; x < card;x++) {//could put card in template to unroll
// 			subset_conf[1] = __ffs(subset_conf[0]) - 1; //find which index is first bit
// 			subset_conf[0] &= ~(1 << subset_conf[1]);//set nth bit to 0
// 			shift[tid][x] = subset_conf[1];
// 		}

// 	}

// 	CHECKPOINT("l2\t %d\n");

// 	__syncthreads();


// #pragma unroll
// 	for(x =0; x < confpkernel; x += nparallelconf) {

// #if (TIMING == 1)
// 		start_time = clock();
// #endif
// 		if(conf[x] > lastval) {//if the permutation is larger than the full set, c> (1 << NAGENTS)
// 			continue;
// 		}

// #pragma unroll
//  		for(z=0; z < nparallelconf;z++) {
//  			subset_value[0][z] = subset_value[1][z] = 0U;
//  		}
// 		CHECKPOINT("l3\t %d\n");
// 		if(ispec >= maxval) {
// 			goto postfetch;
// 		}

// 		CHECKPOINT("l3\t %d\n");
// 		unsigned int tmp[2];
// 		tmp[1]= tmp[0] = 0;
// #pragma unroll
// 		for(z=0;z < nparallelconf;z +=2) {

// 			if(conf[z+x] >= lastval) {
// 				continue;
// 			}
// 			unsigned int tmpstore = 0, tmpstore1 = 0;
// 			unsigned int index;
// 			//unsigned int const tcar = __popc(tmp);
// 			unsigned int wshile;
// 			unsigned int icard;

// 			if(conf[z+x+1] < lastval) {
// 				wshile = ispec;
// 				subset_conf[0] = 0;
// 				subset_conf[1] = 0;
// 				while(wshile) {
// 					index = __ffs(wshile)-1;
// 					subset_conf[1] += (1 << shift[x+z+1][index]);
// 					subset_conf[0] += (1 << shift[x+z][index]);//CHECK
// 					wshile &= ~(1 << index);
// 				}
// 				subset_conf[1] = SUBSET(conf[z+x+1],subset_conf[1]);
// 				// icard = __popc(subset_conf[1]);
// 				// if(icard >= (NAGENTS-card)) {
// 				subset_value[0][z+1] = f[subset_conf[1]];
// 				// } else {
// 				// 	subset_value[0][z+1] = 0;
// 				// }
// 			} else {
// 				wshile = ispec;
// 				subset_conf[0] = 0;
// 				while(wshile) {
// 					index = __ffs(wshile)-1;
// 					subset_conf[0] += (1 << shift[x+z][index]);//CHECK
// 					wshile &= ~(1 << index);
// 				}
// 			}
// 			subset_conf[0] = SUBSET(conf[z+x],subset_conf[0]);
// 			// icard = __popc(subset_conf[0]);
// 			// if(icard > (NAGENTS-card)) {
// 			subset_value[0][z] = f[subset_conf[0]];
// 			// } else {
// 			//subset_value[0][z] = 0;
// 			// }
// 			if((ispec+1) < maxval) {
// 				tmp[0] = SUBSET(conf[z+x],subset_conf[0]);
// 				// if(tmp[0] == subset_conf[1]) {\\expensive
// 				// 	subset_value[1][z] = subset_value[0][z];
// 				// } else
// 				if(tmp[0] != tmp[1]) {//dont remove
// 					subset_value[1][z] = f[tmp[0]];
// 				}

// 				if(conf[z+x+1] < lastval) {
// 					tmp[1] = SUBSET(conf[z+x+1],subset_conf[1]);
// 					if(tmp[1] == tmp[0]) {// Do not remove, cost 1 second
// 						subset_value[1][z+1] = subset_value[1][z];
// 					// } else//  if(tmp[1] == subset_conf[0]) {
// 					// 	subset_value[1][z+1] = subset_value[0][z];
// 					} else {
// 						subset_value[1][z+1] = f[tmp[1]];
// 					}
// 				}
// 			}


// 				//	}

// 			subset_conf[0] = (setdiff(conf[z+x],subset_conf[0]));
// 			tmpstore = f[subset_conf[0]];//

// 			//next splitting
// 			if((ispec+1) < maxval) {
// 				tmp[0] = (setdiff(conf[z+x],tmp[0]));
// 				tmpstore1 = f[tmp[0]];
// 			}

// 			//next configuration
// 			if(conf[z+x+1] < lastval) {
// 				subset_conf[1] = (setdiff(conf[z+x+1],subset_conf[1]));
// 				if(z < (nparallelconf-2)){
// 				 	subset_value[1][z+2] = subset_value[1][z+1];
// 				}
// 				if(subset_conf[1] == subset_conf[0]){// dont remove
// 					subset_value[0][z+1] += tmpstore;
// 					//} // else if(subset_conf[1] == tmp[0]){ // expensive
// 				// 	subset_value[0][z+1] += tmpstore1;
// 				} else{
// 					subset_value[0][z+1] += f[subset_conf[1]];
// 				}
// 				//next splitting
// 				if((ispec+1) < maxval) {
// 					tmp[1] = (setdiff(conf[z+x+1],tmp[1]));
// 					// if(tmp[1] == tmp[0]){// expensive
// 					// 	subset_value[1][z+1] += tmpstore1;
// 					// }else
// 					subset_value[1][z+1] += f[tmp[1]];
// 				}
// 			}
// 			subset_value[0][z] += tmpstore;//f[(setdiff(conf[z+x],subset_conf[0]))];//
// 			subset_value[1][z] += tmpstore1;
// 			tmp[1] = (setdiff(conf[z+x+1],tmp[1]));
// 			//	subset_conf[1] = (setdiff(conf[z+x+1],subset_conf[1]));


// 		}
// 		CHECKPOINT("l4\t %d\n");

// 	postfetch:

// #pragma unroll
// 		for(z = 0; z < nparallelconf;z +=2) {//warp reduction
// 			if(conf[z+x] >= lastval) {
// 				continue;
// 			}
// 			if(subset_value[1][z] > subset_value[0][z]) {
// 				subset_value[0][z] = subset_value[1][z];
// 				//subset_conf[0][z] = subset_conf[1][z];
// 			}
// 			if(subset_value[1][z+1] > subset_value[0][z+1]) {
// 				subset_value[0][z+1] = subset_value[1][z+1];
// 				//subset_conf[0][z] = subset_conf[1][z];
// 			}
// #pragma unroll
// 			for(i = 16;i >=1;i >>=1) {
// 				subset_value[1][z] = __shfl_xor((int)subset_value[0][z],i,32);
// 				subset_value[1][z+1] = __shfl_xor((int)subset_value[0][z+1],i,32);
// 				//int warp_conf = __shfl_xor((int)subset_conf[0][z],i,32);
// 				if(subset_value[1][z] > subset_value[0][z]) {
// 					subset_value[0][z] = subset_value[1][z];// (unsigned int) warp_value;
// 					//	subset_conf[0][z] =(unsigned int) warp_conf;
// 				}
// 				if(subset_value[1][z+1] > subset_value[0][z+1]) {
// 					subset_value[0][z+1] = subset_value[1][z+1];
// 				//subset_conf[0][z] = subset_conf[1][z];
// 				}
// 			}
// 			//tid&(WARPSIZE-1) == tid%WARPSIZE
// 			//Only threads with line id == 0 is allowed to update in the shared memory,
// 			//i.e. the first thread in each warp
// 			if(!(tid&(31))) {
// 				unsigned int index = tid >> 5; // tid >> 5 == tid / 32 which warp it is
// 				shared_value[index][z] = subset_value[0][z];
// 				shared_value[index][z+1] = subset_value[0][z+1];
// 				//shared_conf[index][z] = subset_conf[0][z];
// 			}

// 		}
// 		/*WORKING STOP DELETE*/
// 		CHECKPOINT("l5\t %d\n");

// 		//	CHECKPOINT("l8\t %d\n");
// 		if((currblocksize/32) > 1) {
//  		__syncthreads();
// 		}
// 		//how many warps is it, block dimension divided by warp size
// 		//e.g. 256/32 == 256 >> 5
// //		if((currblocksize/32) > 1) {//evaluated by the pre-processor

//  			i = blockDim.x >> 6;
//  			if(tid<i) {//reduction mby move down if you get wrong results gained ~1000 cycles
//  #pragma unroll
//  				for(; i > 0; i >>= 1) {
// //#pragma unroll
// 						for(z=0; z < nparallelconf;z ++) {

// // 						subset_value[0][z] = shared_value[tid][z];
// // //#pragma unroll
// // 						for(i = (currblocksize/32);i >=1;i >>=1) {
// // 							int warp_value = __shfl_down((int)subset_value[0][z],1,32);
// // 							//int warp_conf = __shfl_xor((int)subset_conf[0][z],i,32);
// // 							if(warp_value > subset_value[0][z]) {
// // 								subset_value[0][z] =(unsigned int) warp_value;
// // 								//	subset_conf[0][z] =(unsigned int) warp_conf;
// // 							}
// // 						}
// // 						if(tid == 0) {
// // //							unsigned int index = tid >> 5; // tid >> 5 == tid / 32 which warp it is
// // 							shared_value[0][z] = subset_value[0][z];
// // 							//shared_conf[index][z] = subset_conf[0][z];
// // 						}
// 						COMP(z);
// 						// COMP(0+1);
// 						// COMP(0+2);
// 						// COMP(0+3);

// 							}
// // 				}
//  			}
// 		}

// 		CHECKPOINT("l6\t %d\n");
// 		if((currblocksize) >= blockSize) {//evaluated by the pre-processor
// 			//__syncthreads();
// 			if(tid == 0) {
// #pragma unroll
// 				for(z=0; z < nparallelconf;z++) {
// 					if(conf[z+x] > lastval) {continue;}
// 					if(f[conf[z+x]] < shared_value[0][z]) {
// 						//	printf("lock val %u shared_val %s\n",lock[count+z] ,shared_value[0][z]);
// 						if(atomicMax(&(lock[count1+x+z]),shared_value[0][z]) < shared_value[0][z]) {
// 							//O[conf[z+x]] = shared_conf[0][z];
// 							if(f[conf[z+x]] < shared_value[0][z]) {
// 								f[conf[z+x]] = shared_value[0][z];
// 							}
// 						}
// 					}
// 				}
// 			}
// 		} else {
// #pragma unroll
// 			for(z=0; z < nparallelconf;z++) {
// 				if(conf[z+x] > lastval) {continue;}
// 				if(f[conf[z+x]] < shared_value[0][z]) {
// 					//O[conf[z+x]] = shared_conf[0][z];
// 					f[conf[z+x]] = shared_value[0][z];

// 				}

// 			}
// 		}
// 		CHECKPOINT("l7\t %d\n");
// //		count += nparallelconf;
// 	}
// }
