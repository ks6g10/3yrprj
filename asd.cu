#include "hip/hip_runtime.h"
#include <stdio.h> 
#include "asd.h"
#include <string.h> // for ffs
#include <stdint.h>
#include <math.h>
#include <time.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

/*Debug enabled gives more print statements of bids and how the "Matrix" gets evaluated*/
#define DEBUG 0
#define TRUE 1 
#define FALSE 0
/*Test sets all bids to one, which should give you n=|ITEMS| bids on output*/
#define TEST 1
/*Defines from 0-Range the random will give out*/
#define RANGE 10000
#define ITEMS 25

#define MAX (2 << (ITEMS-1))
#if ITEMS < 8
#define dint uint8_t  
#elif ITEMS < 16
#undef dint
#define dint uint16_t
#elif ITEMS < 32
#undef dint
#define dint uint32_t
#endif

#define SUBSET(X)((~_conf+(X+1))&_conf)
#define SETSUM(X)(f[setdiff(_conf,X)]+f[X])


static void HandleError( hipError_t err, const char *file,int line ) {
	if (err != hipSuccess) {
		printf( "%s in %s at line %d\n", hipGetErrorString( err ),file, line );
		exit( EXIT_FAILURE );
	}
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


unsigned int pascal[30][30] =  
{{1,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0},
 {1,1,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0},
 {1,2,1,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0},
 {1,3,3,1,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0},
 {1,4,6,4,1,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0},
 {1,5,10,10,5,1,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0},
 {1,6,15,20,15,6,1,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0},
 {1,7,21,35,35,21,7,1,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0},
 {1,8,28,56,70,56,28,8,1,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0},
 {1,9,36,84,126,126,84,36,9,1,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0},
 {1,10,45,120,210,252,210,120,45,10,1,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0},
 {1,11,55,165,330,462,462,330,165,55,11,1,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0},
 {1,12,66,220,495,792,924,792,495,220,66,12,1,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0},
 {1,13,78,286,715,1287,1716,1716,1287,715,286,78,13,1,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0},
 {1,14,91,364,1001,2002,3003,3432,3003,2002,1001,364,91,14,1,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0},
 {1,15,105,455,1365,3003,5005,6435,6435,5005,3003,1365,455,105,15,1,0,0,0,0,0,0,0,0,0,0,0,0,0,0},
 {1,16,120,560,1820,4368,8008,11440,12870,11440,8008,4368,1820,560,120,16,1,0,0,0,0,0,0,0,0,0,0,0,0,0},
 {1,17,136,680,2380,6188,12376,19448,24310,24310,19448,12376,6188,2380,680,136,17,1,0,0,0,0,0,0,0,0,0,0,0,0},
 {1,18,153,816,3060,8568,18564,31824,43758,48620,43758,31824,18564,8568,3060,816,153,18,1,0,0,0,0,0,0,0,0,0,0,0},
 {1,19,171,969,3876,11628,27132,50388,75582,92378,92378,75582,50388,27132,11628,3876,969,171,19,1,0,0,0,0,0,0,0,0,0,0},
 {1,20,190,1140,4845,15504,38760,77520,125970,167960,184756,167960,125970,77520,38760,15504,4845,1140,190,20,1,0,0,0,0,0,0,0,0,0},
 {1,21,210,1330,5985,20349,54264,116280,203490,293930,352716,352716,293930,203490,116280,54264,20349,5985,1330,210,21,1,0,0,0,0,0,0,0,0},
 {1,22,231,1540,7315,26334,74613,170544,319770,497420,646646,705432,646646,497420,319770,170544,74613,26334,7315,1540,231,22,1,0,0,0,0,0,0,0},
 {1,23,253,1771,8855,33649,100947,245157,490314,817190,1144066,1352078,1352078,1144066,817190,490314,245157,100947,33649,8855,1771,253,23,1,0,0,0,0,0,0},
 {1,24,276,2024,10626,42504,134596,346104,735471,1307504,1961256,2496144,2704156,2496144,1961256,1307504,735471,346104,134596,42504,10626,2024,276,24,1,0,0,0,0,0},
 {1,25,300,2300,12650,53130,177100,480700,1081575,2042975,3268760,4457400,5200300,5200300,4457400,3268760,2042975,1081575,480700,177100,53130,12650,2300,300,25,1,0,0,0,0},
 {1,26,325,2600,14950,65780,230230,657800,1562275,3124550,5311735,7726160,9657700,10400600,9657700,7726160,5311735,3124550,1562275,657800,230230,65780,14950,2600,325,26,1,0,0,0},
 {1,27,351,2925,17550,80730,296010,888030,2220075,4686825,8436285,13037895,17383860,20058300,20058300,17383860,13037895,8436285,4686825,2220075,888030,296010,80730,17550,2925,351,27,1,0,0},
 {1,28,378,3276,20475,98280,376740,1184040,3108105,6906900,13123110,21474180,30421755,37442160,40116600,37442160,30421755,21474180,13123110,6906900,3108105,1184040,376740,98280,20475,3276,378,28,1,0},
 {1,29,406,3654,23751,118755,475020,1560780,4292145,10015005,20030010,34597290,51895935,67863915,77558760,77558760,67863915,51895935,34597290,20030010,10015005,4292145,1560780,475020,118755,23751,3654,406,29,1}};


/*		           0 1 2 3 4 5 6 7 8			*/
uint32_t * bids,* f, * O;
// dint bids[MAX] ={0,2,3,6,4,7,6,9}; //conf 5 and 2
//dint bids[MAX] =  {0,2,3,6,4,6,6,9}; //conf 3 and 4
// dint bids[MAX] =  {0,20,3,6,4,6,10,9}; //conf 1 and 6
 
struct _stack {
	dint conf;
	struct _stack * next;
} typedef stack;



#define setdiff(seta,setb) (seta & ~setb)

inline  dint cardinality( dint seta) {
	return __builtin_popcount(seta);
}
int indexa =0;
void gen_rand_bids(dint MAXVAL) {
	register dint i = 0;
#if TEST
	for(i = 1; i < MAXVAL;i++) {
		bids[i] = 1;
		O[i] = i;
	}
//     unsigned int seed = (unsigned)time ( NULL );
	//   srand(seed);
	indexa++;
	//indexa = rand() % MAXVAL;
     bids[indexa] = 100;
     printf("index %d \n",indexa);
     if(indexa >= MAXVAL) {
	     printf("No error\n");
	     exit(0);
     }
	     
//	bids[1] =0;
//	bids[2] = 0;
//	bids[32769] = 20;
#else
	for(i = 1; i < MAXVAL;i++) {
		bids[i] = rand() % RANGE;
		O[i] = i;
	}
	for(i = 1; i < MAXVAL;i*=2) {
		bids[i] = rand() % RANGE;

	}
#endif

}

const char *btb(dint y)
{
	int x = y;
	static char b[9];
	b[0] = '\0';
	
	int z;
	for (z = 128; z > 0; z >>= 1)
	{
		strcat(b, ((x & z) == z) ? "1" : "0");
	}	
	return b;
}

/*Sets all bids with one element in it, |n| = 1*/
inline void set_singleton_bid(dint MAXVAL) {
	register  dint i;
	for(i =1; i< MAXVAL; i*=2) {
		f[i] = bids[i];
		if(bids[i] > 0)
			O[i] = i;
	}
}



void printfo(dint MAXVAL) {
	int i;
	printf("\n");
	for(i = 1; i< MAXVAL; i++)
	{
		printf("Bid[%d]\t%u\tF[%d]\t%u\tO[%d]\t%u\tbin\t%s\n",i,bids[i],i,f[i],i,O[i],btb(i));

	}
}

int parse_wopt(dint MAXVAL) {
	//printf("parse maxval = %u\n",MAXVAL);
	//wopt at start contain MAX at wopt[0] which is the combination that goes in bids[wopt[n]]
	stack * root =(stack *) malloc(sizeof(stack));
	stack * sroot = NULL;
	stack * scurr = NULL;
	//DO N	OT REMOVE -1
	root->conf = (MAXVAL)-1;
	int count = 0;
	root->next = NULL;
	stack * curr = root;
	while(curr) {
		dint conf = curr->conf;
		if(conf == 0) {
			printf("EXIT FAILURE\n");
			return 1;
		}
		/*if something is wrong*/
			if(count > 40) {
				fprintf(stderr,"Something went wrong at line %d in %s\n",__LINE__,__FILE__);
				return 1;
			}
		printf("curr %u\t\n",curr->conf);
		if(conf != O[conf]) {

			dint diff = setdiff(conf,O[conf]);
			curr->conf = O[conf];
			stack * tmp = (stack *) malloc(sizeof(stack));
			printf("diff %u\t conf %u\t O[diff] %u\t O[conf] %u\t f %u\n",diff,conf,O[diff],O[conf],f[conf]);
			tmp->conf = diff;
			tmp->next = curr;
			root = tmp;
			curr = root;
			count++;
			continue;
		}
		if(sroot == NULL) {
			sroot = curr;
			scurr = curr;
		} else {
			/*set next pointer to the next singelton*/
			scurr->next = curr;
		}

		/*set the current singleton to be the pointer*/
		scurr = curr;
		curr = curr->next;
		/*clear the pinter to avoid infinite loop if cross-referenced*/
		scurr->next = NULL;

	}
	printf("\n");
	curr = sroot;
	dint tmp = 0;
	while(curr != NULL) {
		if(bids[curr->conf]) {
			if(curr->conf == indexa) {
				printf("correct bid\n");
				tmp++;
				return 0;
			}
			printf("conf %u value %u\n",curr->conf,bids[curr->conf]);
			
		}
		stack * tmp = curr;
		curr = curr->next;
		free(tmp);
	}
	if(tmp < 1) {
		printf("something is wrong, no bids\n");
			       
		return 1;

	}
	printf("n = %u\n",tmp);
	return 0;
}

#define I (threadIdx.x + blockDim.x * blockIdx.x)
#define SET_TEST_FETCH(STEP,S1,S2) {					\
									\
		STEP = SUBSET(ispec);					\
		if( ispec < maxval) {					\
			S1 = f[(setdiff(_conf,STEP))];			\
			S2 = f[(STEP)];					\
		} else {						\
			S1 = S2 = 0U;					\
		}							\
		ispec++;						\
	}

/* ispec += blockDim.x; change back if not working */

#define COMP_SET(V1,S1,V2,S2) {			\
		if(V1>V2) {			\
			V2 = V1;		\
			S2 = S1;		\
		}				\
	}
  
    
#define MAXBLOCKSIZE 256U
#define NAGENTS 21  
#define NSTREAMS 8 
#define NPERBLOCK 8
#define HALFBLOCK 8

#define PASCALSIZE 30

static __constant__ unsigned int pascl[PASCALSIZE][PASCALSIZE];

static __constant__ unsigned int cardindex[NAGENTS+1];

__device__ unsigned int get_index(unsigned int set) {
	const unsigned int card = __popc(set);
	const unsigned int cardi = cardindex[card];
	unsigned int sum = 0;
	unsigned int tmp = set;
	int i;
#pragma unroll
	for(i = 1;i<= card; i++) {
		unsigned int fsb = __ffs(tmp) -1;
		sum += pascl[fsb][i];
		tmp &= ~(1 << fsb);
	}
	sum += cardi;
	return sum;
}

__global__ void subsetcomp22(
	/*0*/	uint32_t * __restrict__ f, /*Bid value*/ 
	/*1*/	unsigned int * __restrict__ O, /*The move array*/
	/*2*/	unsigned int * __restrict__ lock,
	/*3*/	unsigned int _conf, /*The configuration*/
	/*5*/	unsigned int maxval,
	/*6*/	unsigned int count,
	/*8*/	unsigned int defbid)
{
/*these arrays are shared between all threads in the same block */
	__shared__ unsigned int share[MAXBLOCKSIZE];
	__shared__ unsigned int step[MAXBLOCKSIZE];     
	unsigned int ispec = NPERBLOCK*(threadIdx.x + blockDim.x * blockIdx.x);//I + offset;
	const unsigned int tid = threadIdx.x;
	int i;  
	unsigned int max = 0;
	unsigned int rstep = 0;
	unsigned int val1[NPERBLOCK];//the value for one of the subset sums
	unsigned int val2[NPERBLOCK];//the value for the other subset sums
	unsigned int stept[NPERBLOCK]; // the step array
	if(ispec < maxval) {

/*Local for the thread, check all its bid and pick the biggest*/
#pragma unroll 8
		for(i = 0; i < NPERBLOCK; i++) {
			SET_TEST_FETCH(stept[i],val1[i],val2[i]);	
		}
#pragma unroll 8
		for(i = 0; i < NPERBLOCK; i++) {		
			val1[i] += val2[i];
			COMP_SET(val1[i],stept[i],max,rstep);			
		}

	}
	step[threadIdx.x] = rstep;
	share[threadIdx.x] = max;
	i= blockDim.x >> 1;
	__syncthreads();
/*do max reduction on the shared array for all threads inside the block*/
#pragma unroll
	for (; i>0; i>>=1) {
		if (tid < i /* && (ispec <= maxval) */) {
			if(share[tid] <= share[tid + i]) {
				step[tid] = step[tid+i];
				share[tid] = share[tid+i];
			}
		}
		__syncthreads();
	}

/*thread 0 will attempt to set to global memory the agreed maximum value inside the block,
* if it is greater than the original bid and the bid in the lock array
*/

	if(tid == 0U) {
		i = share[0U];
		if(i == 0)
			return;
		if(defbid >= i)
			return;
		if(lock[count] < i) {
			if(atomicMax(&(lock[count]),i) < i) {
				O[_conf] = step[0U];
				f[_conf] = i; 
				__threadfence();
				return;
			} 
		}
	} else {
		return;
	}
}

int gen_copy_base_index() {
	unsigned int *tmpa =(unsigned int *) malloc(sizeof(unsigned int)*(NAGENTS+1));
	if(tmpa == NULL) {
		fprintf(stderr,"Can not allocate memory at line %s in %s\n",__LINE__,__FILE__);
		exit(1);
	}
	int i;
	int tmp = 0;
	for(i = 1; i <= NAGENTS;i++) {
		tmp += pascal[NAGENTS][i-1];
		tmpa[i] = tmp;
		//printf("tmp = %d for i %d\n",tmp,i);
	       
	}
	HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cardindex), tmpa, sizeof(unsigned int)*(NAGENTS+1), 0, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(pascl), pascal, sizeof(unsigned int)*(PASCALSIZE*PASCALSIZE), 0, hipMemcpyHostToDevice));
	return 0;
}

#define COMBS(X) ((1 << cardinality(X)-1) - 1)

int run_test(dint MAXVAL,dint items) {
/*Setup the environment*/
	//dint perm[MAXVAL];

	register unsigned int i, c,count =0;
 	unsigned int *dev_f,*dev_o;

	i = items/2;
	count = 0;

	HANDLE_ERROR(hipDeviceReset());
//  	hipDeviceSetCacheConfig( hipFuncCachePreferL1 );

	unsigned int * dev_lock1,*dev_lock2,*dev_ptr;
	const	unsigned int devcount = 1024;// count;
	register unsigned int streams = NSTREAMS;
	register unsigned int lock_count = 0;
	register unsigned int streamcount = 0;
	register hipStream_t stream[streams];
	for(int i = 0;i < streams; i++)
		HANDLE_ERROR(hipStreamCreate(&stream[i]));
//	printf("count %u\n",devcount);
	count = 0;
	HANDLE_ERROR(hipMalloc((void **)&dev_lock1,(10+devcount)*sizeof(int)));
	HANDLE_ERROR(hipMalloc((void **)&dev_lock2,(10+devcount)*sizeof(int)));
 	HANDLE_ERROR(hipMalloc((void **)&dev_f, MAXVAL*sizeof(int)));
 	HANDLE_ERROR(hipMalloc((void **)&dev_o, MAXVAL*sizeof(int)));

 	HANDLE_ERROR(hipMemcpy(dev_f,bids,MAXVAL*sizeof(int),hipMemcpyHostToDevice));
 	HANDLE_ERROR(hipMemcpy(dev_o,O,MAXVAL*sizeof(int),hipMemcpyHostToDevice));

	HANDLE_ERROR(hipMemset(dev_lock1,0,devcount*sizeof(int)));
	HANDLE_ERROR(hipMemset(dev_lock2,0,devcount*sizeof(int)));
	gen_copy_base_index();
	/*2.*/
	//printfo(MAXVAL); printf("before\n");
	dev_ptr = dev_lock1;
	register unsigned int bsize = 0;
	register int blocks;
	int prev =0;
	lock_count = 0;
	time_t rstart,rend,rt;
	rstart=clock();
	for(i = 2; i <= items; i++) {
		time_t start,end,t;
		
		start=clock();
		int splittings;
		double threads;
		c = c = (1 << i) -1;
		splittings =  COMBS(c);///NPERBLOCK;
		threads = ((double) splittings)/ NPERBLOCK;
		threads = ceil(threads);
		for(; c <= MAXVAL;) {
			while( bsize < MAXBLOCKSIZE && threads > bsize) {
				bsize += 32;
			}
			blocks =(int)  ceil((threads/bsize));

			subsetcomp22<<<blocks,bsize,0,stream[streamcount]>>>(dev_f,dev_o,dev_ptr,c,splittings,lock_count,bids[c]);

			t = c | (c-1);
			c = (t + 1) | (((~t & -~t) - 1) >> (__builtin_ctz(c) + 1));

			count++;
			lock_count++;	
			streamcount++;
			if(streamcount >= streams)
				streamcount = 0;
			if(lock_count < devcount)
				continue;
			HANDLE_ERROR(hipMemset(dev_ptr,0,devcount*sizeof(int)));

			if(dev_ptr == dev_lock1)
				dev_ptr = dev_lock2;
			else
				dev_ptr = dev_lock1;
			lock_count = 0;
		}

		for (int t = 0; t < streams; ++t) {
			HANDLE_ERROR(hipStreamSynchronize(stream[t]));
		}

		HANDLE_ERROR(hipDeviceSynchronize());
		

		end=clock();
		t=(end-start)/(CLOCKS_PER_SEC/1000);
		printf("ended card %d blocks\t %d threads/block %u, n kernels %u \t time %lu \t splittings %d\n",i,blocks,bsize,count-prev,t,splittings);
		prev =	count;

	}
	for (int i = 0; i < streams; ++i)
	hipStreamDestroy(stream[i]);

	HANDLE_ERROR(hipDeviceSynchronize());

	rend=clock();
	rt=(rend-rstart)/(CLOCKS_PER_SEC);
	printf("real time %lu\n",rt);

	HANDLE_ERROR(hipMemcpy(f,dev_f,MAXVAL*sizeof(int),hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipMemcpy(O,dev_o,MAXVAL*sizeof(int),hipMemcpyDeviceToHost));
	//int i;
	HANDLE_ERROR(hipFree(dev_f));
	HANDLE_ERROR(hipFree(dev_o));
	HANDLE_ERROR(hipFree(dev_lock1));
	HANDLE_ERROR(hipFree(dev_lock2));

	HANDLE_ERROR(hipDeviceReset());
//	printfo(MAXVAL);
	return count;
}



int main(void) {
	/*Start n amount of assets*/
	dint from = NAGENTS;
	dint MAXVAL = (2 << (from-1));
	
	time_t start,end,t;
	O = (dint * ) malloc(sizeof(dint)*(2 << (from-1)));
	bids = (dint * ) malloc(sizeof(dint)*(2 << (from-1)));
	f = bids;
	int ret_val =0;
	int count;
	while(ret_val == 0) {
	
	MAXVAL = (2 << (from-1));
	gen_rand_bids(MAXVAL);
	set_singleton_bid(MAXVAL);
	printf("maxval %u from %u\n",MAXVAL,from);
	start=clock();//predefined  function in c
	 count = run_test(MAXVAL,from);
	end=clock();
	t=(end-start)/CLOCKS_PER_SEC;
	ret_val= parse_wopt(MAXVAL);
	printf("\nTime taken =%lu for n= %u with count %d average per count %lf\n", (unsigned long) t,from,count,(double)t/count);
	}


	free(O);
	free(f);

	return 0;
}
