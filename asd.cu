#include <stdio.h> 
#include <string.h> // for ffs
#include <stdint.h>
#include <math.h>
#include <time.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

/*Debug enabled gives more print statements of bids and how the "Matrix" gets evaluated*/
#define DEBUG 0
#define TRUE 1
#define FALSE 0
/*Test sets all bids to one, which should give you n=|ITEMS| bids on output*/
#define TEST 1
/*Defines from 0-Range the random will give out*/
#define RANGE 10000
#define ITEMS 25
#define MAX (2 << (ITEMS-1))
#if ITEMS < 8
#define dint uint8_t  
#elif ITEMS < 16
#undef dint
#define dint uint16_t
#elif ITEMS < 32
#undef dint
#define dint uint32_t
#elif ITEMS > 32
#undef dint
#define dint uint64_t
#endif

#define BLOCKSIZE 512
#define HALFBLOCK 256
#if HALFBLOCK*2 != BLOCKSIZE
#error HALFBLOCK is not set correctly
#endif
#define SUBSET(X)((~_conf+(X+1))&_conf)
#define SETSUM(X)(f[setdiff(_conf,X)]+f[X])
#define I (threadIdx.x + BLOCKSIZE * blockIdx.x)

static void HandleError( hipError_t err, const char *file,int line ) {
	if (err != hipSuccess) {
		printf( "%s in %s at line %d\n", hipGetErrorString( err ),file, line );
		exit( EXIT_FAILURE );
	}
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))



/*		           0 1 2 3 4 5 6 7 8			*/
dint * bids;
// dint bids[MAX] ={0,2,3,6,4,7,6,9}; //conf 5 and 2
//dint bids[MAX] =  {0,2,3,6,4,6,6,9}; //conf 3 and 4
// dint bids[MAX] =  {0,20,3,6,4,6,10,9}; //conf 1 and 6
dint * f;
dint * O;

struct _stack {
	dint conf;
	struct _stack * next;
} typedef stack;

struct _locklist {
	unsigned int size;
	unsigned int conf;
	unsigned int * dev_f;
	unsigned int * dev_o;
	struct _locklist * next;
} typedef locklist;

struct _lockstruct {
	unsigned int * dev_lock;
	struct _lockstruct * next;
} typedef lockstruct;


#define setdiff(seta,setb) (seta & ~setb)

inline  dint cardinality( dint seta) {
	return __builtin_popcount(seta);
}

void gen_rand_bids(dint MAXVAL) {
	register dint i = 0;
#if TEST
	for(i = 1; i < MAXVAL;i++) {
		bids[i] = 1;
		O[i] = i;
	}
	bids[1] =0;
	bids[2] = 0;
#else
	for(i = 1; i < MAXVAL;i++) {
		bids[i] = rand() % RANGE;
	}
	for(i = 1; i < MAXVAL;i*=2) {
		bids[i] = 1;//rand() % RANGE;
	}
#endif

#if DEBUG
	printf("i =\t");
	for(i = 1; i < MAX;i++) {
		printf("%u\t",i);
	}
	printf("\n");
	printf("val =\t");
	for(i = 1; i < MAX;i++) {
		printf("%u\t",bids[i]);
	}
	printf("\n");
#endif
}

const char *btb(dint y)
{
	int x = y;
	static char b[9];
	b[0] = '\0';
	
	int z;
	for (z = 128; z > 0; z >>= 1)
	{
		strcat(b, ((x & z) == z) ? "1" : "0");
	}	
	return b;
}

inline void printfo() {
#if DEBUG
	dint i;
	printf("i\t");
	for(i =1; i < MAX; i++) {
		printf("%u\t",i);
	}
	printf("\n");
	printf("f[]\t");
	for(i =1; i < MAX; i++) {
		printf("%u\t",f[i]);
	}
	printf("\n");
	printf("O[]\t");
	for(i =1; i < MAX; i++) {
		printf("%u\t",O[i]);
	}
	printf("\n");
#endif
}

/*Sets all bids with one element in it, |n| = 1*/
inline void set_singleton_bid(dint MAXVAL) {
	register  dint i;
	for(i =1; i< MAXVAL; i*=2) {
		f[i] = bids[i];
		if(bids[i] > 0)
			O[i] = i;
	}
}

/* conf a e.g. 1101
 * (~a+i) & a gives a subset of a
 *  i is a integer from 1 to |a|
 *
 * ~1101 = 0010
 * i = 0001
 * (0010+0001)&1101 =
 * 0011&1101 = 0001
 *
 *i = 0011
 * (0010+0011)&1101
 *(0101)&1101 = 0101
 */



void printfo(dint MAXVAL) {
	int i;
	printf("\n");
	for(i = 1; i< MAXVAL; i++)
	{
		printf("Bid[%d]\t%u\tF[%d]\t%u\tO[%d]\t%u\tbin\t%s\n",i,bids[i],i,f[i],i,O[i],btb(i));

	}
}

void parse_wopt(dint MAXVAL) {
	printf("parse maxval = %u\n",MAXVAL);
	//wopt at start contain MAX at wopt[0] which is the combination that goes in bids[wopt[n]]
	stack * root =(stack *) malloc(sizeof(stack));
	stack * sroot = NULL;
	stack * scurr = NULL;
	//DO N	OT REMOVE -1
	root->conf = (MAXVAL)-1;
	int count = 0;
	root->next = NULL;
	stack * curr = root;
	while(curr) {
		dint conf = curr->conf;
		if(conf == 0) {
			printf("EXIT FAILURE\n");
			return;
		}
		/*if something is wrong*/
		if(count > 40) {
			return;
		}
			printf("curr %u\t",curr->conf);
		if(conf != O[conf]) {

			dint diff = setdiff(conf,O[conf]);
			curr->conf = O[conf];
			stack * tmp = (stack *) malloc(sizeof(stack));
			printf("diff %u\t O[conf] %u f %u\n",diff,O[conf],f[conf]);
			tmp->conf = diff;
			tmp->next = curr;
			root = tmp;
			curr = root;
			count++;
			continue;
		}
		if(sroot == NULL) {
			sroot = curr;
			scurr = curr;
		} else {
			/*set next pointer to the next singelton*/
			scurr->next = curr;
		}

		/*set the current singleton to be the pointer*/
		scurr = curr;
		curr = curr->next;
		/*clear the pinter to avoid infinite loop if cross-referenced*/
		scurr->next = NULL;

	}
	curr = sroot;
	dint tmp = 0;
	while(curr != NULL) {
		printf("conf %u value %u\n",curr->conf,bids[curr->conf]);
		tmp++;
		stack * tmp = curr;
		curr = curr->next;
		free(tmp);
	}
	printf("n = %u\n",tmp);
}

/*n 15 t 9 n 16 t 42*/
void max2(dint conf) {
	register dint card = cardinality(conf)/2;
	register dint combinations = 1 << (cardinality(conf)-1);
	register dint max = bids[conf];
	register dint set = conf;
	register dint tmp = 0;
	register dint subset;
	register dint inverse = ~conf;
	register dint i;
	for(i = 1;i<combinations; i++) {
		subset = (inverse+i)&conf;
		if(cardinality(subset) > card)
			continue;
		tmp = f[setdiff(conf,subset)] + f[subset];
		if(max < tmp) {
			max = tmp;
			set = subset;
		}
	}
	f[conf] = max;
	O[conf] = set;
}


//the index




/*
 *
 * 1. gen all combinations of card n
 * 2. for each combination, generate all subset with condition |s| < |c|/2
 * 3. for each subset check if |s| < |c|/2 then compute the sum
 * 4. 
 *
 *
 *
 */
__global__ void setlock(unsigned int * lock) {
		lock[I] = 0;
}
__global__ void subsetcomp22(unsigned int * f, /*Bid value*/
			     unsigned int * O, /*The move array*/
			     unsigned int * lock,
			     unsigned int _conf, /*The configuration*/
			     unsigned int cardmax, /*cardinality of max allowance*/
			     unsigned int maxval,
			     unsigned int count,
			    	unsigned int offset)
{

	__shared__ unsigned int share[BLOCKSIZE];
	__shared__ unsigned int step[BLOCKSIZE];
		/* printf("threadid.x\t%d\tblockDim.x\t%d\tblockIdx.x\t%d\tI\t%u\n", */
	/*        threadIdx.x, */
	/*        blockDim.x, */
	/*        blockIdx.x, */
	/*        conf); */
	unsigned int i = I+offset;
	/*subset var, but also for indexing later on*/
	unsigned int s = SUBSET(i);
//	unsigned int s2 = SUBSET(i);

	unsigned int tid = threadIdx.x;


	step[tid] = share[tid] = 0;
	if(i < maxval) {
		if(__popc(s) <= cardmax ) {
			share[tid] = f[setdiff(_conf,s)] + f[s];
			step[tid] = s;
		}
	}
	s= blockDim.x >> 1;
	__syncthreads();
	for (; s>0; s>>=1) {
		if (tid < s && (i < maxval)) {
			if(share[tid] < share[tid + s]) {
				step[tid] = step[tid+s];
				share[tid] = share[tid+s];
			}
		}
		__syncthreads();
	}

	if(tid == 0) {
		if(lock[count+1] < share[0]) {
			if(atomicMax(&(lock[count]),share[0]) < share[0]) {
				lock[count+1] = share[0];
				O[_conf] = step[0];
				f[_conf] =share[0];
				
			} 
		}
	}
}



#define COMBS(X) ((1 << cardinality(X)) - 1)

int run_test(dint MAXVAL,dint items) {
/*Setup the environment*/
	//dint perm[MAXVAL];
	printfo();
	unsigned int i, c,t,count =0;
//	f = bids;
 	unsigned int *dev_f,*dev_o;

	i = items/2;
	count = 0;
	/* for(c = (1 << i) -1; c <= MAXVAL;) { */
	/* 	count++; */
	/* 	t = c | (c-1); */
	/* 	c = (t + 1) | (((~t & -~t) - 1) >> (__builtin_ctz(c) + 1)); */
	/* } */
	
	unsigned int * dev_lock;
	unsigned int devcount = 1024;// count;
	unsigned int count2 = 0;
	unsigned int * cpy_lock =(unsigned int *)malloc((devcount+10)*sizeof(int));
	for(i = 0; i< devcount+10; i++)
		cpy_lock[i] = 0;
	printf("count %u\n",devcount);
	

	count = 0;
	HANDLE_ERROR(hipMalloc((void **)&dev_lock,(10+devcount)*sizeof(int)));
 	HANDLE_ERROR(hipMalloc((void **)&dev_f, MAXVAL*sizeof(int)));
 	HANDLE_ERROR(hipMalloc((void **)&dev_o, MAXVAL*sizeof(int)));

 	HANDLE_ERROR(hipMemcpy(dev_f,bids,MAXVAL*sizeof(int),hipMemcpyHostToDevice));
 	HANDLE_ERROR(hipMemcpy(dev_o,O,MAXVAL*sizeof(int),hipMemcpyHostToDevice));
//	HANDLE_ERROR(cudaMemcpy(dev_lock,cpy_lock,(10+devcount)*sizeof(int),cudaMemcpyHostToDevice));
	HANDLE_ERROR(hipMemset(dev_lock,0,devcount*sizeof(int)));
	/*2.*/
//	printfo(MAXVAL); printf("before\n");

	double bsize = 0;
	for(i = 2; i <= items; i++) {
		//	count =0;
		/*Generate all combinations of cardinality i*/
		count2 = 0;// bsize = 0;
		//	c = (1 << i) -1;
		//	printf("blocks %d\n",(COMBS(c)/BLOCKSIZE)+1);
		for(c = (1 << i) -1; c <= MAXVAL;) {

			double tmp = (double) COMBS(c);
			
			while( bsize <= 128 && tmp > bsize) {
				bsize += 32;
			}
			int blocks =(int)  ceil((tmp/bsize));
			int remainder = blocks - 65535;
	//		while( blocks > 65535 ) {
//				bsize += 32;
//				blocks =(int)  ceil((tmp/bsize));
//			}
			//double bsize = BLOCKSIZE;
			if(remainder > 0) {
				blocks =65535;
				subsetcomp22<<<remainder,bsize>>>(dev_f,dev_o,dev_lock,c,i/2,tmp,count2,65535*bsize);
			
			}

			subsetcomp22<<<blocks,bsize>>>(dev_f,dev_o,dev_lock,c,i/2,tmp,count2,0);
		
			t = c | (c-1);
			c = (t + 1) | (((~t & -~t) - 1) >> (__builtin_ctz(c) + 1));
			count++;
			count2 +=2;	
			if(count2 < devcount)
				continue;
			HANDLE_ERROR(hipDeviceSynchronize());		
			HANDLE_ERROR(hipMemset(dev_lock,0,devcount*sizeof(int)));
//			HANDLE_ERROR(cudaMemcpy(dev_lock,cpy_lock,(devcount)*sizeof(int),cudaMemcpyHostToDevice));
			count2 = 0;
		}

		HANDLE_ERROR(hipDeviceSynchronize());
		HANDLE_ERROR(hipMemset(dev_lock,0,devcount*sizeof(int)));
//		HANDLE_ERROR(cudaMemcpy(dev_lock,cpy_lock,(10+devcount)*sizeof(int),cudaMemcpyHostToDevice));
      
		printfo();
	}
	HANDLE_ERROR(hipDeviceSynchronize());
	HANDLE_ERROR(hipMemcpy(f,dev_f,MAXVAL*sizeof(int),hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipMemcpy(O,dev_o,MAXVAL*sizeof(int),hipMemcpyDeviceToHost));
	//int i;
	HANDLE_ERROR(hipFree(dev_f));
	HANDLE_ERROR(hipFree(dev_o));
	HANDLE_ERROR(hipFree(dev_lock));


	HANDLE_ERROR(hipDeviceReset());
	free(cpy_lock);
//	printfo(MAXVAL);
	//printf("items %u F[%u] = %u\n",items,MAXVAL,f[MAXVAL-1]);
	parse_wopt(MAXVAL);
	return count;
}



int main(void) {
	/*Start n amount of assets*/
	dint from = 23;
	/*End amount of assets , inclusive*/
	dint till = 23;
	dint MAXVAL = (2 << (from-1));


	time_t start,end,t;
	O = (dint * ) malloc(sizeof(dint)*(2 << (till-1)));
	bids = (dint * ) malloc(sizeof(dint)*(2 << (till-1)));
	f = bids;
	/*Run all tests*/
	for(;from <= till;from++) {
		MAXVAL = (2 << (from-1));
		gen_rand_bids(MAXVAL);
		set_singleton_bid(MAXVAL);
		printf("maxval %u from %u\n",MAXVAL,from);
		  start=clock();//predefined  function in c
		  int count = run_test(MAXVAL,from);
		  end=clock();
		  t=(end-start)/CLOCKS_PER_SEC;
		  printf("\nTime taken =%lu for n= %u with count %d average per count %lf\n", (unsigned long) t,from,count,(double)t/count);
/*Reset the arrays*/
		memset(&f,'\0',sizeof(f));
		memset(&O,'\0',sizeof(O));
	}
	free(O);
	free(f);

	return 0;
}








///old stuff

__global__ void setglobal(unsigned int * f, /*Bid value*/
			  unsigned int * O, /*The move array*/
			  unsigned int * tf,
			  unsigned int * to, /*The configuration*/
			  unsigned int conf /*cardinality of max allowance*/
			  ) {

	unsigned int tid = threadIdx.x;
//	while(to[threadIdx.x] == 0);
//	while(tf[threadIdx.x] == 0);
	extern __shared__ unsigned int share[];
	extern __shared__ unsigned int step[];
	share[tid] = tf[tid];
	step[tid]  = to[tid];

	__syncthreads();
	for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
		if (tid < s) {
			if(share[tid] < share[tid + s]) {
				share[tid] = share[tid+s];
				step[tid] = step[tid+s];
			}
		}
		__syncthreads();
	}

	if(tid == 0) {
		f[conf] = share[0];
		O[conf] = step[0];

		__threadfence();
	}
	
}

__global__ void subsetcomp2(unsigned int * f, /*Bid value*/
			   unsigned int * O, /*The move array*/
			   unsigned int cardset, /*The configuration*/
			    unsigned int cardmax, /*cardinality of max allowance*/
			    unsigned int bidperthread, /*how many bids should be looked at per thread*/
			    unsigned int maxval) 
/*The maximum value it can take, e.g tid can not be greater than 129*/
{
	__shared__ unsigned int _conf;
//	__shared__ unsigned int max[128];
//	__shared__ unsigned int step[128];
	unsigned int i,tid;
	unsigned int subset=0,sum = 0;
	unsigned int tmpset = 0,tmpsum = 0;
	/*thread 0 sets up variables*/
	if(threadIdx.x == 0) {
		/*set up first permutaion eg 0011 for cardset 2*/
		_conf = (1 << cardset) -1;
		
		/*generate the conf value*/
		for(i=0;i<blockIdx.x;i++) {
			tid = _conf | (_conf-1);
			_conf = (tid + 1) | (((~tid & -~tid) - 1) >> (__ffs(_conf)));
		}
		/*set the configuration value to shared memory*/
//		_conf = c;
		/*put it also in the global memory*/

		/*make sure that all blocks sees the change, could possibly discard it*/
		__threadfence();
		O[_conf] = _conf;
		/* printf("threadid.x\t%d\tblockDim.x\t%d\tblockIdx.x\t%d\tI\t%u\n", */
		/*        threadIdx.x, */
		/*        blockDim.x, */
		/*        blockIdx.x, */
		/*        _conf); */
	}
//	max[threadIdx.x] = 0;
//	step[threadIdx.x] = 0;
	__syncthreads();		
//	unsigned int comb = (1 << (__popc(_conf)-1));
	
	tid = threadIdx.x;//*2;
	
	for(i = 1; i<= bidperthread && tid < maxval;i++) {
		tmpset = SUBSET(tid);
		if(__popc(tmpset) <= cardmax) {
			tmpsum = f[tmpset]+f[setdiff(_conf,tmpset)];
			if(sum < tmpsum) {
				subset = tmpset;
				sum = tmpsum;
				
			}
			
		}
		tid += blockDim.x;
		//	printf("subset %u sum %u\n",tmpset,f[tmpset]+f[setdiff(_conf,tmpset)]);
	}

	__syncthreads();

//	unsigned int temp = 0;

	for(i = 0; i < blockDim.x;i++) {
		if(threadIdx.x == i) {
			if(f[_conf] < sum) {
				f[_conf] = sum;
				O[_conf] = subset;
			}
		}
		__syncthreads();
	}
}
__global__ void subsetcomp(unsigned int * f, /*Bid value*/
			   unsigned int * O, /*The move array*/
			   unsigned int conf, /*The configuration*/
			   unsigned int cardinality) /*cardinality of max allowance*/
{
	unsigned int max;
	/*tmp_max is a temporary max variable that is not subject to mutex lock*/
//	__shared__ unsigned int tmp_max;
	__shared__ unsigned int tmpstore[192];
	/* printf("threadid.x\t%d\tblockDim.x\t%d\tblockIdx.x\t%d\tI\t%d\n", */
	/*        threadIdx.x, */
	/*        blockDim.x, */
	/*        blockIdx.x, */
	/*        I); */
	unsigned int subset = (~conf+(I+1))&conf;
	if(__popc(subset) <= cardinality)
		tmpstore[I] = f[setdiff(conf,subset)] + f[subset];
	__syncthreads();
//	__threadfence();
	if(threadIdx.x == 0) {
		unsigned int i = 0;
		unsigned int c = 0;
		for(;i < blockDim.x;i++)
		{
			if(tmpstore[i] > max) {
				max = tmpstore[i];
				c = i;
			}
		}
		if(atomicMax(&f[conf],max) < max) {
			subset = (~conf+((c+blockDim.x*blockIdx.x)+1))&conf;
			atomicExch(&O[conf],subset);
		}
	}
}
__global__ void add(unsigned int * p, unsigned int * f, unsigned int * O)
{
	int tid = blockIdx.x;

	unsigned int conf = p[tid];
	unsigned int card = (unsigned int) __popc(conf)/2;
	unsigned int combinations = 1 << (__popc(conf) -1);
	unsigned int max = f[conf];
	unsigned int set = p[tid];

	unsigned int tmp = 0;
	unsigned int subset;
	unsigned int inverse = ~set;
	unsigned int i;
	if(max == 0) {
		printf("hello");
		return;
	}
/**/
	
	for(i = 1;i<combinations; i++) {
		subset = (inverse+i)&conf;
		if(__popc(subset) > card)
			continue;
		tmp = f[setdiff(conf,subset)] + f[subset];
		if(max < tmp) {
			max = tmp;
			set = subset;
		}
	}
	f[conf] = max;
	O[conf] = set;
}
