#include <stdio.h>
#include <string.h> // for ffs
#include <stdint.h>
#include <math.h>
#include <time.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>  
#include <hip/hip_runtime_api.h>
/*Debug enabled gives more print statements of bids and how the "Matrix" gets evaluated*/
#define DEBUG 0   
#define TRUE 1
#define FALSE 0
/*Test sets all bids to one, which should give you n=|ITEMS| bids on output*/
#define TEST 1
/*Defines from 0-Range the random will give out*/
#define RANGE 10000
#define ITEMS 25

//#define MAX (2 << (ITEMS-1))
#if ITEMS < 8
#define dint uint8_t
#elif ITEMS < 16
#undef dint
#define dint uint16_t
#elif ITEMS < 32
#undef dint
#define dint uint32_t
#endif

static void HandleError( hipError_t err, const char *file,int line ) {
	if (err != hipSuccess) {
		printf( "%s in %s at line %d\n", hipGetErrorString( err ),file, line );
		exit( EXIT_FAILURE );
	}
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


/*		           0 1 2 3 4 5 6 7 8			*/
unsigned short  * f, * bids;
// dint bids[MAX] ={0,2,3,6,4,7,6,9}; //conf 5 and 2
//dint bids[MAX] =  {0,2,3,6,4,6,6,9}; //conf 3 and 4
// dint bids[MAX] =  {0,20,3,6,4,6,10,9}; //conf 1 and 6

struct _stack {
	dint conf;
	struct _stack * next;
} typedef stack;


inline  dint cardinality( dint seta) {
	return __builtin_popcount(seta);
}
int indexa =0;
void gen_rand_bids(dint MAXVAL) {
	register dint i = 0;
#if TEST
	unsigned int seed = (unsigned)time ( NULL );
	srand(seed);
	indexa++;
	for(i = 1; i < MAXVAL;i++) {
		bids[i] = 1;//rand()%10+1;
		//	O[i] = i;
	}

	//indexa = rand() % MAXVAL;
	bids[indexa] = 100;
	printf("index %d \n",indexa);
	if(indexa >= MAXVAL) {
		printf("No error\n");
		exit(0);
	}

//	bids[1] =0;
//	bids[2] = 0;
//	bids[32769] = 20;
#else
	for(i = 1; i < MAXVAL;i++) {
		bids[i] = rand() % RANGE;
		//	O[i] = i;
	}
	for(i = 1; i < MAXVAL;i*=2) {
		bids[i] = rand() % RANGE;

	}
#endif

}

const char *btb(dint y)
{
	int x = y;
	static char b[9];
	b[0] = '\0';

	int z;
	for (z = 128; z > 0; z >>= 1)
	{
		strcat(b, ((x & z) == z) ? "1" : "0");
	}
	return b;
}

/*Sets all bids with one element in it, |n| = 1*/
inline void set_singleton_bid(dint MAXVAL) {
	register  dint i;
	for(i =1; i< MAXVAL; i*=2) {
		f[i] = bids[i];
	}
}



void printfo(dint MAXVAL) {
	int i;
	printf("\n");
	for(i = 1; i< MAXVAL; i++)
	{
		//printf("Bid[%d]\t%u\tF[%d]\t%u\tO[%d]\t%u\tbin\t%s\n",i,bids[i],i,f[i],i,O[i],btb(i));

	}
}


#define SUBSET(Y,X)(((~Y+1)+(X))&Y)

#define setdiff(seta,setb) (seta & ~setb)
//maxreg 32
//256
#define MAXBLOCKSIZE (1024)
//32
#define WARPSIZE (32)
//0
#define MIN_BLOCKS_PER_MP 4
#define NAGENTS (20)
//32  
#define NSTREAMS (16)
//2
#define NPERBLOCK (2)
#define confpwarp (2)
//32
#define CONFPKERNEL ((MAXBLOCKSIZE/32)*confpwarp)
//4
#define parasplittings (32)
#define NPARALLELCONF (4)
#define TIMING (0)

#define COMP(Z) {							\
		if(shared_value[tid][Z] < shared_value[tid+i][Z]) {	\
			shared_value[tid][Z] = shared_value[tid+i][Z];	\
		}							\
	}

#define CHECKPOINT(X) {							\
		stop_time = clock();					\
		if(tid == 0 && blockIdx.x == 0) {			\
			total = stop_time - start_time;			\
			printf(X,stop_time - start_time);		\
		}							\
		start_time =clock();					\
	}

#if (TIMING == 0)
#undef CHECKPOINT
#define CHECKPOINT(X) {}
#endif

#define MAX(X,Y) ((X) > (Y) ? (X) : (Y))


template<int blockSize, int overlastval>
__global__ void
__launch_bounds__(blockSize,MIN_BLOCKS_PER_MP)
	subsetcomp33(
		/*0*/	unsigned short * __restrict__ f, /*Bid value*/
		const unsigned int splittings,
		const unsigned int lastval,//the value a permutation can not exceed.
		const unsigned int card,
		const unsigned int conf1,
		const unsigned int conf2,
		const unsigned int conf3,
		const unsigned int conf4
		
		)
{
	__shared__ unsigned int conf[confpwarp][(blockSize/32)+1];
	__shared__ unsigned short value[confpwarp][(blockSize/32)+1];
	__shared__  unsigned char shift [(blockSize/32)*confpwarp][NAGENTS];// the shift matrix/array
	const unsigned int tid = threadIdx.x;
	const unsigned int laneId = (tid&31);
	const unsigned int warpId = tid/32;
	const unsigned int specsplittings = (splittings/32)+!!(splittings&31);
//(laneId < splittings)*(splittings/32)+(laneId < (splittings%32));,COMBS(29),(!!(COMBS(29)%32))+(COMBS(29)/32)
	const unsigned int initsplit = laneId*specsplittings;//+(laneId >= (splittings%32))*(splittings%32);
	
	// const unsigned int initsplit = tid*specsplittings;
	unsigned int leafsplit[2];
//	unsigned int rootsplit[confpwarp];
	unsigned int rvalue[confpwarp][2];
	/*Thread 0 of each warp*/
	if(!tid) {
		unsigned int tmp;
		unsigned int conftmp;
		if(blockIdx.x == 0) {
			conftmp = conf[0][warpId] = conf1;
		} else if(blockIdx.x == 1) {
			conftmp = conf[0][warpId] = conf2;
		} else if(blockIdx.x == 2) {
			conftmp = conf[0][warpId] = conf3;
		} else {
			conftmp = conf[0][warpId] = conf4;
		}
		int x;
		tmp = conftmp | (conftmp-1);
		conftmp = (tmp + 1) | (((~tmp & -~tmp) - 1) >> (__ffs(conftmp)));
		conf[1][warpId] = conftmp;
		for(x =1; x < (blockSize/32);x++) {
			tmp = conftmp | (conftmp-1);
			conftmp = (tmp + 1) | (((~tmp & -~tmp) - 1) >> (__ffs(conftmp)));
			conf[0][x] = conftmp;
			if(conftmp > lastval) {
				conf[1][x] = conftmp;
				continue;
			}
			tmp = conftmp | (conftmp-1);
			conftmp = (tmp + 1) | (((~tmp & -~tmp) - 1) >> (__ffs(conftmp)));
			conf[1][x] = conftmp;
		}
	}
	__syncthreads();
	if(conf[0][warpId] > lastval) {
		return;
	}
	if((__popc(conf[1][warpId] & conf[0][warpId]) == (card - 1)) && conf[1][warpId] < lastval) {
		if(laneId == 0) {
			unsigned int index;
			unsigned int count = 0;
			unsigned int conftmp = conf[0][warpId] & conf[1][warpId];

			//if(conf[1][warpId] < lastval) {
			value[1][warpId] = f[conf[1][warpId]];
			//}
			value[0][warpId] = f[conf[0][warpId]];
//#pragma unroll
			while(conftmp) {
				index = __ffs(conftmp) - 1; //find which index is first bit
				conftmp &= ~(1 << index);//set nth bit to 0
				shift[warpId][count] = index;
				shift[warpId+1][count] = index;
				count++;
			}
		}
	} else if(laneId < confpwarp) { //generate the shift arrays
		unsigned int index;
		unsigned int conftmp = conf[laneId][warpId];

		if(conftmp < lastval) {
			value[laneId][warpId] = f[conftmp];
		}
#pragma unroll
		for(int x = 0; x < card;x++) {//could put card in template to unroll
			index = __ffs(conftmp) - 1; //find which index is first bit
			conftmp &= ~(1 << index);//set nth bit to 0
			shift[warpId+laneId][x] = index;
		}

	}
	//__syncthreads();



	if(conf[1][warpId] < lastval) {
		unsigned int index;
		unsigned int splittmp = initsplit;
		leafsplit[0] = leafsplit[1] = 0;
		if((__popc(conf[1][warpId] & conf[0][warpId]) == (card - 1))) {
			while(splittmp) {
			index = __ffs(splittmp)-1;
			leafsplit[1] += (1 << shift[warpId+1][index]);			
			splittmp &= ~(1 << index);
			}
			leafsplit[0] = leafsplit[1] = SUBSET((conf[1][warpId] & conf[0][warpId]),leafsplit[1]);
		} else {
			while(splittmp) {
			index = __ffs(splittmp)-1;
			leafsplit[1] += (1 << shift[warpId+1][index]);
			leafsplit[0] += (1 << shift[warpId][index]);//CHECK
			splittmp &= ~(1 << index);
		}
			leafsplit[1] = SUBSET(conf[1][warpId],leafsplit[1]);
			leafsplit[0] = SUBSET(conf[0][warpId],leafsplit[0]);
		}
	} else {
		unsigned int index;
		unsigned int splittmp= initsplit;
		leafsplit[0] = 0;
		while(splittmp) {
			index = __ffs(splittmp)-1;
			leafsplit[0] += (1 << shift[warpId][index]);//CHECK
			splittmp &= ~(1 << index);
		}
		leafsplit[0] = SUBSET(conf[0][warpId],leafsplit[0]);
	}
	if(!specsplittings) {
		return;
	}

	int y;


//	for(x = 0; x < specsplittings;x += parasplittings) {
	rvalue[0][0] =rvalue[0][1] = rvalue[1][1] = 0;
	if((__popc(conf[1][warpId] & conf[0][warpId]) == (card - 1)) && (conf[1][warpId] < lastval)) {
		
#pragma unroll 4		
		for(y = 0;y < specsplittings ;y +=2) {				
			if(y+initsplit < splittings) {
				
				int tmp = __popc(leafsplit[0]);
				if((NAGENTS-card) <= tmp && (NAGENTS-card+tmp) <= (card)) {
					rvalue[1][1] = rvalue[0][1] = f[leafsplit[0]];
						
					rvalue[0][1] += f[setdiff(conf[0][warpId],leafsplit[0])];						
					rvalue[1][1] += f[setdiff(conf[1][warpId],leafsplit[1])];					
				}
				leafsplit[1] = SUBSET((conf[1][warpId] & conf[0][warpId]),leafsplit[1]);
				leafsplit[0] = SUBSET((conf[1][warpId] & conf[0][warpId]),leafsplit[0]);
				if(rvalue[0][0] < rvalue[0][1]) {
					rvalue[0][0] = rvalue[0][1];
				}
				if(rvalue[1][0] < rvalue[1][1]) {
					rvalue[1][0] = rvalue[1][1];
				}

			}
			if(y+1+initsplit < splittings) {
//				rvalue[0][1] = rvalue[1][1] = 0;
				int tmp = __popc(leafsplit[0]);
				if((NAGENTS-card) <= tmp && (NAGENTS-card+tmp) <= (card)) {
					rvalue[1][1] = rvalue[0][1] = f[leafsplit[0]];
						
					rvalue[0][1] += f[setdiff(conf[0][warpId],leafsplit[0])];						
					rvalue[1][1] += f[setdiff(conf[1][warpId],leafsplit[1])];					
				}
				leafsplit[1] = SUBSET((conf[1][warpId] & conf[0][warpId]),leafsplit[1]);
				leafsplit[0] = SUBSET((conf[1][warpId] & conf[0][warpId]),leafsplit[0]);
				if(rvalue[0][0] < rvalue[0][1]) {
					rvalue[0][0] = rvalue[0][1];
				}
				if(rvalue[1][0] < rvalue[1][1]) {
					rvalue[1][0] = rvalue[1][1];
				}

			}

		}

	} else {		
		//	rvalue[0][0] =rvalue[0][1] = rvalue[1][1] = 0;
		if(conf[1][warpId] < lastval) {
#pragma unroll 4		
		for(y = 0;y < specsplittings ;y+=2) {
			if(y+initsplit < splittings) {
				
				int tmp = __popc(leafsplit[0]);
				if((NAGENTS-card) <= tmp && (NAGENTS-card+tmp) <= (card)) {

					rvalue[0][1] = f[leafsplit[0]];
					rvalue[1][1] = f[leafsplit[1]];								
					rvalue[0][1] += f[setdiff(conf[0][warpId],leafsplit[0])];

					rvalue[1][1] += f[setdiff(conf[1][warpId],leafsplit[1])];
		
				}
				leafsplit[0] = SUBSET(conf[0][warpId],leafsplit[0]);
				leafsplit[1] = SUBSET(conf[1][warpId],leafsplit[1]);
				if(rvalue[0][0] < rvalue[0][1]) {
					rvalue[0][0] = rvalue[0][1];
				}
				if(rvalue[1][0] < rvalue[1][1]) {
					rvalue[1][0] = rvalue[1][1];
				}
	
			}
			if(y+1+initsplit < splittings) {
				//rvalue[0][1] = rvalue[1][1] = 0;
				int tmp = __popc(leafsplit[0]);
				if((NAGENTS-card) <= tmp && (NAGENTS-card+tmp) <= (card)) {

					rvalue[0][1] = f[leafsplit[0]];
					rvalue[1][1] = f[leafsplit[1]];								
					rvalue[0][1] += f[setdiff(conf[0][warpId],leafsplit[0])];
					rvalue[1][1] += f[setdiff(conf[1][warpId],leafsplit[1])];					
				}
				leafsplit[0] = SUBSET(conf[0][warpId],leafsplit[0]);
				leafsplit[1] = SUBSET(conf[1][warpId],leafsplit[1]);
				if(rvalue[0][0] < rvalue[0][1]) {
					rvalue[0][0] = rvalue[0][1];
				}
				if(rvalue[1][0] < rvalue[1][1]) {
					rvalue[1][0] = rvalue[1][1];
				}
	
			}
		}
		} else {
#pragma unroll 4
		for(y = 0;y < specsplittings ;y+=2) {
			if(y+initsplit < splittings) {
				
				int tmp = __popc(leafsplit[0]);
				if((NAGENTS-card) <= tmp && (NAGENTS-card+tmp) <= (card)) {

					rvalue[0][1] = f[leafsplit[0]];
						
					rvalue[0][1] += f[setdiff(conf[0][warpId],leafsplit[0])];

				}
				leafsplit[0] = SUBSET(conf[0][warpId],leafsplit[0]);

				if(rvalue[0][0] < rvalue[0][1]) {
					rvalue[0][0] = rvalue[0][1];
				}
			}
			if(y+1+initsplit < splittings) {
				//rvalue[0][1] = rvalue[1][1] = 0;
				int tmp = __popc(leafsplit[0]);
				if((NAGENTS-card) <= tmp && (NAGENTS-card+tmp) <= (card)) {

					rvalue[0][1] = f[leafsplit[0]];
						
					rvalue[0][1] += f[setdiff(conf[0][warpId],leafsplit[0])];

				}
				leafsplit[0] = SUBSET(conf[0][warpId],leafsplit[0]);

				if(rvalue[0][0] < rvalue[0][1]) {
					rvalue[0][0] = rvalue[0][1];
				}
			}
		}
	}

	}

	#// pragma unroll	       
	// 		for(y = 1;y < parasplittings;y++) {
	// 			if(x+y+initsplit >= splittings) {
	// 				continue;
	// 			}

	// 		}


	 // if(__any( ( (rvalue[0][0] > value[0][warpId]) || (rvalue[1][0] > value[1][warpId]) ) ) == 0) {
	 // 	 return;
	 // }
		
	if(__any( ( rvalue[0][0] > value[0][warpId] ) ) ) {
		rvalue[0][1] = __shfl_xor((int)rvalue[0][0],16,32);
		rvalue[0][0] = MAX(rvalue[0][0],rvalue[0][1]);
		rvalue[0][1] = __shfl_xor((int)rvalue[0][0],8,32);
		rvalue[0][0] = MAX(rvalue[0][0],rvalue[0][1]);
		rvalue[0][1] = __shfl_xor((int)rvalue[0][0],4,32);
		rvalue[0][0] = MAX(rvalue[0][0],rvalue[0][1]);
		rvalue[0][1] = __shfl_xor((int)rvalue[0][0],2,32);
		rvalue[0][0] = MAX(rvalue[0][0],rvalue[0][1]);
		rvalue[0][1] = __shfl_xor((int)rvalue[0][0],1,32);
		rvalue[0][0] = MAX(rvalue[0][0],rvalue[0][1]);
	}
		
	if(__any( ( rvalue[1][0] > value[1][warpId] ) ) ) {
		if(conf[1][warpId] < lastval) {
			rvalue[1][1] = __shfl_xor((int)rvalue[1][0],16,32);
			rvalue[1][0] = MAX(rvalue[1][0],rvalue[1][1]);
			rvalue[1][1] = __shfl_xor((int)rvalue[1][0],8,32);
			rvalue[1][0] = MAX(rvalue[1][0],rvalue[1][1]);
			rvalue[1][1] = __shfl_xor((int)rvalue[1][0],4,32);
			rvalue[1][0] = MAX(rvalue[1][0],rvalue[1][1]);
			rvalue[1][1] = __shfl_xor((int)rvalue[1][0],2,32);
			rvalue[1][0] = MAX(rvalue[1][0],rvalue[1][1]);
			rvalue[1][1] = __shfl_xor((int)rvalue[1][0],1,32);
			rvalue[1][0] = MAX(rvalue[1][0],rvalue[1][1]);
		}
	}
		
	if(laneId == 0) {
		if(rvalue[0][0] > value[0][warpId]) {
			value[0][warpId] = rvalue[0][0];
		}
		if(rvalue[1][0] > value[1][warpId]) {
			value[1][warpId] = rvalue[1][0];
		}
	}


	if(laneId == 0) {
		if(conf[0][warpId] < lastval) {
			if(value[0][warpId] > f[conf[0][warpId]]) {
				f[conf[0][warpId]] = value[0][warpId];
			}
		}
		
		if(conf[1][warpId] < lastval) {
			if(value[1][warpId] > f[conf[1][warpId]]) {
				f[conf[1][warpId]] = value[1][warpId];
			}
		}
	}
	return;
}



#define COMBS(X) ((1 << (X-1)) - 1)

int run_test(unsigned int MAXVAL,dint items) {

	register unsigned int i,c1,count =0;
	unsigned short *dev_bids;

	count = 0;

	HANDLE_ERROR(hipDeviceReset());
	HANDLE_ERROR(hipSetDeviceFlags(hipDeviceScheduleYield));
  	HANDLE_ERROR(hipDeviceSetCacheConfig(hipFuncCachePreferL1));
//	HANDLE_ERROR(cudaDeviceSetLimit(cudaLimitMallocHeapSize,0));
	HANDLE_ERROR(hipDeviceSetLimit(hipLimitStackSize,0));
	HANDLE_ERROR(hipDeviceSetLimit(hipLimitMallocHeapSize,0));
	HANDLE_ERROR(hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte));

	register unsigned int streams = NSTREAMS;
	register unsigned int streamcount = 0;
	register hipStream_t stream[streams];
	for(i = 0;i < streams; i++)
		HANDLE_ERROR(hipStreamCreate(&stream[i]));

	count = 0;

 	HANDLE_ERROR(hipMalloc((void **)&dev_bids, MAXVAL*sizeof(short)));

 	HANDLE_ERROR(hipMemcpy(dev_bids,bids,MAXVAL*sizeof(short),hipMemcpyHostToDevice));

	register unsigned int bsize = MAXBLOCKSIZE;
	register int blocks;
	int prev =0;
//	lock_count = 0;
	time_t rstart,rend,rt;
	rstart=clock();
	for(i = 2; i <= NAGENTS; i++) {
		time_t start,end,t;

		start=clock();
		unsigned int splittings;
		blocks =4;//(int)  ceil((threads/bsize));
		double threads;
		c1 = (1 << i) -1;
		unsigned int c2 = c1;
		unsigned int c3;
		unsigned int ca[blocks];
		unsigned int cacount;
		splittings = ((1 << (i-1))-1);// COMBS(c1);///NPERBLOCK;
		threads = ((double) splittings)/ NPERBLOCK;
		threads = ceil(threads);
		const int o = 1;
		const int a[3] = {0,1,(2 << 2)};
		for(; c1 <= MAXVAL;) {

			cacount = 1;
			ca[0] = c1;			
			for(int x = 0, cacount = 1; x < CONFPKERNEL*blocks;x++) {
				t = c2 | (c2-1);
				c2 = (t + 1) | (((~t & -~t) - 1) >> (__builtin_ctz(c2) + 1));
				if(x%CONFPKERNEL == 0 && x > 1) {
					ca[cacount] = c2;
					cacount++;
				}
			}
			if((c2 > MAXVAL)) {
						      
		
					subsetcomp33 < MAXBLOCKSIZE , 1> <<<blocks,MAXBLOCKSIZE,0,stream[streamcount]>>>(dev_bids,splittings,MAXVAL,i,ca[0],ca[1],ca[2],ca[3]);
		
			}else{
		
					subsetcomp33 < MAXBLOCKSIZE , 0> <<<blocks,MAXBLOCKSIZE,0,stream[streamcount]>>>(dev_bids,splittings,MAXVAL,i,ca[0],ca[1],ca[2],ca[3]);
			
			}
			c1 = c2;


			streamcount++;
			count++;

			if(streamcount >= streams)
				streamcount = 0;
		}
		

		for (int t = 0; t < streams; ++t) {
			HANDLE_ERROR(hipStreamSynchronize(stream[t]));
		}

		HANDLE_ERROR(hipDeviceSynchronize());


		end=clock();
		t=(end-start)/(CLOCKS_PER_SEC/1000);
		printf("ended card %d blocks\t %d threads/block %u, n kernels %u \t time %lu \t splittings %d time per kernel %lf\n",i,blocks,bsize,count-prev,t,splittings,(double)t/(count-prev));
		prev =	count;

	}
	for (int i = 0; i < streams; ++i)
		hipStreamDestroy(stream[i]);

	HANDLE_ERROR(hipDeviceSynchronize());

	rend=clock();
	rt=(rend-rstart)/(CLOCKS_PER_SEC/1000);
	printf("real time %lu ms\n",rt);

	HANDLE_ERROR(hipMemcpy(f,dev_bids,MAXVAL*sizeof(short),hipMemcpyDeviceToHost));
//	HANDLE_ERROR(cudaMemcpy(O,dev_o,MAXVAL*sizeof(int),cudaMemcpyDeviceToHost));
	//int i;
	HANDLE_ERROR(hipFree(dev_bids));
//	HANDLE_ERROR(cudaFree(dev_o));
	// HANDLE_ERROR(cudaFree(dev_lock1));
	// HANDLE_ERROR(cudaFree(dev_lock2));

	HANDLE_ERROR(hipDeviceReset());
//	printfo(MAXVAL);
	return count;
}

dint max2(dint conf) {
     register dint card = cardinality(conf)/2;
     register dint combinations = (1 << cardinality(conf)-1)-1;
     register dint max = f[conf];
     register dint tmp = 0;
     register dint subset = 0;
     register const dint inverse = ~conf;
     register dint i;
     for(i = 1;i<=combinations; i++) {
	     subset = ((inverse+1)+subset)&conf;
	     tmp = f[setdiff(conf,subset)] + f[subset];
	     if(max == tmp) {
		     break;
		     //  return subset;
	     }

     }
     return subset;
}

int recur_parse_wopt(dint MAXVAL) {
	stack * root = (stack *) malloc(sizeof(stack));
	stack * sroot = NULL;
	stack * scurr = NULL;
		root->conf = (MAXVAL)-1;
	int count = 0;
	root->next = NULL;
	stack * curr = root;
	while(curr) {
		dint conf = curr->conf;
		if(conf == 0) {
			printf("EXIT FAILURE\n");
			return 1;
		}
		/*if something is wrong*/
		if(count > 40) {
			fprintf(stderr,"Something went wrong at line %d in %s\n",__LINE__,__FILE__);
			return 1;
		}
//		printf("curr %u\t\n",curr->conf);
		if(f[conf] != bids[conf]) {
			dint proper_subset = max2(conf);
			dint diff = setdiff(conf,proper_subset);
			curr->conf = proper_subset;
			stack * tmp = (stack *) malloc(sizeof(stack));
			//printf("diff %u\t conf %u\t O[diff] %u\t O[conf]\t f %u\n",diff,conf,O[diff],O[conf],f[conf]);
			tmp->conf = diff;
			tmp->next = curr;
			root = tmp;
			curr = root;
			count++;
			continue;
		}
		if(sroot == NULL) {
			sroot = curr;
			scurr = curr;
		} else {
			/*set next pointer to the next singelton*/
			scurr->next = curr;
		}

		/*set the current singleton to be the pointer*/
		scurr = curr;
		curr = curr->next;
		/*clear the pinter to avoid infinite loop if cross-referenced*/
		scurr->next = NULL;

	}
	printf("\n");
	curr = sroot;
	dint tmp = 0;
	while(curr != NULL) {
		if(bids[curr->conf]) {
			if(curr->conf == indexa) {
				printf("correct bid\n");
				tmp++;
				return 0;
			}
			//printf("conf %u value %u\n",curr->conf,bids[curr->conf]);

		}
		stack * tmp = curr;
		curr = curr->next;
		free(tmp);
	}
	if(tmp < 1) {
		printf("something is wrong, no bids\n");
		return 1;

	}
	printf("n = %u\n",tmp);
	return 0;
}


int main(void) {
	/*Start n amount of assets*/
	dint from = NAGENTS;
	const unsigned long  MAXVAL = (2 << (from-1));
	time_t start,end,t;
//	O = (dint * ) malloc(sizeof(dint)*(2 << (from-1)));
	bids = (unsigned short * ) malloc(sizeof(short)*(2 << (from-1)));

	f =  (unsigned short * ) malloc(sizeof(short)*(2 << (from-1)));
	
//	return;
//	f = bids;
	int ret_val =1;
	int count;
	while(ret_val == 1) {
		
		//MAXVAL = (2 << (from-1));
		gen_rand_bids(MAXVAL);
		printf("hello\n");
		set_singleton_bid(MAXVAL);

		printf("maxval %u from %u\n",MAXVAL,from);
		start=clock();//predefined  function in c
		count = run_test(MAXVAL,from);
		end=clock();
		t=(end-start)/CLOCKS_PER_SEC;
		ret_val= recur_parse_wopt(MAXVAL);// parse_wopt(MAXVAL);
		printf("\nTime taken =%lu for n= %u with count %d average per count %lf\n", (unsigned long) t,from,count,(double)t/count);
	}
	free(f);

	return 0;
}
