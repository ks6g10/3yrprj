#include <stdio.h>
#include <string.h> // for ffs
#include <stdint.h>
#include <math.h>
#include <time.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

/*Debug enabled gives more print statements of bids and how the "Matrix" gets evaluated*/
#define DEBUG 0
//legacy code
#define TRUE 1
#define FALSE 0
/*Test sets all bids to one, which should give you n=|ITEMS| bids on output*/
#define TEST 1
/*Defines from 0-Range the random will give out*/
//legacy code
#define RANGE 10000
#define ITEMS 25

#define MAX (2 << (ITEMS-1))

//legacy code
#if ITEMS < 8
#define dint uint8_t
#elif ITEMS < 16
#undef dint
#define dint uint16_t
#elif ITEMS < 32
#undef dint
#define dint uint32_t
#endif

static void HandleError( hipError_t err, const char *file,int line ) {
	if (err != hipSuccess) {
		printf( "%s in %s at line %d\n", hipGetErrorString( err ),file, line );
		exit( EXIT_FAILURE );
	}
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


//legacy var
uint32_t  * O;

unsigned short  * f, * bids;

struct _stack {
	dint conf;
	struct _stack * next;
} typedef stack;


inline  dint cardinality( dint seta) {
	return __builtin_popcount(seta);
}
int indexa =0;

//generate the bids, indexa is the controll bid, should always be present
void gen_rand_bids(dint MAXVAL) {
	register dint i = 0;
#if TEST
	unsigned int seed = (unsigned)time ( NULL );
	srand(seed);
	indexa++;
	for(i = 1; i < MAXVAL;i++) {
		bids[i] = rand()%10+1;
		O[i] = i;
	}
	bids[indexa] = 100;
	printf("index %d \n",indexa);
	if(indexa >= MAXVAL) {
		printf("No error\n");
		exit(0);
	}
#else
	for(i = 1; i < MAXVAL;i++) {
		bids[i] = rand() % RANGE;
		O[i] = i;
	}
	for(i = 1; i < MAXVAL;i*=2) {
		bids[i] = rand() % RANGE;

	}
#endif

}
//legacy code
const char *btb(dint y)
{
	int x = y;
	static char b[9];
	b[0] = '\0';

	int z;
	for (z = 128; z > 0; z >>= 1)
	{
		strcat(b, ((x & z) == z) ? "1" : "0");
	}
	return b;
}

//legacy code
/*Sets all bids with one element in it, |n| = 1*/
inline void set_singleton_bid(dint MAXVAL) {
	register  dint i;
	for(i =1; i< MAXVAL; i*=2) {
		f[i] = bids[i];
		if(bids[i] > 0)
			O[i] = i;
	}
}


//legacy code
void printfo(dint MAXVAL) {
	int i;
	printf("\n");
	for(i = 1; i< MAXVAL; i++)
	{
		printf("Bid[%d]\t%u\tF[%d]\t%u\tO[%d]\t%u\tbin\t%s\n",i,bids[i],i,f[i],i,O[i],btb(i));

	}
}


#define SUBSET(Y,X)(((~Y+1)+(X))&Y)

#define setdiff(seta,setb) (seta & ~setb)
//maxreg 32
//256
#define MAXBLOCKSIZE (256)
//32
#define WARPSIZE (32)
//0
#define MIN_BLOCKS_PER_MP 8
#define NAGENTS (24)
//32 
#define NSTREAMS (32) 
//2
#define NPERBLOCK (2)
//32
#define CONFPKERNEL (32)
//4
#define NPARALLELCONF (4) 
#define TIMING (0)

#define COMP(Z) {							\
		if(shared_value[tid][Z] < shared_value[tid+i][Z]) {	\
			shared_value[tid][Z] = shared_value[tid+i][Z];	\
		}							\
	}

//timing function
#define CHECKPOINT(X) {							\
		stop_time = clock();					\
		if(tid == 0 && blockIdx.x == 0) {			\
			total = stop_time - start_time;			\
			printf(X,stop_time - start_time);		\
		}							\
		start_time =clock();					\
	}

#if (TIMING == 0)
#undef CHECKPOINT
#define CHECKPOINT(X) {}
#endif

template<int blockSize,int nparallelconf,int confpkernel,int nperblock,int currblocksize>
__global__ void
__launch_bounds__(currblocksize,MIN_BLOCKS_PER_MP)  
	subsetcomp33(
		/*0*/	unsigned short * __restrict__ f, /*value*/	       
		/*2*/	unsigned int * __restrict__ lock, /*pointer to the atomic lock array*/
		/*5*/	unsigned int maxval, /*the maximum number of subsets possible*/
		/*6*/	unsigned short count1, /*which index in the lock array*/
		unsigned int conf1, /*the intitial coalition structure*/
		unsigned int lastval//the value a coalition permutation can not exceed.
		)
{  
	//shared memory
	__shared__  unsigned short shared_value[(currblocksize >> 5)+1][nparallelconf]; // the values for each warp
	__shared__  unsigned int conf[confpkernel];// the configurations needed for the whole execution
	__shared__  unsigned char shift [confpkernel][NAGENTS];// the shift matrix/array
//registers
 	register unsigned int subset_value[nperblock][nparallelconf];//the value for one of the subset sums
	register unsigned int subset_conf[2];// temporary array to hold the subset coalition
	register unsigned int const tid = threadIdx.x;
	register unsigned int const ispec = nperblock*(threadIdx.x + blockDim.x * blockIdx.x);//I + offset;
	register int x,i,z; //counter

#if (TIMING == 1)
	clock_t stop_time, total,start_time;
	total = 0;
	start_time = clock(); 
#endif

	if(tid == 0) {
		subset_conf[0] = conf[0] = conf1;
		/* Generates the coalition structures using the bit tricks
		 * google "Compute the lexicographically next bit permutation"
		 *
		 */
#pragma unroll	
		for(x = 1;x < confpkernel; x++) { 			
			z = subset_conf[0] | (subset_conf[0]-1);
			subset_conf[0] = (z + 1) | (((~z & -~z) - 1) >> (__ffs(subset_conf[0])));
			conf[x]= subset_conf[0];
		}
		CHECKPOINT("l1\t %d\n"); // first checkpoint, see paper
	}
//	__syncthreads();

#if (TIMING == 1)
	start_time = clock(); 
#endif
	/* Generates the shift array in order to in the future to create subsets.
	 * as confpkernel is <= 32, this will be done in one warp, hence no need
	 * for the syncthreads above.
	 */
	
	if(tid < confpkernel) { //generate the shift arrays		
		subset_conf[0] = conf[tid]; // re-use registers tmpval
		unsigned int cnt = 0;
		while(subset_conf[0]) {
			subset_conf[1] = __ffs(subset_conf[0]) - 1; //find which index is first bit
			subset_conf[0] &= ~(1 << subset_conf[1]);//set nth bit to 0
			shift[tid][cnt++] = subset_conf[1];
		}
	}

	CHECKPOINT("l2\t %d\n");

	__syncthreads(); //all other warps except the first one will wait here for the first one to finish

#pragma unroll	
	for(x =0; x < confpkernel; x += nparallelconf) { // fetches the values in batches

#if (TIMING == 1)
		start_time = clock(); 
#endif
		if(conf[x] > lastval) {//if the permutation is larger than the full set, c> (1 << NAGENTS)
			continue;
		}
#pragma unroll	
 		for(z=0; z < nparallelconf;z++) {//reset values
 			subset_value[0][z] = subset_value[1][z] = 0U;			
 		}

		CHECKPOINT("l3\t %d\n");

		if(ispec >= maxval) { // no need to fetch, have greater splitting index than splittings possible
			goto postfetch;
		}

		CHECKPOINT("l3\t %d\n");

		/* Hold onto your hat, this will be a wild ride
		 * Here is where the collision detection happens
		 * Which collitions do I try to find?
		 * The ones that causes collisions and those that make the runtime faster
		 * ergo not all that causes a collision
		 * How did I find which causes collision?
		 * if(coalision_x == coalision_y) print(hello world)
		 * I evaluate two coallisions at the same time, 
		 * will reference them as second and first coalition
		 * and subset means one half of a splitting, and setdiff means the other half
		 */

		unsigned int tmp[2];
		tmp[1]= tmp[0] = 0; //reset
		subset_conf[1] = 0; //reset
#pragma unroll
		for(z=0;z < nparallelconf;z +=2) {

			if(conf[z+x] >= lastval) {
				continue;
			}		 
			unsigned int tmpstore = 0, tmpstore1 = 0;
			unsigned int index;
			//unsigned int const tcar = __popc(tmp);
			unsigned int wshile = ispec;
			subset_conf[0] = 0;
			
			/* Generate the subset (intitialsplit) and fetch the value for the second coalition structure first
			 * Why? It is so much faster than generating and fetching the first
			 * Why? Do not know, but my guess is that it is better to block early one time than to block
			 * twice even though it should be the same time, I guess better scheduling, did not profile that part.
			 * Escept for speed, gained like 5-6 seconds @ 24 agents
			 */
			if(conf[z+x+1] < lastval) {
				wshile = ispec;
				subset_conf[1] = 0;
				while(wshile) {
					index = __ffs(wshile)-1;
					subset_conf[1] += (1 << shift[x+z+1][index]);
					wshile &= ~(1 << index);	
				}
				/*As I start from index 0, need to use makro SUBSET( nextSplit) once before you get the right value*/
				subset_conf[1] = SUBSET(conf[z+x+1],subset_conf[1]); 
				/*Fetch the value*/
				subset_value[0][z+1] = f[subset_conf[1]];
			}
			while(wshile) {
				index = __ffs(wshile)-1;
				subset_conf[0] += (1 << shift[x+z][index]);
				wshile &= ~(1 << index);
				if(wshile) {
					index = __ffs(wshile)-1;
					subset_conf[0] += (1 << shift[x+z][index]);
					wshile &= ~(1 << index);
				}
			}
			subset_conf[0] = SUBSET(conf[z+x],subset_conf[0]);

			/*If there is enough splittings, we can get the next splitting as well
			* just use SUBSET (nextsplit) to generate the next splitting
			* this is for the first coalition structure
			**/
			if((ispec+1) < maxval) {
				tmp[0] = SUBSET(conf[z+x],subset_conf[0]);
				/*The first collision, I have pre-emptively stored the previouse 
				 * subset of the second splitting of the second coalision structure's
				 * value in the register for this second splitting of the first coalision structure
				 * if they are not the same you shal fetch me a new value
				 */
				if(tmp[0] != tmp[1]) {
					subset_value[1][z] = f[tmp[0]];
				}
			}
			/* Fetch the first coalitions first splittings value
			 */
			subset_value[0][z] = f[subset_conf[0]];		
			
			/*The second splitting of the second coalition
			 */
			if((ispec+1) < maxval && conf[z+x+1] < lastval) {
				tmp[1] = SUBSET(conf[z+x+1],subset_conf[1]);
				//Collision
				if(tmp[1] == tmp[0]) {
					subset_value[1][z+1] = subset_value[1][z];
				} else {
					subset_value[1][z+1] = f[tmp[1]];
				}
			}

			subset_conf[0] = (setdiff(conf[z+x],subset_conf[0]));
			tmpstore = f[subset_conf[0]];//

			//the setdiff of the second splitting @ first coalition
			if((ispec+1) < maxval) {
				tmp[0] = (setdiff(conf[z+x],tmp[0]));
				tmpstore1 = f[tmp[0]];
			}
			
			//second coalision
			if(conf[z+x+1] < lastval) {
				subset_conf[1] = (setdiff(conf[z+x+1],subset_conf[1]));
				if(z < (nparallelconf-2)){
				 	subset_value[1][z+2] = subset_value[1][z+1]; // preemptive storage for first Collision
				}
				//Collision
				if(subset_conf[1] == subset_conf[0]){// dont remove
					subset_value[0][z+1] += tmpstore;					
				} else{
					subset_value[0][z+1] += f[subset_conf[1]];
				}
				//next splitting
				if((ispec+1) < maxval) {
					tmp[1] = (setdiff(conf[z+x+1],tmp[1]));
					subset_value[1][z+1] += f[tmp[1]];										
				}
			}
			subset_value[0][z] += tmpstore;
			subset_value[1][z] += tmpstore1;
			tmp[1] = (setdiff(conf[z+x+1],tmp[1])); //get the subset instead of the setdiff in order to do the first Collision

		}
		CHECKPOINT("l4\t %d\n");
					
	postfetch:

#pragma unroll	
		for(z = 0; z < nparallelconf;z++) {
			if(conf[z+x] >= lastval) {
				continue;
			}
			if(subset_value[1][z] > subset_value[0][z]) { // see which one is greater
				subset_value[0][z] = subset_value[1][z];
			}
#pragma unroll	
			for(i = 16;i >=1;i >>=1) {//warp reduction
				int warp_value = __shfl_xor((int)subset_value[0][z],i,32); // exchange values between threads
				if(warp_value > subset_value[0][z]) {
					subset_value[0][z] =(unsigned int) warp_value;
				}
			}
			//tid&(WARPSIZE-1) == tid%WARPSIZE
			//Only threads with line id == 0 is allowed to update in the shared memory,
			//i.e. the first thread in each warp
			if(!(tid&(31))) {
				unsigned int index = tid >> 5; // tid >> 5 == tid / 32 which warp it is
				shared_value[index][z] = subset_value[0][z];
			}

		}

		CHECKPOINT("l5\t %d\n");		

		//template optimisation
		if((currblocksize/32) > 1) {
 		__syncthreads();
		}
		//how many warps is it, block dimension divided by warp size
		//e.g. 256/32 == 256 >> 5
//		if((currblocksize/32) > 1) {//evaluated by the pre-processor
			
		i = blockDim.x >> 6;
		if(tid<i) {
			for(; i > 0; i >>= 1) {//reduction in shared memory
				for(z=0; z < nparallelconf;z +=4) {
					COMP(z); // small macro, lazy to put it back
				}
			}
		}
		
		CHECKPOINT("l6\t %d\n");
		/*
		 *Update the value in f using the atomic lock array
		 */
		if((currblocksize) >= blockSize) {//evaluated by the pre-processor
			//__syncthreads(); // no need as the shared memory reduction is in the same warp
			if(tid == 0) {
#pragma unroll	
				for(z=0; z < nparallelconf;z++) {
					if(conf[z+x] > lastval) {continue;}
					if(f[conf[z+x]] < shared_value[0][z]) {
						if(atomicMax(&(lock[count1+x+z]),shared_value[0][z]) < shared_value[0][z]) {
							if(f[conf[z+x]] < shared_value[0][z]) {
								f[conf[z+x]] = shared_value[0][z];
							}
						}
					}
				}
			}
		} else {
			/*if only one block*/
#pragma unroll
			for(z=0; z < nparallelconf;z++) {
				if(conf[z+x] > lastval) {continue;}
				if(f[conf[z+x]] < shared_value[0][z]) {
					f[conf[z+x]] = shared_value[0][z];
						
				}

			}
		}
		CHECKPOINT("l7\t %d\n");
	}
}


#define COMBS(X) ((1 << cardinality(X)-1) - 1)

int run_test(dint MAXVAL,dint items) {
/*Setup the environment*/
	//dint perm[MAXVAL];

	register unsigned int i,c1,count =0;
	unsigned short *dev_bids;

	count = 0;
 
	HANDLE_ERROR(hipDeviceReset());
	HANDLE_ERROR(hipSetDeviceFlags(hipDeviceScheduleYield)); 
  	HANDLE_ERROR(hipDeviceSetCacheConfig(hipFuncCachePreferL1));
	HANDLE_ERROR(hipDeviceSetLimit(hipLimitMallocHeapSize,0));
	HANDLE_ERROR(hipDeviceSetLimit(hipLimitStackSize,0));
	HANDLE_ERROR(hipDeviceSetLimit(hipLimitMallocHeapSize,0));
	HANDLE_ERROR(hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte));
	unsigned int * dev_lock1,*dev_lock2,*dev_ptr;
	const	unsigned int devcount = 1024*CONFPKERNEL;// count;
	register unsigned int streams = NSTREAMS;
	register unsigned short lock_count = 0;
	register unsigned int streamcount = 0;
	register hipStream_t stream[streams];
	for(i = 0;i < streams; i++)
		HANDLE_ERROR(hipStreamCreate(&stream[i]));

	count = 0;
	HANDLE_ERROR(hipMalloc((void **)&dev_lock1,(devcount)*sizeof(int)));
	HANDLE_ERROR(hipMalloc((void **)&dev_lock2,(devcount)*sizeof(int)));

 	HANDLE_ERROR(hipMalloc((void **)&dev_bids, MAXVAL*sizeof(short)));

 	HANDLE_ERROR(hipMemcpy(dev_bids,bids,MAXVAL*sizeof(short),hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemset(dev_lock1,0,devcount*sizeof(int)));
	HANDLE_ERROR(hipMemset(dev_lock2,0,devcount*sizeof(int)));
	/*2.*/
	dev_ptr = dev_lock1;
	register unsigned int bsize = 32;
	register int blocks;
	int prev =0;
	lock_count = 0;
	time_t rstart,rend,rt;
	rstart=clock();
	// start with small coalition structures and go big
	for(i = 2; i <= items; i++) {
		time_t start,end,t;

		start=clock();
		int splittings;

		double threads;
		c1 = (1 << i) -1;
		splittings =  COMBS(c1);///NPERBLOCK;
		threads = ((double) splittings)/ NPERBLOCK;
		threads = ceil(threads);
		
		for(; c1 <= MAXVAL;) {
			while( bsize < MAXBLOCKSIZE && threads > bsize ) {//dynamicly adjust the blocksize, 32 -> 256
				bsize += 32;
			}
			blocks =(int)  ceil((threads/bsize)); // number of blocks
			

			switch(bsize) {
			case 32:
				subsetcomp33 < MAXBLOCKSIZE , NPARALLELCONF , CONFPKERNEL , NPERBLOCK,32> <<<blocks,32,0,stream[streamcount]>>>(dev_bids,dev_ptr,splittings,lock_count,c1,MAXVAL);
				break;
			case 64:
				subsetcomp33 < MAXBLOCKSIZE , NPARALLELCONF , CONFPKERNEL , NPERBLOCK,64> <<<blocks,64,0,stream[streamcount]>>>(dev_bids,dev_ptr,splittings,lock_count,c1,MAXVAL);
				break;
			case 128:
				subsetcomp33 < MAXBLOCKSIZE , NPARALLELCONF , CONFPKERNEL , NPERBLOCK,128> <<<blocks,128,0,stream[streamcount]>>>(dev_bids,dev_ptr,splittings,lock_count,c1,MAXVAL);
				break;
			case 256:
				subsetcomp33 < MAXBLOCKSIZE , NPARALLELCONF , CONFPKERNEL , NPERBLOCK,256> <<<blocks,256,0,stream[streamcount]>>>(dev_bids,dev_ptr,splittings,lock_count,c1,MAXVAL);
				break;
			// case 512:
			// 	subsetcomp33 < MAXBLOCKSIZE , NPARALLELCONF , CONFPKERNEL , NPERBLOCK,512> <<<blocks,512,0,stream[streamcount]>>>(dev_bids,dev_ptr,splittings,lock_count,c1,MAXVAL,i);
			// 	break;
			//  case 1024:
			//  	subsetcomp33 < MAXBLOCKSIZE , NPARALLELCONF , CONFPKERNEL , NPERBLOCK,1024> <<<blocks,1024,0,stream[streamcount]>>>(dev_bids,dev_ptr,splittings,lock_count,c1,MAXVAL,i);
			//  	break;	
				
			}
			//subsetcomp33 < MAXBLOCKSIZE , NPARALLELCONF , CONFPKERNEL , NPERBLOCK > <<<blocks,bsize,0,stream[streamcount]>>>(dev_bids,dev_o,dev_ptr,splittings,lock_count,c1,MAXVAL,i);

			for(int x = 0; x < CONFPKERNEL;x++) {
				t = c1 | (c1-1);
				c1 = (t + 1) | (((~t & -~t) - 1) >> (__builtin_ctz(c1) + 1));
			}
			streamcount++;
			count++;
  			
			lock_count += CONFPKERNEL;
			
			/*
			 *Handle the streams and locks
			 *have two lock arrays I switch between
			 * once one is full, switch and issue an memset on the full
			 *
			 */
			
			if(streamcount >= streams)
				streamcount = 0;
			if(lock_count < devcount)
				continue;
			HANDLE_ERROR(hipMemset(dev_ptr,0,devcount*sizeof(int)));

			if(dev_ptr == dev_lock1)
				dev_ptr = dev_lock2;
			else
				dev_ptr = dev_lock1;
			lock_count = 0;
		}

		for (int t = 0; t < streams; ++t) {
			HANDLE_ERROR(hipStreamSynchronize(stream[t]));
		}

		HANDLE_ERROR(hipDeviceSynchronize());


		end=clock();
		t=(end-start)/(CLOCKS_PER_SEC/1000);
		printf("ended card %d blocks\t %d threads/block %u, n kernels %u \t time %lu \t splittings %d\n",i,blocks,bsize,count-prev,t,splittings);
		prev =	count;

	}
	for (int i = 0; i < streams; ++i)
		hipStreamDestroy(stream[i]);

	HANDLE_ERROR(hipDeviceSynchronize());

	rend=clock();
	rt=(rend-rstart)/(CLOCKS_PER_SEC/1000);
	printf("real time %lu\n",rt);

	HANDLE_ERROR(hipMemcpy(f,dev_bids,MAXVAL*sizeof(short),hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipFree(dev_bids));
	HANDLE_ERROR(hipFree(dev_lock1));
	HANDLE_ERROR(hipFree(dev_lock2));

	HANDLE_ERROR(hipDeviceReset());
	return count;
}
//max function in DP in order to get the final splittings just like in idp of removing the splittings table of a given C
dint max2(dint conf) {
     register dint card = cardinality(conf)/2;
     register dint combinations = (1 << cardinality(conf)-1)-1;
     register dint max = f[conf];
     register dint tmp = 0;
     register dint subset = 0;
     register const dint inverse = ~conf;
     register dint i;
     for(i = 1;i<=combinations; i++) {
	     subset = ((inverse+1)+subset)&conf;
	     tmp = f[setdiff(conf,subset)] + f[subset];		     
	     if(max == tmp) {
		     break;
		     //  return subset;
	     }
	  
     }     
     return subset;
}
//this one handels the the retrival of the splittings like IDP
int recur_parse_wopt(dint MAXVAL) {
	stack * root = (stack *) malloc(sizeof(stack));
	stack * sroot = NULL;
	stack * scurr = NULL;
	root->conf = (MAXVAL)-1; //just like CS = {A} in DP
	int count = 0;
	root->next = NULL;
	stack * curr = root;
	while(curr) {
		dint conf = curr->conf;
		if(conf == 0) {
			printf("EXIT FAILURE\n");
			return 1;
		}
		/*if something is wrong*/
		if(count > 40) {
			fprintf(stderr,"Something went wrong at line %d in %s\n",__LINE__,__FILE__);
			return 1;
		}

		if(f[conf] != bids[conf]) {
			dint proper_subset = max2(conf);
			dint diff = setdiff(conf,proper_subset);
			curr->conf = proper_subset;
			stack * tmp = (stack *) malloc(sizeof(stack));
			//printf("diff %u\t conf %u\t O[diff] %u\t O[conf]\t f %u\n",diff,conf,O[diff],O[conf],f[conf]);
			tmp->conf = diff;
			tmp->next = curr;
			root = tmp;
			curr = root;
			count++;
			continue;
		}
		if(sroot == NULL) {
			sroot = curr;
			scurr = curr;
		} else {
			/*set next pointer to the next singelton*/
			scurr->next = curr;
		}

		/*set the current singleton to be the pointer*/
		scurr = curr;
		curr = curr->next;
		/*clear the pinter to avoid infinite loop if cross-referenced*/
		scurr->next = NULL;

	}
	printf("\n");
	curr = sroot;
	dint tmp = 0;
	while(curr != NULL) {
		if(bids[curr->conf]) {
			if(curr->conf == indexa) {
				printf("correct bid\n");
				tmp++;
				return 0;
			}
			//printf("conf %u value %u\n",curr->conf,bids[curr->conf]);

		}
		stack * tmp = curr;
		curr = curr->next;
		free(tmp);
	}
	if(tmp < 1) {
		printf("something is wrong, no bids\n");
		return 1;

	}
	printf("n = %u\n",tmp);
	return 0;
}
//legacy code of the above function with the final splittings in an array like DP
int parse_wopt(dint MAXVAL) {
	//printf("parse maxval = %u\n",MAXVAL);
	//wopt at start contain MAX at wopt[0] which is the combination that goes in bids[wopt[n]]
	stack * root =(stack *) malloc(sizeof(stack));
	stack * sroot = NULL;
	stack * scurr = NULL;
	//DO N	OT REMOVE -1
	root->conf = (MAXVAL)-1;
	int count = 0;
	root->next = NULL;
	stack * curr = root;
	while(curr) {
		dint conf = curr->conf;
		if(conf == 0) {
			printf("EXIT FAILURE\n");
			return 1;
		}
		/*if something is wrong*/
		if(count > 40) {
			fprintf(stderr,"Something went wrong at line %d in %s\n",__LINE__,__FILE__);
			return 1;
		}
		printf("curr %u\t\n",curr->conf);
		if(conf != O[conf]) {

			dint diff = setdiff(conf,O[conf]);
			curr->conf = O[conf];
			stack * tmp = (stack *) malloc(sizeof(stack));
			printf("diff %u\t conf %u\t O[diff] %u\t O[conf] %u\t f %u\n",diff,conf,O[diff],O[conf],f[conf]);
			tmp->conf = diff;
			tmp->next = curr;
			root = tmp;
			curr = root;
			count++;
			continue;
		}
		if(sroot == NULL) {
			sroot = curr;
			scurr = curr;
		} else {
			/*set next pointer to the next singelton*/
			scurr->next = curr;
		}

		/*set the current singleton to be the pointer*/
		scurr = curr;
		curr = curr->next;
		/*clear the pinter to avoid infinite loop if cross-referenced*/
		scurr->next = NULL;

	}
	printf("\n");
	curr = sroot;
	dint tmp = 0;
	while(curr != NULL) {
		if(bids[curr->conf]) {
			if(curr->conf == indexa) {
				printf("correct bid\n");
				tmp++;
				return 0;
			}
			printf("conf %u value %u\n",curr->conf,bids[curr->conf]);

		}
		stack * tmp = curr;
		curr = curr->next;
		free(tmp);
	}
	if(tmp < 1) {
		printf("something is wrong, no bids\n");

		return 1;

	}
	printf("n = %u\n",tmp);
	return 0;
}

int main(void) {
	/*Start n amount of assets*/
	dint from = NAGENTS;
	dint MAXVAL = (2 << (from-1));

	time_t start,end,t;
	O = (dint * ) malloc(sizeof(dint)*(2 << (from-1)));
	bids = (unsigned short * ) malloc(sizeof(short)*(2 << (from-1)));
	f = (unsigned short * ) malloc(sizeof(short)*(2 << (from-1)));
//	f = bids;
	int ret_val =0;
	int count;
	while(ret_val == 0) { // remove this to only test one valye, not all

		MAXVAL = (2 << (from-1));
		gen_rand_bids(MAXVAL);
		set_singleton_bid(MAXVAL);
		printf("maxval %u from %u\n",MAXVAL,from);
		start=clock();//predefined  function in c
		count = run_test(MAXVAL,from);
		end=clock();
		t=(end-start)/CLOCKS_PER_SEC;
		ret_val= recur_parse_wopt(MAXVAL);// parse_wopt(MAXVAL);
		printf("\nTime taken =%lu for n= %u with count %d average per count %lf\n", (unsigned long) t,from,count,(double)t/count);
	}


	free(O);
	free(f);

	return 0;
}

//legacy code
template<int blockSize,int nparallelconf,int confpkernel,int nperblock,int currblocksize>
__global__ void
__launch_bounds__(currblocksize,MIN_BLOCKS_PER_MP)  
	subsetcomp32(
		/*0*/	unsigned short * __restrict__ f, /*Bid value*/
		/*1*/	unsigned int * __restrict__ O, /*The move array*/
		/*2*/	unsigned int * __restrict__ lock,
		/*5*/	unsigned int maxval,
		/*6*/	unsigned short count1,
		unsigned int conf1,
		unsigned int lastval,//the value a permutation can not exceed.
		unsigned int card)
{  
	//confpkernel = how many configurations the kernel will evaluate
	//nparallelconf = how many configurations the kernel will evaluate at the same time
	__shared__  unsigned short shared_value[(currblocksize >> 5)+1][nparallelconf];
	__shared__  unsigned int shared_conf[(currblocksize >> 5)+1][nparallelconf];
	__shared__  unsigned int conf[confpkernel];// the configurations needed for the whole execution
	__shared__  unsigned short shift [confpkernel][NAGENTS];// the shift matrix/array
	//__shared__ volatile unsigned int tmp [confpkernel][2];
 	register unsigned int subset_value[2][8];//the value for one of the subset sums
	register unsigned int subset_conf[2][8];
   
	register unsigned int count = count1;
	register unsigned int const tid = threadIdx.x;
//	__shared__ unsigned int
	register unsigned int ispec = nperblock*(threadIdx.x + blockDim.x * blockIdx.x);//I + offset;
	register int x,i,z; //counter

#if (TIMING == 1)
	clock_t stop_time, total,start_time;
	total = 0;
	start_time = clock(); 
#endif

	if(tid == 0) {
		conf[0] = conf1;
#pragma unroll	
		for(x = 1;x < confpkernel; x++) { // generate the configurations
			conf[x] = conf[x-1];
			z = conf[x] | (conf[x]-1);
			conf[x] = (z + 1) | (((~z & -~z) - 1) >> (__ffs(conf[x])));
		}
		CHECKPOINT("l1 %d\n");
	}
	if(confpkernel > 32) {
		__syncthreads();
	}
	
	count = count1;
	if(tid < confpkernel) { //generate the shift arrays
		subset_conf[0][0] = conf[tid]; // re-use registers tmpval
		subset_conf[0][1] = 0;// re-use registers index
#pragma unroll		
		for(x = 0,i=0; x < card;x++) {//could put card in template to unroll
			subset_conf[0][1] = __ffs(subset_conf[0][0]) - 1; //find which index is first bit
			subset_conf[0][0] &= ~(1 << subset_conf[0][1]);//set nth bit to 0
			shift[tid][x] = subset_conf[0][1];
		}
	}

	CHECKPOINT("l2 %d\n");

	__syncthreads();

#pragma unroll	
	for(x =0; x < confpkernel; x += nparallelconf) {

		if(conf[x] > lastval) {//if the permutation is larger than the full set, c> (1 << NAGENTS)
			continue;
		}
//#pragma unroll	
//		for(i = 0; i < nperblock; i++) {
#pragma unroll	
 		for(z=0; z < nparallelconf;z++) {
 			subset_value[0][z] = subset_value[1][z] = 0U;			
 		}
//		}
		CHECKPOINT("l3 %d\n");
		if(ispec >= maxval) {
			goto postfetch;
		}
			//This for loop initilize the first subset configuration.

#pragma unroll	
		for(i = 0; i < nparallelconf; i++) { 
			unsigned int tmp = ispec;
			//unsigned int const tcar = __popc(tmp);
			subset_conf[0][i] = 0;
			while(tmp) {
				unsigned short index = __ffs(tmp)-1;
				subset_conf[0][i] += (1 << shift[x+i][index]);//CHECK
				tmp &= ~(1 << index);				
			}
		}
		CHECKPOINT("l4 %d\n");	
#pragma unroll
		for(z=0;z < nparallelconf;z++) {
			if(conf[z+x] > lastval) {
				continue;
			}
			subset_conf[0][z] = SUBSET(conf[z+x],subset_conf[0][z]);
			subset_value[0][z] = f[(setdiff(conf[z+x],subset_conf[0][z]))] + f[subset_conf[0][z]];
			//ispec++;
			if((ispec+1) >= maxval) {
				continue;
			}
			subset_conf[1][z] = SUBSET(conf[z+x],subset_conf[0][z]);
			subset_value[1][z] = f[(setdiff(conf[z+x],subset_conf[1][z]))] + f[subset_conf[1][z]];	
		}
		CHECKPOINT("l6 %d\n");
					
	postfetch:

#pragma unroll	
		for(z = 0; z < nparallelconf;z++) {//warp reduction
			if(subset_value[1][z] > subset_value[0][z]) {
				subset_value[0][z] = subset_value[1][z];
				subset_conf[0][z] = subset_conf[1][z];
			}
#pragma unroll	
			for(i = 16;i >=1;i >>=1) {
				int warp_value = __shfl_xor((int)subset_value[0][z],i,32);
				int warp_conf = __shfl_xor((int)subset_conf[0][z],i,32);
				if(warp_value > subset_value[0][z]) {
					subset_value[0][z] =(unsigned int) warp_value;
					subset_conf[0][z] =(unsigned int) warp_conf;
				}
			}
			//tid&(WARPSIZE-1) == tid%WARPSIZE
			//Only threads with line id == 0 is allowed to update in the shared memory,
			//i.e. the first thread in each warp
			if(!(tid&(31))) {
				unsigned int index = tid >> 5; // tid >> 5 == tid / 32 which warp it is
				shared_value[index][z] = subset_value[0][z];
				shared_conf[index][z] = subset_conf[0][z];
			}

		}
		CHECKPOINT("l7 %d\n");		

		//	CHECKPOINT("l8 %d\n");
		if((currblocksize/32) > 1) {
 		__syncthreads();
		}
		//how many warps is it, block dimension divided by warp size
		//e.g. 256/32 == 256 >> 5
		if((currblocksize/32) > 1) {//evaluated by the pre-processor
			i = (currblocksize >> 6);//blockDim.x >> 6;
			if(tid<i) {//reduction mby move down if you get wrong results gained ~1000 cycles

#pragma unroll	
			for(; i > 0; i >>= 1) {
#pragma unroll	
  					for(z=0; z < nparallelconf;z +=4) {
						COMP(z);
  					}
			}
			}
		}
		
		CHECKPOINT("l9 %d\n");
		if((currblocksize) >= blockSize) {//evaluated by the pre-processor
			//__syncthreads();
			if(tid == 0) {
#pragma unroll	
				for(z=0; z < nparallelconf;z++) {
					if(f[conf[z+x]] < shared_value[0][z]) {
						//	printf("lock val %u shared_val %s\n",lock[count+z] ,shared_value[0][z]);
						if(atomicMax(&(lock[count+z]),shared_value[0][z]) < shared_value[0][z]) {
							//	O[conf[z+x]] = shared_conf[0][z];
							f[conf[z+x]] = shared_value[0][z];
						}
					}
				}
			}
		} else {
#pragma unroll
			for(z=0; z < nparallelconf;z++) {
				if(f[conf[z+x]] < shared_value[0][z]) {
					//	O[conf[z+x]] = shared_conf[0][z];
					f[conf[z+x]] = shared_value[0][z];
						
				}

			}
		}

		count += nparallelconf;
		ispec = NPERBLOCK*(threadIdx.x + blockDim.x * blockIdx.x);//I + offset;

	}
}
