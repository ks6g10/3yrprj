#include <stdio.h>
#include <string.h> // for ffs
#include <stdint.h>
#include <math.h>
#include <time.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


//Not removing useless stuff that may have traces left in the code as I am on my laptop which can not compile cuda code.

//LEGACY START - Could possibly be removed, do not listen to the comments here, it does not work
/*Debug enabled gives more print statements of bids and how the "Matrix" gets evaluated*/
#define DEBUG 0
#define TRUE 1
#define FALSE 0
/*Test sets all bids to one, which should give you n=|ITEMS| bids on output*/
#define TEST 1
/*Defines from 0-Range the random will give out*/
#define RANGE 10000
#define ITEMS 25


#if ITEMS < 8
#define dint uint8_t
#elif ITEMS < 16
#undef dint
#define dint uint16_t
#elif ITEMS < 32
#undef dint
#define dint uint32_t
#endif
//LEGACY END


//Handle errors for cuda
static void HandleError( hipError_t err, const char *file,int line ) {
	if (err != hipSuccess) {
		printf( "%s in %s at line %d\n", hipGetErrorString( err ),file, line );
		exit( EXIT_FAILURE );
	}
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

//useless
uint32_t  * O;

//the bids and f array, why not make them global eh.
unsigned short  * f, * bids;

//Small stack implementation
struct _stack {
	dint conf;
	struct _stack * next;
} typedef stack;


inline  dint cardinality( dint seta) {
	return __builtin_popcount(seta);
}
int indexa =0; //the joker coalition
void gen_rand_bids(dint MAXVAL) {
	register dint i = 0;
#if TEST
	unsigned int seed = (unsigned)time ( NULL );
	srand(seed);
	indexa++;
	for(i = 1; i < MAXVAL;i++) {
		bids[i] = 1;//rand()%10+1;
		O[i] = i;
	}

	//indexa = rand() % MAXVAL;
	bids[indexa] = 100; //set the joker to something high
	printf("index %d \n",indexa);
	if(indexa >= MAXVAL) {
		printf("No error\n");
		exit(0);
	}

#else
	for(i = 1; i < MAXVAL;i++) {
		bids[i] = rand() % RANGE;
		O[i] = i;
	}
	for(i = 1; i < MAXVAL;i*=2) {
		bids[i] = rand() % RANGE;

	}
#endif

}

const char *btb(dint y)
{
	int x = y;
	static char b[9];
	b[0] = '\0';

	int z;
	for (z = 128; z > 0; z >>= 1)
	{
		strcat(b, ((x & z) == z) ? "1" : "0");
	}
	return b;
}

/*Sets all bids with one element in it, |n| = 1*/
inline void set_singleton_bid(dint MAXVAL) {
	register  dint i;
	for(i =1; i< MAXVAL; i*=2) {
		f[i] = bids[i];
		if(bids[i] > 0)
			O[i] = i;
	}
}



void printfo(dint MAXVAL) {
	int i;
	printf("\n");
	for(i = 1; i< MAXVAL; i++)
	{
		printf("Bid[%d]\t%u\tF[%d]\t%u\tO[%d]\t%u\tbin\t%s\n",i,bids[i],i,f[i],i,O[i],btb(i));

	}
}


#define SUBSET(Y,X)(((~Y+1)+(X))&Y)

#define setdiff(seta,setb) (seta & ~setb)
//maxreg 32
// //You can touch this
#define MAXBLOCKSIZE (1024)
//32
#define WARPSIZE (32)
//0 //You can touch this
#define MIN_BLOCKS_PER_MP 4


//You can touch this
#define NAGENTS (25)

//32  //You can touch this
#define NSTREAMS (32)

//2 //You should probably not touch this
#define NPERBLOCK (2)

//DO NOT TOUCH THIS, IT IS NOT DYNAMIC
#define confpwarp (2)

//Cant touch this na na na na
#define CONFPKERNEL ((MAXBLOCKSIZE/32)*confpwarp)

//You can touch this
#define parasplittings (8)

//not used any more
#define NPARALLELCONF (4)

//have not inserted any timings in the code
#define TIMING (0)

#define CHECKPOINT(X) {							\
		stop_time = clock();					\
		if(tid == 0 && blockIdx.x == 0) {			\
			total = stop_time - start_time;			\
			printf(X,stop_time - start_time);		\
		}							\
		start_time =clock();					\
	}

#if (TIMING == 0)
#undef CHECKPOINT
#define CHECKPOINT(X) {}
#endif

#define MAX(X,Y) ((X) > (Y) ? (X) : (Y))


template<int blockSize, int overlastval>
__global__ void
__launch_bounds__(blockSize,MIN_BLOCKS_PER_MP)
	subsetcomp33(
		/*0*/	unsigned short * __restrict__ f, /*Bid value*/
		/*5*/	const unsigned int splittings,		
		const unsigned int lastval,//the value a permutation can not exceed.
		const unsigned int card,
		const unsigned int conf1,
		const unsigned int conf2,
		const unsigned int conf3,
		const unsigned int conf4
)
{
     //coalition structures  indexed [0-1][warpId]
	__shared__ unsigned int conf[confpwarp][(blockSize/32)+1];
	//values
	__shared__ unsigned short value[confpwarp][(blockSize/32)+1];
	//shift array for init split
	__shared__  unsigned char shift [(blockSize/32)*confpwarp][NAGENTS];// the shift matrix/array
	const unsigned int tid = threadIdx.x;
	const unsigned int laneId = (tid%32);
	const unsigned int warpId = tid/32;
	//How many splittings can we do per thread
	const unsigned int specsplittings = (!!(splittings%32))+(splittings/32);
	//the first splitting
	const unsigned int initsplit = tid*specsplittings;
	//Holds splittings
	unsigned int leafsplit[2];
	//values in registers
	unsigned int rvalue[confpwarp][parasplittings];
	/*Thread 0 of each warp*/
	if(!tid) {
		unsigned int tmp;
		unsigned int conftmp;
		//assign right coalition to right block
		if(blockIdx.x == 0) {
			conftmp = conf[0][warpId] = conf1;
		} else if(blockIdx.x == 1) {
			conftmp = conf[0][warpId] = conf2;
		} else if(blockIdx.x == 2) {
			conftmp = conf[0][warpId] = conf3;
		} else {
			conftmp = conf[0][warpId] = conf4;
		}
		int x;
		//genereate the next coalition structures
		tmp = conftmp | (conftmp-1);
		conftmp = (tmp + 1) | (((~tmp & -~tmp) - 1) >> (__ffs(conftmp)));
		conf[1][warpId] = conftmp;
		//genereate the next coalition structures
		for(x =1; x < (blockSize/32);x++) {
			tmp = conftmp | (conftmp-1);
			conftmp = (tmp + 1) | (((~tmp & -~tmp) - 1) >> (__ffs(conftmp)));
			conf[0][x] = conftmp;

			tmp = conftmp | (conftmp-1);
			conftmp = (tmp + 1) | (((~tmp & -~tmp) - 1) >> (__ffs(conftmp)));
			conf[1][x] = conftmp;
		}
	}
	__syncthreads();
	//if coalition greater than what we can calculate, i.e. out of range, e.g. 24th bit set in a 23 sized problem
	if(conf[0][warpId] > lastval) {
		return;
	}

	//the collision detection, if |c & c'| == |c|-1 && |c| == |c'| then we can generate all splittings for both of c and c' using their intersection
	// here we set up the shift array
	if((__popc(conf[1][warpId] & conf[0][warpId]) == (card - 1)) && conf[1][warpId] < lastval) {
	     //one thread
		if(laneId == 0) {
			unsigned int index;
			unsigned int count = 0;
			unsigned int conftmp = conf[0][warpId] & conf[1][warpId];


			//fetch the coalitions values to shared memory
			value[1][warpId] = f[conf[1][warpId]];

			value[0][warpId] = f[conf[0][warpId]];

			//look at the paper
			while(conftmp) {
				index = __ffs(conftmp) - 1; //find which index is first bit
				conftmp &= ~(1 << index);//set nth bit to 0
				shift[warpId][count] = index;
				shift[warpId+1][count] = index;
				count++;
			}
		}
		//two threads
	} else if(laneId < confpwarp) { //generate the shift arrays
		unsigned int index;
		unsigned int conftmp = conf[laneId][warpId];

		if(conftmp < lastval) {
			value[laneId][warpId] = f[conftmp];
		}
#pragma unroll
		for(int x = 0; x < card;x++) {//could put card in template to unroll
			index = __ffs(conftmp) - 1; //find which index is first bit
			conftmp &= ~(1 << index);//set nth bit to 0
			shift[warpId+laneId][x] = index;
		}

	}
	//__syncthreads();


	//Generate the initial splitting, if the second coalition is less than lastval do both else do the single one
	if(conf[1][warpId] < lastval) {
		unsigned int index;
		unsigned int splittmp= initsplit;
		leafsplit[0] = leafsplit[1] = 0;
		while(splittmp) {
			index = __ffs(splittmp)-1;
			leafsplit[1] += (1 << shift[warpId+1][index]);
			leafsplit[0] += (1 << shift[warpId][index]);//CHECK
			splittmp &= ~(1 << index);
		}
		//as we base our index from zero, do one run with nextsplit
		if((__popc(conf[1][warpId] & conf[0][warpId]) == (card - 1))) {
			leafsplit[0] = leafsplit[1] = SUBSET((conf[1][warpId] & conf[0][warpId]),leafsplit[1]);
		} else {
			leafsplit[1] = SUBSET(conf[1][warpId],leafsplit[1]);
			leafsplit[0] = SUBSET(conf[0][warpId],leafsplit[0]);
		}
	} else {
		unsigned int index;
		unsigned int splittmp= initsplit;
		leafsplit[0] = 0;
		while(splittmp) {
			index = __ffs(splittmp)-1;
			leafsplit[0] += (1 << shift[warpId][index]);//CHECK
			splittmp &= ~(1 << index);
		}
		leafsplit[0] = SUBSET(conf[0][warpId],leafsplit[0]);
	}

	if(!specsplittings) {
		return;
	}

	int x,y;

	//fetch splittings
	for(x = 0; x < specsplittings;x += parasplittings) {
	     //if there is a collision
		if((__popc(conf[1][warpId] & conf[0][warpId]) == (card - 1)) && (conf[1][warpId] < lastval)) {
#pragma unroll 8
		     for(y = 0;y < parasplittings ;y++) {
			  //reset value
				rvalue[0][y] = 	rvalue[1][y] = 0;
				if(x+y+initsplit < splittings) {
				     //IDP
					int tmp = __popc(leafsplit[0]);
					if((NAGENTS-card) <= tmp && (NAGENTS-card) <= (card-tmp)) {
					     //fetch the subset in common
						rvalue[1][y] = rvalue[0][y] = f[leafsplit[0]];
						//fetch the set difference
						rvalue[0][y] += f[setdiff(conf[0][warpId],leafsplit[0])];
						rvalue[1][y] += f[setdiff(conf[1][warpId],leafsplit[1])];
					}
					//nextsplit
					leafsplit[1] = SUBSET((conf[1][warpId] & conf[0][warpId]),leafsplit[1]);
					leafsplit[0] = SUBSET((conf[1][warpId] & conf[0][warpId]),leafsplit[0]);
				}
			}

		} else {			
#pragma unroll 8			
			for(y = 0;y < parasplittings ;y++) {
				rvalue[0][y] = rvalue[1][y] = 0;
				if(x+y+initsplit < splittings) {
				     //idp
					int tmp = __popc(leafsplit[0]);
					if((NAGENTS-card) <= tmp && (NAGENTS-card) <= (card-tmp)) {
						rvalue[0][y] = f[leafsplit[0]];
						rvalue[0][y] += f[setdiff(conf[0][warpId],leafsplit[0])];
						//template optimisation in order to remove one if statement from kernel launches which does not exceed
						//the maximum coalition structure
						if(overlastval) {
						if(conf[1][warpId] < lastval) {
							rvalue[1][y] = f[leafsplit[1]];	
							rvalue[1][y] += f[setdiff(conf[1][warpId],leafsplit[1])];
						}
						} else {
							rvalue[1][y] = f[leafsplit[1]];	
							rvalue[1][y] += f[setdiff(conf[1][warpId],leafsplit[1])];
						}
					}
					leafsplit[0] = SUBSET(conf[0][warpId],leafsplit[0]);
					leafsplit[1] = SUBSET(conf[1][warpId],leafsplit[1]);
					
				}
			}

		}

		//register reduction
#pragma unroll	       
		for(y = 1;y < parasplittings;y++) {
			if(x+y+initsplit >= splittings) {
				continue;
			}
			if(rvalue[0][0] < rvalue[0][y]) {
				rvalue[0][0] = rvalue[0][y];
			}
			if(rvalue[1][0] < rvalue[1][y]) {
				rvalue[1][0] = rvalue[1][y];
			}
		}

		//if their values are less than the value in shared memory, continue onto the next loop
		if(__ballot( ( (rvalue[0][0] > value[0][warpId]) || (rvalue[1][0] > value[1][warpId]) ) ) == 0) {
			continue;
		}

		//if rvalue is gfreater than value in shared memory, do warp reduction, this is a unrolled version
		if(__ballot( ( rvalue[0][0] > value[0][warpId] ) ) ) {
			rvalue[0][1] = __shfl_xor((int)rvalue[0][0],16,32);
			rvalue[0][0] = MAX(rvalue[0][0],rvalue[0][1]);
			rvalue[0][1] = __shfl_xor((int)rvalue[0][0],8,32);
			rvalue[0][0] = MAX(rvalue[0][0],rvalue[0][1]);
			rvalue[0][1] = __shfl_xor((int)rvalue[0][0],4,32);
			rvalue[0][0] = MAX(rvalue[0][0],rvalue[0][1]);
			rvalue[0][1] = __shfl_xor((int)rvalue[0][0],2,32);
			rvalue[0][0] = MAX(rvalue[0][0],rvalue[0][1]);
			rvalue[0][1] = __shfl_xor((int)rvalue[0][0],1,32);
			rvalue[0][0] = MAX(rvalue[0][0],rvalue[0][1]);
		}
		//same but different value
		if(__ballot( ( rvalue[1][0] > value[1][warpId] ) ) ) {
			if(conf[1][warpId] < lastval) {
			rvalue[1][1] = __shfl_xor((int)rvalue[1][0],16,32);
			rvalue[1][0] = MAX(rvalue[1][0],rvalue[1][1]);
			rvalue[1][1] = __shfl_xor((int)rvalue[1][0],8,32);
			rvalue[1][0] = MAX(rvalue[1][0],rvalue[1][1]);
			rvalue[1][1] = __shfl_xor((int)rvalue[1][0],4,32);
			rvalue[1][0] = MAX(rvalue[1][0],rvalue[1][1]);
			rvalue[1][1] = __shfl_xor((int)rvalue[1][0],2,32);
			rvalue[1][0] = MAX(rvalue[1][0],rvalue[1][1]);
			rvalue[1][1] = __shfl_xor((int)rvalue[1][0],1,32);
			rvalue[1][0] = MAX(rvalue[1][0],rvalue[1][1]);
			}
		}
		//update shared memory
		if(laneId == 0) {
			if(rvalue[0][0] > value[0][warpId]) {
				value[0][warpId] = rvalue[0][0];
			}
			if(rvalue[1][0] > value[1][warpId]) {
				value[1][warpId] = rvalue[1][0];
			}
		}
	}
	//lastly update global memory, no need for atomics as there is no contender
	if(laneId == 0) {
		if(conf[0][warpId] < lastval) {
			if(value[0][warpId] > f[conf[0][warpId]]) {
				f[conf[0][warpId]] = value[0][warpId];
			}
		}
		if(conf[1][warpId] < lastval) {
			if(value[1][warpId] > f[conf[1][warpId]]) {
				f[conf[1][warpId]] = value[1][warpId];
			}
		}
	}
	return;
}



#define COMBS(X) ((1 << cardinality(X)-1) - 1)

int run_test(unsigned int MAXVAL,dint items) {

	register unsigned int i,c1,count =0;
	unsigned short *dev_bids;

	count = 0;

	HANDLE_ERROR(hipDeviceReset());
	HANDLE_ERROR(hipSetDeviceFlags(hipDeviceScheduleYield));
  	HANDLE_ERROR(hipDeviceSetCacheConfig(hipFuncCachePreferL1));
	HANDLE_ERROR(hipDeviceSetLimit(hipLimitStackSize,0));
	HANDLE_ERROR(hipDeviceSetLimit(hipLimitMallocHeapSize,0));
	HANDLE_ERROR(hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte));

	register unsigned int streams = NSTREAMS;
	register unsigned int streamcount = 0;
	register hipStream_t stream[streams];
	for(i = 0;i < streams; i++)
		HANDLE_ERROR(hipStreamCreate(&stream[i]));

	count = 0;

 	HANDLE_ERROR(hipMalloc((void **)&dev_bids, MAXVAL*sizeof(short)));

 	HANDLE_ERROR(hipMemcpy(dev_bids,bids,MAXVAL*sizeof(short),hipMemcpyHostToDevice));

	register unsigned int bsize = MAXBLOCKSIZE;
	register int blocks;
	int prev =0;

	time_t rstart,rend,rt;
	rstart=clock();
	for(i = 2; i <= items; i++) {
		time_t start,end,t;

		start=clock();
		unsigned int splittings;
		blocks =4;//(int)  ceil((threads/bsize));
		double threads;
		c1 = (1 << i) -1;
		unsigned int c2 = c1;
		unsigned int c3;
		unsigned int ca[blocks];
		unsigned int cacount;
		splittings =  COMBS(c1);///NPERBLOCK;
		threads = ((double) splittings)/ NPERBLOCK;
		threads = ceil(threads);
		
		for(; c1 <= MAXVAL;) {

			cacount = 1;
			ca[0] = c1;			
			for(int x = 0, cacount = 1; x < CONFPKERNEL*blocks;x++) {
				t = c2 | (c2-1);
				c2 = (t + 1) | (((~t & -~t) - 1) >> (__builtin_ctz(c2) + 1));
				if(x%CONFPKERNEL == 0 && x > 1) {
					ca[cacount] = c2;
					cacount++;
				}
			}
	
			switch((c2 > MAXVAL)) {
			case 1:
				subsetcomp33 < MAXBLOCKSIZE , 1> <<<blocks,MAXBLOCKSIZE,0,stream[streamcount]>>>(dev_bids,splittings,MAXVAL,i,ca[0],ca[1],ca[2],ca[3]);
				break;
			case 0:
				subsetcomp33 < MAXBLOCKSIZE , 0> <<<blocks,MAXBLOCKSIZE,0,stream[streamcount]>>>(dev_bids,splittings,MAXVAL,i,ca[0],ca[1],ca[2],ca[3]);
				break;

			}
			c1 = c2;

			streamcount++;
			count++;

			if(streamcount >= streams)
				streamcount = 0;
		}
		

		for (int t = 0; t < streams; ++t) {
			HANDLE_ERROR(hipStreamSynchronize(stream[t]));
		}

		HANDLE_ERROR(hipDeviceSynchronize());


		end=clock();
		t=(end-start)/(CLOCKS_PER_SEC/1000);
		printf("ended card %d blocks\t %d threads/block %u, n kernels %u \t time %lu \t splittings %d time per kernel %u\n",i,blocks,bsize,count-prev,t,splittings,t/(count-prev));
		prev =	count;

	}
	for (int i = 0; i < streams; ++i)
		hipStreamDestroy(stream[i]);

	HANDLE_ERROR(hipDeviceSynchronize());

	rend=clock();
	rt=(rend-rstart)/(CLOCKS_PER_SEC/1000);
	printf("real time %lu\n",rt);

	HANDLE_ERROR(hipMemcpy(f,dev_bids,MAXVAL*sizeof(short),hipMemcpyDeviceToHost));

	HANDLE_ERROR(hipFree(dev_bids));

	HANDLE_ERROR(hipDeviceReset());

	return count;
}

dint max2(dint conf) {
     register dint card = cardinality(conf)/2;
     register dint combinations = (1 << cardinality(conf)-1)-1;
     register dint max = f[conf];
     register dint tmp = 0;
     register dint subset = 0;
     register const dint inverse = ~conf;
     register dint i;
     for(i = 1;i<=combinations; i++) {
	     subset = ((inverse+1)+subset)&conf;
	     tmp = f[setdiff(conf,subset)] + f[subset];
	     if(max == tmp) {
		     break;
		     //  return subset;
	     }

     }
     return subset;
}

int recur_parse_wopt(dint MAXVAL) {
	stack * root = (stack *) malloc(sizeof(stack));
	stack * sroot = NULL;
	stack * scurr = NULL;
		root->conf = (MAXVAL)-1;
	int count = 0;
	root->next = NULL;
	stack * curr = root;
	while(curr) {
		dint conf = curr->conf;
		if(conf == 0) {
			printf("EXIT FAILURE\n");
			return 1;
		}
		/*if something is wrong*/
		if(count > 40) {
			fprintf(stderr,"Something went wrong at line %d in %s\n",__LINE__,__FILE__);
			return 1;
		}
//		printf("curr %u\t\n",curr->conf);
		if(f[conf] != bids[conf]) {
			dint proper_subset = max2(conf);
			dint diff = setdiff(conf,proper_subset);
			curr->conf = proper_subset;
			stack * tmp = (stack *) malloc(sizeof(stack));
			//printf("diff %u\t conf %u\t O[diff] %u\t O[conf]\t f %u\n",diff,conf,O[diff],O[conf],f[conf]);
			tmp->conf = diff;
			tmp->next = curr;
			root = tmp;
			curr = root;
			count++;
			continue;
		}
		if(sroot == NULL) {
			sroot = curr;
			scurr = curr;
		} else {
			/*set next pointer to the next singelton*/
			scurr->next = curr;
		}

		/*set the current singleton to be the pointer*/
		scurr = curr;
		curr = curr->next;
		/*clear the pinter to avoid infinite loop if cross-referenced*/
		scurr->next = NULL;

	}
	printf("\n");
	curr = sroot;
	dint tmp = 0;
	while(curr != NULL) {
		if(bids[curr->conf]) {
			if(curr->conf == indexa) {
				printf("correct bid\n");
				tmp++;
				return 0;
			}
			//printf("conf %u value %u\n",curr->conf,bids[curr->conf]);

		}
		stack * tmp = curr;
		curr = curr->next;
		free(tmp);
	}
	if(tmp < 1) {
		printf("something is wrong, no bids\n");
		return 1;

	}
	printf("n = %u\n",tmp);
	return 0;
}

int main(void) {
	/*Start n amount of assets*/
	dint from = NAGENTS;
	dint MAXVAL = (2 << (from-1));

	time_t start,end,t;
//	O = (dint * ) malloc(sizeof(dint)*(2 << (from-1)));
	bids = (unsigned short * ) malloc(sizeof(short)*(2 << (from-1)));
	f = (unsigned short * ) malloc(sizeof(short)*(2 << (from-1)));
//	f = bids;
	int ret_val =0;
	int count;
	while(ret_val == 0) {

		MAXVAL = (2 << (from-1));
		gen_rand_bids(MAXVAL);
		set_singleton_bid(MAXVAL);
		printf("maxval %u from %u\n",MAXVAL,from);
		start=clock();//predefined  function in c
		count = run_test(MAXVAL,from);
		end=clock();
		t=(end-start)/CLOCKS_PER_SEC;
		ret_val= recur_parse_wopt(MAXVAL);// parse_wopt(MAXVAL);
		printf("\nTime taken =%lu for n= %u with count %d average per count %lf\n", (unsigned long) t,from,count,(double)t/count);
	}


	free(O);
	free(f);

	return 0;
}

