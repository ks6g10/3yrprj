#include "hip/hip_runtime.h"
#include <stdio.h> 
#include <string.h> // for ffs
#include <stdint.h>
#include <math.h>
#include <time.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

/*Debug enabled gives more print statements of bids and how the "Matrix" gets evaluated*/
#define DEBUG 0
#define TRUE 1 
#define FALSE 0
/*Test sets all bids to one, which should give you n=|ITEMS| bids on output*/
#define TEST 1
/*Defines from 0-Range the random will give out*/
#define RANGE 10000
#define ITEMS 25

#define MAX (2 << (ITEMS-1))
#if ITEMS < 8
#define dint uint8_t  
#elif ITEMS < 16
#undef dint
#define dint uint16_t
#elif ITEMS < 32
#undef dint
#define dint uint32_t
#elif ITEMS > 32
#undef dint
#define dint uint64_t
#endif

#define SUBSET(X)((~_conf+(X+1))&_conf)
#define SETSUM(X)(f[setdiff(_conf,X)]+f[X])


static void HandleError( hipError_t err, const char *file,int line ) {
	if (err != hipSuccess) {
		printf( "%s in %s at line %d\n", hipGetErrorString( err ),file, line );
		exit( EXIT_FAILURE );
	}
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))



/*		           0 1 2 3 4 5 6 7 8			*/
dint * bids;
// dint bids[MAX] ={0,2,3,6,4,7,6,9}; //conf 5 and 2
//dint bids[MAX] =  {0,2,3,6,4,6,6,9}; //conf 3 and 4
// dint bids[MAX] =  {0,20,3,6,4,6,10,9}; //conf 1 and 6
dint * f;
dint * O;
 
struct _stack {
	dint conf;
	struct _stack * next;
} typedef stack;

struct _locklist {
	unsigned int size;
	unsigned int conf;
	unsigned int * dev_f;
	unsigned int * dev_o;
	struct _locklist * next;
} typedef locklist;

struct _lockstruct {
	unsigned int * dev_lock;
	struct _lockstruct * next;
} typedef lockstruct;


#define setdiff(seta,setb) (seta & ~setb)

inline  dint cardinality( dint seta) {
	return __builtin_popcount(seta);
}

void gen_rand_bids(dint MAXVAL) {
	register dint i = 0;
#if TEST
	for(i = 1; i < MAXVAL;i++) {
		bids[i] = 1;
		O[i] = i;
	}
//	bids[1] =0;
//	bids[2] = 0;
//	bids[32769] = 20;
#else
	for(i = 1; i < MAXVAL;i++) {
		bids[i] = rand() % RANGE;
		O[i] = i;
	}
	for(i = 1; i < MAXVAL;i*=2) {
		bids[i] = rand() % RANGE;

	}
#endif

#if DEBUG
	printf("i =\t");
	for(i = 1; i < MAX;i++) {
		printf("%u\t",i);
	}
	printf("\n");
	printf("val =\t");
	for(i = 1; i < MAX;i++) {
		printf("%u\t",bids[i]);
	}
	printf("\n");
#endif
}

const char *btb(dint y)
{
	int x = y;
	static char b[9];
	b[0] = '\0';
	
	int z;
	for (z = 128; z > 0; z >>= 1)
	{
		strcat(b, ((x & z) == z) ? "1" : "0");
	}	
	return b;
}

inline void printfo() {
#if DEBUG
	dint i;
	printf("i\t");
	for(i =1; i < MAX; i++) {
		printf("%u\t",i);
	}
	printf("\n");
	printf("f[]\t");
	for(i =1; i < MAX; i++) {
		printf("%u\t",f[i]);
	}
	printf("\n");
	printf("O[]\t");
	for(i =1; i < MAX; i++) {
		printf("%u\t",O[i]);
	}
	printf("\n");
#endif
}

/*Sets all bids with one element in it, |n| = 1*/
inline void set_singleton_bid(dint MAXVAL) {
	register  dint i;
	for(i =1; i< MAXVAL; i*=2) {
		f[i] = bids[i];
		if(bids[i] > 0)
			O[i] = i;
	}
}

/* conf a e.g. 1101
 * (~a+i) & a gives a subset of a
 *  i is a integer from 1 to |a|
 *
 * ~1101 = 0010
 * i = 0001
 * (0010+0001)&1101 =
 * 0011&1101 = 0001
 *
 *i = 0011
 * (0010+0011)&1101
 *(0101)&1101 = 0101
 */



void printfo(dint MAXVAL) {
	int i;
	printf("\n");
	for(i = 1; i< MAXVAL; i++)
	{
		printf("Bid[%d]\t%u\tF[%d]\t%u\tO[%d]\t%u\tbin\t%s\n",i,bids[i],i,f[i],i,O[i],btb(i));

	}
}

void parse_wopt(dint MAXVAL) {
	printf("parse maxval = %u\n",MAXVAL);
	//wopt at start contain MAX at wopt[0] which is the combination that goes in bids[wopt[n]]
	stack * root =(stack *) malloc(sizeof(stack));
	stack * sroot = NULL;
	stack * scurr = NULL;
	//DO N	OT REMOVE -1
	root->conf = (MAXVAL)-1;
	int count = 0;
	root->next = NULL;
	stack * curr = root;
	while(curr) {
		dint conf = curr->conf;
		if(conf == 0) {
			printf("EXIT FAILURE\n");
			return;
		}
		/*if something is wrong*/
		if(count > 40) {
			fprintf(stderr,"Something went wrong at line %d in %s\n",__LINE__,__FILE__);
			return;
		}
			printf("curr %u\t",curr->conf);
		if(conf != O[conf]) {

			dint diff = setdiff(conf,O[conf]);
			curr->conf = O[conf];
			stack * tmp = (stack *) malloc(sizeof(stack));
			printf("diff %u\t O[conf] %u f %u\n",diff,O[conf],f[conf]);
			tmp->conf = diff;
			tmp->next = curr;
			root = tmp;
			curr = root;
			count++;
			continue;
		}
		if(sroot == NULL) {
			sroot = curr;
			scurr = curr;
		} else {
			/*set next pointer to the next singelton*/
			scurr->next = curr;
		}

		/*set the current singleton to be the pointer*/
		scurr = curr;
		curr = curr->next;
		/*clear the pinter to avoid infinite loop if cross-referenced*/
		scurr->next = NULL;

	}
	curr = sroot;
	dint tmp = 0;
	while(curr != NULL) {
		printf("conf %u value %u\n",curr->conf,bids[curr->conf]);
		tmp++;
		stack * tmp = curr;
		curr = curr->next;
		free(tmp);
	}
	printf("n = %u\n",tmp);
}

#define I (threadIdx.x + blockDim.x * blockIdx.x)
#define SET_TEST_FETCH(STEP,S1,S2) {					\
		S1 = S2 = 0U;						\
		STEP = SUBSET(ispec);					\
		if(__popc(STEP) <= cardmax && ispec <= maxval) {	\
			S1 = f[setdiff(_conf,STEP)];			\
			S2 = f[STEP];					\
		}							\
		ispec += inc;						\
	}

/* ispec += blockDim.x; change back if not working */

#define COMP_SET(V1,S1,V2,S2) {			\
		if(V1>V2) {			\
			V2 = V1;		\
			S2 = S1;		\
		}				\
	}
  

#define MAXBLOCKSIZE 256U
#define NAGENTS 23
#define NSTREAMS 16 
#define NPERBLOCK 8
#define HALFBLOCK 4
 
__global__ void subsetcomp22(
		 	     unsigned int * __restrict__ f, /*Bid value*/
			     unsigned int * __restrict__ O, /*The move array*/
			     unsigned int * __restrict__ lock,
			     unsigned int _conf, /*The configuration*/
			     unsigned int cardmax, /*cardinality of max allowance*/
			     unsigned int maxval,
			     unsigned int count,
			     unsigned int offset,
			     unsigned int defbid)
{
/*these arrays are shared between all threads in the same block */
	__shared__ unsigned int share[MAXBLOCKSIZE];
	__shared__ unsigned int step[MAXBLOCKSIZE];     

	unsigned int inc = gridDim.x*blockDim.x; //corrected the increment
	unsigned int ispec = I + offset;
//	int val11;
	int i;
	unsigned int val1[HALFBLOCK];//the value for one of the subset sums
	unsigned int val2[HALFBLOCK];//the value for the other subset sums
	unsigned int stept[HALFBLOCK]; // the step array
	step[threadIdx.x] = share[threadIdx.x] = 0U;
	if(ispec <= maxval) {

/*Local for the thread, check all its bid and pick the biggest*/
#pragma unroll 4
		for(i = 0; i < HALFBLOCK; i++) {				
			SET_TEST_FETCH(stept[i],val1[i],val2[i]);
		}
#pragma unroll 4
		for(i = 0; i < HALFBLOCK; i++) {		
			val1[i] += val2[i];
			COMP_SET(val1[i],stept[i],share[threadIdx.x],step[threadIdx.x]);			
			SET_TEST_FETCH(stept[i],val1[i],val2[i]); // pipelined fetch
		}

#pragma unroll 4
		for(i = 0; i < HALFBLOCK; i++) {		
			val1[i] += val2[i];
			COMP_SET(val1[i],stept[i],share[threadIdx.x],step[threadIdx.x]);
			//	SET_TEST_FETCH(stept[i],val1[i],val2[i]);
		}
	}

	ispec = I;
       
	i= blockDim.x >> 1;
	__syncthreads();
/*do max reduction on the shared array for all threads inside the block*/
#pragma unroll
	for (; i>0; i>>=1) {
		if (threadIdx.x < i && (ispec <= maxval)) {
			if(share[threadIdx.x] < share[threadIdx.x + i]) {
				step[threadIdx.x] = step[threadIdx.x+i];
				share[threadIdx.x] = share[threadIdx.x+i];
			}
		}
		__syncthreads();
	}

/*thread 0 will attempt to set to global memory the agreed maximum value inside the block,
* if it is greater than the original bid and the bid in the lock array
*/
	if(threadIdx.x == 0U) {
		i = share[0U];
		if(defbid => i)
			return;
		if(lock[count] < i) {
			if(atomicMax(&(lock[count]),i) < i) {
				O[_conf] = step[0U];
				f[_conf] = i; 
				
			} 
		}
	}
}



#define COMBS(X) ((1 << cardinality(X)) - 1)

int run_test(dint MAXVAL,dint items) {
/*Setup the environment*/
	//dint perm[MAXVAL];
	printfo();
	register unsigned int i, c,count =0;
 	unsigned int *dev_f,*dev_o;

	i = items/2;
	count = 0;

	HANDLE_ERROR(hipDeviceReset());

	unsigned int * dev_lock1,*dev_lock2,*dev_ptr;
	const	unsigned int devcount = 1024;// count;
	register unsigned int streams = NSTREAMS;
	register unsigned int count2 = 0;
	register unsigned int streamcount = 0;
	register hipStream_t stream[streams];
	for(int i = 0;i < streams; i++)
		HANDLE_ERROR(hipStreamCreate(&stream[i]));
	printf("count %u\n",devcount);
	count = 0;
	HANDLE_ERROR(hipMalloc((void **)&dev_lock1,(10+devcount)*sizeof(int)));
	HANDLE_ERROR(hipMalloc((void **)&dev_lock2,(10+devcount)*sizeof(int)));
 	HANDLE_ERROR(hipMalloc((void **)&dev_f, MAXVAL*sizeof(int)));
 	HANDLE_ERROR(hipMalloc((void **)&dev_o, MAXVAL*sizeof(int)));

 	HANDLE_ERROR(hipMemcpy(dev_f,bids,MAXVAL*sizeof(int),hipMemcpyHostToDevice));
 	HANDLE_ERROR(hipMemcpy(dev_o,O,MAXVAL*sizeof(int),hipMemcpyHostToDevice));

	HANDLE_ERROR(hipMemset(dev_lock1,0,devcount*sizeof(int)));
	HANDLE_ERROR(hipMemset(dev_lock2,0,devcount*sizeof(int)));
	/*2.*/
//	printfo(MAXVAL); printf("before\n");
	dev_ptr = dev_lock1;
	register unsigned int bsize = 0;
	register int blocks;
	int prev =0;
	count2 = 0;
	for(i = 2; i <= items; i++) {
		time_t start,end,t;
		start=clock();
		for(c = (1 << i) -1; c <= MAXVAL;) {

			double tmp = (double) COMBS(c)/NPERBLOCK;
			
			while( bsize < MAXBLOCKSIZE && tmp > bsize) {
				bsize += 32;
			}
			blocks =(int)  ceil((tmp/bsize));
/* #if __CUDA_ARCH__ < 300 */
/* 			int remaindern = blocks - 65535; */
/* 			while( blocks > 65535 ) { */
/* 				bsize += 32; */
/* 				blocks =(int)  ceil((tmp/bsize)); */
/* 			} */
/* 			printf("hello"); */
/* 			//double bsize = BLOCKSIZE; */
/* 			if(remaindern > 0) { */
/* 				blocks =65535; */
/* 				subsetcomp22<<<remaindern,bsize,0,stream[streamcount]>>>(dev_f,dev_o,dev_ptr,c,i/2,tmp,count2,65535*bsize,bids[c]); */
/* 			} */
/* #endif */
			subsetcomp22<<<blocks,bsize,0,stream[streamcount]>>>(dev_f,dev_o,dev_ptr,c,i/2,tmp,count2,0,bids[c]);
			//	printf("blocks %d block size %u stream count %d\n",blocks,bsize,streamcount);
			t = c | (c-1);
			c = (t + 1) | (((~t & -~t) - 1) >> (__builtin_ctz(c) + 1));
			count++;
			count2++;	
			streamcount++;
			if(streamcount >= streams)
				streamcount = 0;
			if(count2 < devcount)
				continue;
			HANDLE_ERROR(hipMemset(dev_ptr,0,devcount*sizeof(int)));

			if(dev_ptr == dev_lock1)
				dev_ptr = dev_lock2;
			else
				dev_ptr = dev_lock1;
			count2 = 0;
		}
		end=clock();
		t=(end-start)/CLOCKS_PER_SEC;
		printf("ended card %d blocks\t %d threads/block %u, n kernels %u \t time %lu\n",i,blocks,bsize,count-prev,t);
		prev =	count;
		for (int i = 0; i < streams; ++i)
			HANDLE_ERROR(hipStreamSynchronize(stream[i]));

		HANDLE_ERROR(hipDeviceSynchronize());
		printfo();
	}
	for (int i = 0; i < streams; ++i)
	hipStreamDestroy(stream[i]);

	HANDLE_ERROR(hipDeviceSynchronize());
	HANDLE_ERROR(hipMemcpy(f,dev_f,MAXVAL*sizeof(int),hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipMemcpy(O,dev_o,MAXVAL*sizeof(int),hipMemcpyDeviceToHost));
	//int i;
	HANDLE_ERROR(hipFree(dev_f));
	HANDLE_ERROR(hipFree(dev_o));
	HANDLE_ERROR(hipFree(dev_lock1));
	HANDLE_ERROR(hipFree(dev_lock2));

	HANDLE_ERROR(hipDeviceReset());
//	printfo(MAXVAL);
	return count;
}



int main(void) {
	/*Start n amount of assets*/
	dint from = NAGENTS;
	dint MAXVAL = (2 << (from-1));

	time_t start,end,t;
	O = (dint * ) malloc(sizeof(dint)*(2 << (from-1)));
	bids = (dint * ) malloc(sizeof(dint)*(2 << (from-1)));
	f = bids;


	MAXVAL = (2 << (from-1));
	gen_rand_bids(MAXVAL);
	set_singleton_bid(MAXVAL);
	printf("maxval %u from %u\n",MAXVAL,from);
	start=clock();//predefined  function in c
	int count = run_test(MAXVAL,from);
	end=clock();
	t=(end-start)/CLOCKS_PER_SEC;
	parse_wopt(MAXVAL);
	printf("\nTime taken =%lu for n= %u with count %d average per count %lf\n", (unsigned long) t,from,count,(double)t/count);


	free(O);
	free(f);

	return 0;
}
